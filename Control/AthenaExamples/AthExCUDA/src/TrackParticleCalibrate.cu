#include "hip/hip_runtime.h"
// Dear emacs, this is -*- c++ -*-
//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//

// Local include(s).
#include "TrackParticleCalibrate.h"

// CUDA include(s).
#include <hip/hip_runtime.h>

// System include(s).
#include <cassert>
#include <sstream>
#include <stdexcept>

namespace {

void throw_error(hipError_t errorCode, const char* expression,
                 const char* file, int line) {

   // Create a nice error message.
   std::ostringstream errorMsg;
   errorMsg << file << ":" << line << " Failed to execute: " << expression
            << " (" << hipGetErrorString(errorCode) << ")";

   // Now throw a runtime error with this message.
   throw std::runtime_error(errorMsg.str());
}

}  // private namespace

/// Helper macro used for checking @c hipError_t type return values.
#define CUDA_ERROR_CHECK(EXP)                                             \
    do {                                                                  \
        hipError_t errorCode = EXP;                                      \
        if (errorCode != hipSuccess) {                                   \
            ::throw_error(errorCode, #EXP, __FILE__, __LINE__);           \
        }                                                                 \
    } while (false)

namespace AthCUDAExamples {
namespace kernels {

/// Dummy kernel performing a trivial transformation on the track particle
/// parameters.
__global__
void trackParticleCalibrate(const TrackParticleContainer::const_view input_view,
                            TrackParticleContainer::view output_view) {

   // Get the current thread's index.
   const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

   // Create the device containers.
   TrackParticleContainer::const_device input(input_view);
   TrackParticleContainer::device output(output_view);
   assert(input.size() == output.size());

   // Check that the index is in range.
   if (index < input.size()) {
      // Copy the angle parameters as they are.
      output.theta()[index] = input.theta()[index];
      output.phi()[index] = input.phi()[index];

      // Transform the momentum in some silly way.
      output.qOverP()[index] =
         input.qOverP()[index] * std::abs((input.theta()[index] -
                                           input.phi()[index]) /
                                          input.phi()[index]);
   }

   return;
}


}  // namespace kernels

void calibrate(const TrackParticleContainer::const_view& input,
               TrackParticleContainer::view output) {

   // Launch the kernel.
   static const unsigned int block_size = 256;
   const unsigned int num_blocks =
      (input.capacity() + block_size - 1) / block_size;
   kernels::trackParticleCalibrate<<<num_blocks, block_size>>>(input, output);

   // Check for errors.
   CUDA_ERROR_CHECK(hipGetLastError());
   CUDA_ERROR_CHECK(hipDeviceSynchronize());
}

}  // namespace AthCUDAExamples
