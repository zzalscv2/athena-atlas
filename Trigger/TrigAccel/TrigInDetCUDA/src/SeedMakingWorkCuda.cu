/*
  Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "TrigAccelEvent/TrigInDetAccelEDM.h"
#include "SeedMakingWorkCuda.h"

#include "tbb/tick_count.h"
#include <cstring>
#include <cmath>
#include <iostream>
#include <algorithm>
#include "device_context.h" //for SeedMakingDeviceContext


#include "DoubletCountingKernelCuda.cuh"
#include "DoubletMakingKernelCuda.cuh"
#include "DoubletMatchingKernelCuda.cuh"

SeedMakingWorkCuda::SeedMakingWorkCuda(unsigned int id, SeedMakingDeviceContext* ctx, std::shared_ptr<TrigAccel::OffloadBuffer> data, 
  tbb::concurrent_vector<WorkTimeStamp>* TL) : 
  m_workId(id),
  m_context(ctx), 
  m_input(data),
  m_timeLine(TL)
 {
  
  m_output = std::make_shared<TrigAccel::OffloadBuffer>(sizeof(TrigAccel::OUTPUT_SEED_STORAGE));//output data
}

SeedMakingWorkCuda::~SeedMakingWorkCuda() {
  
  SeedMakingDeviceContext* p = m_context;

  int id = p->m_deviceId;

  hipSetDevice(id);

  hipStreamDestroy(p->m_stream);

  hipFree(p->d_settings);
  hipFree(p->d_spacepoints);
  
  hipFree(p->d_outputseeds);
  hipFree(p->d_doubletstorage);
  hipFree(p->d_doubletinfo);

  hipHostFree(p->h_settings);
  hipHostFree(p->h_spacepoints);
  hipHostFree(p->h_outputseeds);

  delete p;
  m_context = 0;
}

std::shared_ptr<TrigAccel::OffloadBuffer> SeedMakingWorkCuda::getOutput() {
  return m_output;
}

bool SeedMakingWorkCuda::run() {

  m_timeLine->push_back(WorkTimeStamp(m_workId, 0, tbb::tick_count::now()));

  const SeedMakingDeviceContext& p = *m_context;
  
  int id = p.m_deviceId;  
  
  TrigAccel::OUTPUT_SEED_STORAGE* ps = reinterpret_cast<TrigAccel::OUTPUT_SEED_STORAGE*>(p.h_outputseeds);
  
  hipSetDevice(id);

  checkError();
  
  hipMemcpyAsync(p.d_settings, p.h_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS), hipMemcpyHostToDevice, p.m_stream);

  checkError();

  hipMemcpyAsync(p.d_spacepoints, p.h_spacepoints, sizeof(TrigAccel::SPACEPOINT_STORAGE), hipMemcpyHostToDevice, p.m_stream);

  checkError();

  hipStreamSynchronize(p.m_stream);
    
  TrigAccel::SEED_FINDER_SETTINGS* dSettings  = reinterpret_cast<TrigAccel::SEED_FINDER_SETTINGS *>(p.d_settings);
  TrigAccel::SPACEPOINT_STORAGE* dSpacepoints = reinterpret_cast<TrigAccel::SPACEPOINT_STORAGE *>(p.d_spacepoints);
  TrigAccel::DETECTOR_MODEL* dDetModel        = reinterpret_cast<TrigAccel::DETECTOR_MODEL*>(p.d_detmodel);
  TrigAccel::OUTPUT_SEED_STORAGE* dOutput     = reinterpret_cast<TrigAccel::OUTPUT_SEED_STORAGE*>(p.d_outputseeds);

  DOUBLET_INFO* dInfo                         = reinterpret_cast<DOUBLET_INFO*>(p.d_doubletinfo);
  DOUBLET_STORAGE* dStorage                   = reinterpret_cast<DOUBLET_STORAGE*>(p.d_doubletstorage);

  hipMemset(p.d_outputseeds,0,10*sizeof(int));

  checkError();

  hipMemset(p.d_doubletstorage,0,3*sizeof(int));

  checkError();
  
  const TrigAccel::SPACEPOINT_STORAGE* pSPS = reinterpret_cast<const TrigAccel::SPACEPOINT_STORAGE *>(p.h_spacepoints);
  int nSlices = pSPS->m_nPhiSlices;
  int nLayers = pSPS->m_nLayers;
  
  int nMiddleSp = NUM_MIDDLE_THREADS;//determines size of the doublet/triplet buffers
  int nOtherSp = OUTER_THREADS_MULTIPLIER*p.m_gpuParams.m_nNUM_SMX_CORES/NUM_MIDDLE_THREADS;//the size of the spacepoint buffer

  dim3 gridDimensions(nSlices, nLayers);
  dim3 blockDimensions(nMiddleSp, nOtherSp);

  hipMemset(p.d_doubletinfo,0,sizeof(DOUBLET_INFO));

  checkError();

  hipStreamSynchronize(p.m_stream);

  checkError();

  doubletCountingKernel<<<gridDimensions, blockDimensions, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dInfo, nLayers, nSlices);

  hipStreamSynchronize(p.m_stream);

  checkError();

  doubletMakingKernel<<<gridDimensions, blockDimensions, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dOutput, 
    dInfo, dStorage, nLayers, nSlices);

  hipStreamSynchronize(p.m_stream);

  checkError();

  int nStats[3];

  hipMemcpy(&nStats[0], p.d_doubletstorage, 3*sizeof(int), hipMemcpyDeviceToHost);

  
  doubletMatchingKernel<<<p.m_gpuParams.m_nNUM_TRIPLET_BLOCKS, NUM_TRIPLET_THREADS, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dInfo, 
    dStorage,  dOutput, nStats[0]);

  hipStreamSynchronize(p.m_stream);

  checkError();

  TrigAccel::OUTPUT_SEED_STORAGE* pOutput = reinterpret_cast<TrigAccel::OUTPUT_SEED_STORAGE *>(m_output->m_rawBuffer);

  //Read back GPU results

  pOutput->m_nMiddleSps = 0;
  pOutput->m_nSeeds = 0;		
  pOutput->m_nI = 0;
  pOutput->m_nO = 0;
  pOutput->m_nCandidates = 0;

  hipMemcpyAsync(p.h_outputseeds, p.d_outputseeds, sizeof(TrigAccel::OUTPUT_SEED_STORAGE), hipMemcpyDeviceToHost, p.m_stream);

  checkError();
  
  hipStreamSynchronize(p.m_stream);

  checkError();

  memcpy(pOutput, ps, sizeof(TrigAccel::OUTPUT_SEED_STORAGE));
  
  m_timeLine->push_back(WorkTimeStamp(m_workId, 1, tbb::tick_count::now()));

  return true;
}

SeedMakingWorkCudaManaged::SeedMakingWorkCudaManaged(unsigned int id, SeedMakingManagedDeviceContext* ctx, std::shared_ptr<TrigAccel::OffloadBuffer> data, 
  tbb::concurrent_vector<WorkTimeStamp>* TL) : 
  m_workId(id),
  m_context(ctx), 
  m_input(data),
  m_timeLine(TL)
 {
  
  m_output = std::make_shared<TrigAccel::OffloadBuffer>(sizeof(TrigAccel::OUTPUT_SEED_STORAGE));//output data
}

SeedMakingWorkCudaManaged::~SeedMakingWorkCudaManaged() {
  
  SeedMakingManagedDeviceContext* p = m_context;

  int id = p->m_deviceId;

  hipSetDevice(id);

  hipStreamDestroy(p->m_stream);

  hipFree(p->m_settings);
  hipFree(p->m_spacepoints);
  
  hipFree(p->m_outputseeds);
  hipFree(p->d_doubletstorage);
  hipFree(p->d_doubletinfo);

  delete p;
  m_context = 0;
}

std::shared_ptr<TrigAccel::OffloadBuffer> SeedMakingWorkCudaManaged::getOutput() {
  return m_output;
}

bool SeedMakingWorkCudaManaged::run() {

  m_timeLine->push_back(WorkTimeStamp(m_workId, 0, tbb::tick_count::now()));

  const SeedMakingManagedDeviceContext& p = *m_context;
  
  int id = p.m_deviceId;  
  
  TrigAccel::OUTPUT_SEED_STORAGE* ps = reinterpret_cast<TrigAccel::OUTPUT_SEED_STORAGE*>(p.m_outputseeds);
  
  hipSetDevice(id);

  checkError();
  


  hipMemPrefetchAsync(p.m_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS), id, p.m_stream);

  checkError();



  hipMemPrefetchAsync(p.m_spacepoints, sizeof(TrigAccel::SPACEPOINT_STORAGE), id, p.m_stream);

  checkError();

  hipStreamSynchronize(p.m_stream);
    
  TrigAccel::SEED_FINDER_SETTINGS* dSettings  = reinterpret_cast<TrigAccel::SEED_FINDER_SETTINGS *>(p.m_settings);
  TrigAccel::SPACEPOINT_STORAGE* dSpacepoints = reinterpret_cast<TrigAccel::SPACEPOINT_STORAGE *>(p.m_spacepoints);
  TrigAccel::DETECTOR_MODEL* dDetModel        = reinterpret_cast<TrigAccel::DETECTOR_MODEL*>(p.d_detmodel);
  TrigAccel::OUTPUT_SEED_STORAGE* dOutput     = reinterpret_cast<TrigAccel::OUTPUT_SEED_STORAGE*>(p.m_outputseeds);

  DOUBLET_INFO* dInfo                         = reinterpret_cast<DOUBLET_INFO*>(p.d_doubletinfo);
  DOUBLET_STORAGE* dStorage                   = reinterpret_cast<DOUBLET_STORAGE*>(p.d_doubletstorage);

  hipMemset(p.m_outputseeds,0,10*sizeof(int));

  checkError();

  hipMemset(p.d_doubletstorage,0,3*sizeof(int));

  checkError();
  
  const TrigAccel::SPACEPOINT_STORAGE* pSPS = reinterpret_cast<const TrigAccel::SPACEPOINT_STORAGE *>(p.m_spacepoints);
  int nSlices = pSPS->m_nPhiSlices;
  int nLayers = pSPS->m_nLayers;
  
  int nMiddleSp = NUM_MIDDLE_THREADS;//determines size of the doublet/triplet buffers
  int nOtherSp = OUTER_THREADS_MULTIPLIER*p.m_gpuParams.m_nNUM_SMX_CORES/NUM_MIDDLE_THREADS;//the size of the spacepoint buffer

  dim3 gridDimensions(nSlices, nLayers);
  dim3 blockDimensions(nMiddleSp, nOtherSp);

  hipMemset(p.d_doubletinfo,0,sizeof(DOUBLET_INFO));

  checkError();

  hipStreamSynchronize(p.m_stream);

  checkError();

  doubletCountingKernel<<<gridDimensions, blockDimensions, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dInfo, nLayers, nSlices);

  hipStreamSynchronize(p.m_stream);

  checkError();

  doubletMakingKernel<<<gridDimensions, blockDimensions, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dOutput, 
    dInfo, dStorage, nLayers, nSlices);

  hipStreamSynchronize(p.m_stream);

  checkError();

  int nStats[3];

  hipMemcpy(&nStats[0], p.d_doubletstorage, 3*sizeof(int), hipMemcpyDeviceToHost);

  
  doubletMatchingKernel<<<p.m_gpuParams.m_nNUM_TRIPLET_BLOCKS, NUM_TRIPLET_THREADS, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dInfo, 
    dStorage,  dOutput, nStats[0]);

  hipStreamSynchronize(p.m_stream);

  checkError();

  TrigAccel::OUTPUT_SEED_STORAGE* pOutput = reinterpret_cast<TrigAccel::OUTPUT_SEED_STORAGE *>(m_output->m_rawBuffer);

  //Read back GPU results

  pOutput->m_nMiddleSps = 0;
  pOutput->m_nSeeds = 0;		
  pOutput->m_nI = 0;
  pOutput->m_nO = 0;
  pOutput->m_nCandidates = 0;




  


  checkError();

  memcpy(pOutput, ps, sizeof(TrigAccel::OUTPUT_SEED_STORAGE));
  
  m_timeLine->push_back(WorkTimeStamp(m_workId, 1, tbb::tick_count::now()));

  return true;
}
