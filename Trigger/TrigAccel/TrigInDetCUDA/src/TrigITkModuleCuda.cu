/*
  Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <atomic>

#include "device_context.h"
#include "TrigITkModuleCuda.h"
#include "SeedMakingDataStructures_ITk.h"
#include "SeedMakingWorkCuda_ITk.h"

#include "TrigAccelEvent/TrigInDetAccelCodes.h"
#include "gpu_helpers.h"

#include <sstream>


TrigITkModuleCuda::TrigITkModuleCuda() : m_maxDevice(0), m_dumpTimeLine(false) {

  m_h_detmodel = 0;

  m_maxDevice = GPUHelpers::getNumberOfGPUs();

  for(unsigned int i=0;i<getProvidedAlgs().size();i++) {
    m_workItemCounters[i] = 0;
  }
  
  m_h_detmodel = (unsigned char*) malloc(sizeof(TrigAccel::ITk::DETECTOR_MODEL));

  m_timeLine.clear();

}

TrigITkModuleCuda::~TrigITkModuleCuda() {
  
  free(m_h_detmodel);
  m_h_detmodel = 0;
  if(m_dumpTimeLine) {
    if(m_timeLine.size() > 0) {
       tbb::tick_count t0 = m_timeLine[0].m_time;
       std::ostringstream fileName;
       fileName <<"timeLine_"<<getpid()<<".csv";
       std::ofstream tl(fileName.str());
       tl<<"workId,eventType,time"<<std::endl;
       tl<<m_timeLine[0].m_workId<<","<<m_timeLine[0].m_eventType<<",0"<<std::endl;
       for(unsigned int tIdx = 1;tIdx < m_timeLine.size();++tIdx) {
          tbb::tick_count t1 = m_timeLine[tIdx].m_time;
          auto duration = t1-t0;
          tl<<m_timeLine[tIdx].m_workId<<","<<m_timeLine[tIdx].m_eventType<<","<<1000*duration.seconds()<<std::endl;
       }
       tl.close();
      m_timeLine.clear();
    }
 }
}

bool TrigITkModuleCuda::configure() {

  if(m_maxDevice == 0) {
     std::cout<<"No CUDA devices found"<<std::endl;
     return false;
  }
  return true;
}

SeedMakingDeviceContext* TrigITkModuleCuda::createSeedMakingContext(int id) const {

  hipSetDevice(id);
  checkError(11);
  SeedMakingDeviceContext* p = new SeedMakingDeviceContext;

  p->m_deviceId = id;

  //set stream

  hipStreamCreate(&p->m_stream);
  checkError(12);
  //check device property

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, id);

  p->m_gpuParams.m_nSMX = deviceProp.multiProcessorCount;

  int ncores = GPUHelpers::getNumberOfCores(deviceProp.major, deviceProp.minor);
  
  p->m_gpuParams.m_nNUM_SMX_CORES = ncores;//_ConvertSMVer2Cores_local(deviceProp.major, deviceProp.minor);
  p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = NUM_TRIPLET_BLOCKS_ITk;
  if(deviceProp.maxThreadsPerBlock < p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS) 
    p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = deviceProp.maxThreadsPerBlock;

  //Allocate memory
  
  hipMalloc((void **)&p->d_settings,    sizeof(TrigAccel::ITk::SEED_FINDER_SETTINGS));
  hipMalloc((void **)&p->d_spacepoints, sizeof(TrigAccel::ITk::SPACEPOINT_STORAGE));
  hipMalloc((void **)&p->d_detmodel,    sizeof(TrigAccel::ITk::DETECTOR_MODEL));
  checkError();
  hipMalloc((void **)&p->d_outputseeds, sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE));
  hipMalloc((void **)&p->d_doubletstorage, sizeof(DOUBLET_STORAGE_ITk));
  hipMalloc((void **)&p->d_doubletinfo, sizeof(DOUBLET_INFO_ITk));
  checkError(13);

  p->d_size = sizeof(TrigAccel::ITk::SEED_FINDER_SETTINGS) +  
              sizeof(TrigAccel::ITk::SPACEPOINT_STORAGE) + sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE) + sizeof(DOUBLET_STORAGE_ITk) + sizeof(DOUBLET_INFO_ITk) + 
              sizeof(TrigAccel::ITk::DETECTOR_MODEL);
  
  hipHostMalloc((void **)&p->h_settings, sizeof(TrigAccel::ITk::SEED_FINDER_SETTINGS));
  hipHostMalloc((void **)&p->h_spacepoints, sizeof(TrigAccel::ITk::SPACEPOINT_STORAGE));
  hipHostMalloc((void **)&p->h_outputseeds, sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE));

  p->h_size = sizeof(TrigAccel::ITk::SEED_FINDER_SETTINGS) + sizeof(TrigAccel::ITk::SPACEPOINT_STORAGE) + sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE);
  
  checkError(14);
  return p;
}

SeedMakingManagedDeviceContext* TrigITkModuleCuda::createManagedSeedMakingContext(int id) const {

  hipSetDevice(id);
  checkError(11);
  SeedMakingManagedDeviceContext* p = new SeedMakingManagedDeviceContext;

  p->m_deviceId = id;

  //set stream

  hipStreamCreate(&p->m_stream);
  checkError(12);
  //check device property

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, id);

  p->m_gpuParams.m_nSMX = deviceProp.multiProcessorCount;

  int ncores = GPUHelpers::getNumberOfCores(deviceProp.major, deviceProp.minor);

  p->m_gpuParams.m_nNUM_SMX_CORES = ncores;//_ConvertSMVer2Cores_local(deviceProp.major, deviceProp.minor);
  p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = NUM_TRIPLET_BLOCKS_ITk;
  if(deviceProp.maxThreadsPerBlock < p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS) 
    p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = deviceProp.maxThreadsPerBlock;

  //Allocate memory
  
  hipMallocManaged((void **)&p->m_settings,    sizeof(TrigAccel::ITk::SEED_FINDER_SETTINGS));
  hipMallocManaged((void **)&p->m_spacepoints, sizeof(TrigAccel::ITk::SPACEPOINT_STORAGE));
  hipMallocManaged((void **)&p->m_outputseeds, sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE));

  hipMalloc((void **)&p->d_detmodel,    sizeof(TrigAccel::ITk::DETECTOR_MODEL));
  checkError();
  hipMalloc((void **)&p->d_doubletstorage, sizeof(DOUBLET_STORAGE_ITk));
  hipMalloc((void **)&p->d_doubletinfo, sizeof(DOUBLET_INFO_ITk));
  checkError(13);

  p->d_size = sizeof(DOUBLET_STORAGE_ITk) + sizeof(DOUBLET_INFO_ITk) + sizeof(TrigAccel::ITk::DETECTOR_MODEL);
  
  p->h_size = 0;
  
  p->m_size = sizeof(TrigAccel::ITk::SEED_FINDER_SETTINGS) + sizeof(TrigAccel::ITk::SPACEPOINT_STORAGE) + sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE);

  checkError(14);
  return p;
}

TrigAccel::Work* TrigITkModuleCuda::createWork(int workType, std::shared_ptr<TrigAccel::OffloadBuffer> data){
  
  if(workType == TrigAccel::InDetJobControlCode::SIL_LAYERS_EXPORT){

    memcpy(m_h_detmodel, (unsigned char*)data->get(), sizeof(TrigAccel::ITk::DETECTOR_MODEL));

    return 0;
  }

  if(workType == TrigAccel::InDetJobControlCode::FIND_SEEDS){
 
    int deviceId = 0;//always using device 0 for the time being

    //TO-DO: to support mult-GPU load balancing get deviceId from a tbb_concurrent_queue

    SeedMakingDeviceContext* ctx = createSeedMakingContext(deviceId);

    hipMemcpy(ctx->d_detmodel, m_h_detmodel, sizeof(TrigAccel::ITk::DETECTOR_MODEL), hipMemcpyHostToDevice);
    checkError(21);
    TrigAccel::ITk::SEED_MAKING_JOB *pArray = reinterpret_cast<TrigAccel::ITk::SEED_MAKING_JOB*>(data->get());
    
    //1. copy settings to the context host array

    TrigAccel::ITk::SEED_FINDER_SETTINGS* p_settings = reinterpret_cast<TrigAccel::ITk::SEED_FINDER_SETTINGS*>(ctx->h_settings);
    memcpy(p_settings, &pArray->m_settings, sizeof(TrigAccel::ITk::SEED_FINDER_SETTINGS));

    //2. copy spacepoints to the context host array

    TrigAccel::ITk::SPACEPOINT_STORAGE* p_spacePoints = reinterpret_cast<TrigAccel::ITk::SPACEPOINT_STORAGE*>(ctx->h_spacepoints);
    memcpy(p_spacePoints, &pArray->m_data, sizeof(TrigAccel::ITk::SPACEPOINT_STORAGE));
    
    unsigned int workNum = m_workItemCounters[0]++;//seed making uses counter #0
    
    unsigned int workId = workNum*100;
    
    SeedMakingWorkCudaITk* w = new SeedMakingWorkCudaITk(workId, ctx, data, &m_timeLine);
    
    return w;
  }

  if(workType == TrigAccel::InDetJobControlCode::MAKE_SEEDS){
 
    int deviceId = 0;//always using device 0 for the time being

    //TO-DO: to support mult-GPU load balancing get deviceId from a tbb_concurrent_queue

    SeedMakingManagedDeviceContext* ctx = createManagedSeedMakingContext(deviceId);

    hipMemcpy(ctx->d_detmodel, m_h_detmodel, sizeof(TrigAccel::ITk::DETECTOR_MODEL), hipMemcpyHostToDevice);//TO-DO: try CoW here
    checkError(21);
    TrigAccel::ITk::SEED_MAKING_JOB *pArray = reinterpret_cast<TrigAccel::ITk::SEED_MAKING_JOB*>(data->get());
    
    //1. copy settings to the context host array

    TrigAccel::ITk::SEED_FINDER_SETTINGS* p_settings = reinterpret_cast<TrigAccel::ITk::SEED_FINDER_SETTINGS*>(ctx->m_settings);
    memcpy(p_settings, &pArray->m_settings, sizeof(TrigAccel::ITk::SEED_FINDER_SETTINGS));

    //2. copy spacepoints to the context host array

    TrigAccel::ITk::SPACEPOINT_STORAGE* p_spacePoints = reinterpret_cast<TrigAccel::ITk::SPACEPOINT_STORAGE*>(ctx->m_spacepoints);
    memcpy(p_spacePoints, &pArray->m_data, sizeof(TrigAccel::ITk::SPACEPOINT_STORAGE));
    
    unsigned int workNum = m_workItemCounters[0]++;//seed making uses counter #0
    
    unsigned int workId = workNum*100;
    
    SeedMakingWorkCudaManagedITk* w = new SeedMakingWorkCudaManagedITk(workId, ctx, data, &m_timeLine);
    
    return w;
  }

  return 0;
}

const std::vector<int> TrigITkModuleCuda::getProvidedAlgs(){
  std::vector<int> v{
      TrigAccel::InDetJobControlCode::SIL_LAYERS_EXPORT,
      TrigAccel::InDetJobControlCode::MAKE_SEEDS,//the default
      TrigAccel::InDetJobControlCode::FIND_SEEDS //the alternative
  };
  return v;
}

