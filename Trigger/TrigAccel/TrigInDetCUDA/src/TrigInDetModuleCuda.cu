/*
  Copyright (C) 2002-2020 CERN for the benefit of the ATLAS collaboration
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <atomic>

#include "TrigInDetModuleCuda.h"
#include "SeedMakingDataStructures.h"
#include "SeedMakingWorkCuda.h"

#include "TrigAccelEvent/TrigInDetAccelCodes.h"

#include <sstream>

extern "C" TrigAccel::WorkFactory* getFactory() {
  return new TrigInDetModuleCuda();
}

extern "C" int getFactoryId() { 
  return TrigAccel::TrigInDetModuleID_CUDA; 
} 	  

extern "C" void deleteFactory(TrigAccel::WorkFactory* c){
  TrigInDetModuleCuda* mod=reinterpret_cast<TrigInDetModuleCuda*>(c);
  delete mod;
}

void TrigInDetModuleCuda::getNumberOfGPUs() {
  pid_t childpid;
  int fd[2];
  // create pipe descriptors
  pipe(fd);

  childpid = fork();
  if(childpid != 0) {  // parent
    close(fd[1]);
    // read the data (blocking operation)
    read(fd[0], &m_maxDevice, sizeof(m_maxDevice));
    // close the read-descriptor
    close(fd[0]);
  }
  else {  // child
    // writing only, no need for read-descriptor
    close(fd[0]);
    int maxDevice = 0;
    hipGetDeviceCount(&maxDevice);
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
      maxDevice = 0;
      std::cout << "ERROR: " << hipGetErrorString( error ) << std::endl;
    }
    // send the value on the write-descriptor
    write(fd[1], &maxDevice, sizeof(maxDevice)); 
    // close the write descriptor
    close(fd[1]);
    exit(0);
  }
}

TrigInDetModuleCuda::TrigInDetModuleCuda() : m_maxDevice(0), m_dumpTimeLine(false) {

  m_h_detmodel = 0;

  getNumberOfGPUs();

  for(unsigned int i=0;i<getProvidedAlgs().size();i++) {
    m_workItemCounters[i] = 0;
  }
  
  m_h_detmodel = (unsigned char*) malloc(sizeof(TrigAccel::DETECTOR_MODEL));

  m_timeLine.clear();

}

TrigInDetModuleCuda::~TrigInDetModuleCuda() {
  
  free(m_h_detmodel);
  m_h_detmodel = 0;
  if(m_dumpTimeLine) {
    if(m_timeLine.size() > 0) {
       tbb::tick_count t0 = m_timeLine[0].m_time;
       std::ostringstream fileName;
       fileName <<"timeLine_"<<getpid()<<".csv";
       std::ofstream tl(fileName.str());
       tl<<"workId,eventType,time"<<std::endl;
       tl<<m_timeLine[0].m_workId<<","<<m_timeLine[0].m_eventType<<",0"<<std::endl;
       for(unsigned int tIdx = 1;tIdx < m_timeLine.size();++tIdx) {
          tbb::tick_count t1 = m_timeLine[tIdx].m_time;
          auto duration = t1-t0;
          tl<<m_timeLine[tIdx].m_workId<<","<<m_timeLine[tIdx].m_eventType<<","<<1000*duration.seconds()<<std::endl;
       }
       tl.close();
      m_timeLine.clear();
    }
 }
}

bool TrigInDetModuleCuda::configure() {

  if(m_maxDevice == 0) {
     std::cout<<"No CUDA devices found"<<std::endl;
     return false;
  }
  return true;
}

int TrigInDetModuleCuda::getNumberOfCores(int major, int minor) const {
    
    int ncores = 0;
    
    if ((major == 7) && (minor == 5)) {
       ncores = 64;//Turing
    }
    if ((minor == 1) || (minor == 2)) ncores = 128;
    else if (minor == 0) ncores = 64;
    else if ((major == 8) && (minor == 6) ){
       ncores = 32;
    }

    if(ncores == 0) {
       printf("Cannot determine the number of cores: unknown device type, major=%d minor=%d\n", major, minor);
    }
    return ncores;
}

SeedMakingDeviceContext* TrigInDetModuleCuda::createSeedMakingContext(int id) const {

  hipSetDevice(id);
  checkError(11);
  SeedMakingDeviceContext* p = new SeedMakingDeviceContext;

  p->m_deviceId = id;

  //set stream

  hipStreamCreate(&p->m_stream);
  checkError(12);
  //check device property

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, id);

  p->m_gpuParams.m_nSMX = deviceProp.multiProcessorCount;

  int ncores = getNumberOfCores(deviceProp.major, deviceProp.minor);
  
  p->m_gpuParams.m_nNUM_SMX_CORES = ncores;//_ConvertSMVer2Cores_local(deviceProp.major, deviceProp.minor);
  p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = NUM_TRIPLET_BLOCKS;
  if(deviceProp.maxThreadsPerBlock < p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS) 
    p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = deviceProp.maxThreadsPerBlock;

  //Allocate memory
  
  hipMalloc((void **)&p->d_settings,    sizeof(TrigAccel::SEED_FINDER_SETTINGS));
  hipMalloc((void **)&p->d_spacepoints, sizeof(TrigAccel::SPACEPOINT_STORAGE));
  hipMalloc((void **)&p->d_detmodel,    sizeof(TrigAccel::DETECTOR_MODEL));
  checkError();
  hipMalloc((void **)&p->d_outputseeds, sizeof(TrigAccel::OUTPUT_SEED_STORAGE));
  hipMalloc((void **)&p->d_doubletstorage, sizeof(DOUBLET_STORAGE));
  hipMalloc((void **)&p->d_doubletinfo, sizeof(DOUBLET_INFO));
  checkError(13);

  p->d_size = sizeof(TrigAccel::SEED_FINDER_SETTINGS) +  
              sizeof(TrigAccel::SPACEPOINT_STORAGE) + sizeof(TrigAccel::OUTPUT_SEED_STORAGE) + sizeof(DOUBLET_STORAGE) + sizeof(DOUBLET_INFO) + 
              sizeof(TrigAccel::DETECTOR_MODEL);
  
  hipHostMalloc((void **)&p->h_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS));
  hipHostMalloc((void **)&p->h_spacepoints, sizeof(TrigAccel::SPACEPOINT_STORAGE));
  hipHostMalloc((void **)&p->h_outputseeds, sizeof(TrigAccel::OUTPUT_SEED_STORAGE));

  p->h_size = sizeof(TrigAccel::SEED_FINDER_SETTINGS) + sizeof(TrigAccel::SPACEPOINT_STORAGE) + sizeof(TrigAccel::OUTPUT_SEED_STORAGE);
  
  checkError(14);
  return p;
}

SeedMakingManagedDeviceContext* TrigInDetModuleCuda::createManagedSeedMakingContext(int id) const {

  hipSetDevice(id);
  checkError(11);
  SeedMakingManagedDeviceContext* p = new SeedMakingManagedDeviceContext;

  p->m_deviceId = id;

  //set stream

  hipStreamCreate(&p->m_stream);
  checkError(12);
  //check device property

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, id);

  p->m_gpuParams.m_nSMX = deviceProp.multiProcessorCount;

  int ncores = getNumberOfCores(deviceProp.major, deviceProp.minor);

  p->m_gpuParams.m_nNUM_SMX_CORES = ncores;//_ConvertSMVer2Cores_local(deviceProp.major, deviceProp.minor);
  p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = NUM_TRIPLET_BLOCKS;
  if(deviceProp.maxThreadsPerBlock < p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS) 
    p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = deviceProp.maxThreadsPerBlock;

  //Allocate memory
  
  hipMallocManaged((void **)&p->m_settings,    sizeof(TrigAccel::SEED_FINDER_SETTINGS));
  hipMallocManaged((void **)&p->m_spacepoints, sizeof(TrigAccel::SPACEPOINT_STORAGE));
  hipMallocManaged((void **)&p->m_outputseeds, sizeof(TrigAccel::OUTPUT_SEED_STORAGE));

  hipMalloc((void **)&p->d_detmodel,    sizeof(TrigAccel::DETECTOR_MODEL));
  checkError();
  hipMalloc((void **)&p->d_doubletstorage, sizeof(DOUBLET_STORAGE));
  hipMalloc((void **)&p->d_doubletinfo, sizeof(DOUBLET_INFO));
  checkError(13);

  p->d_size = sizeof(DOUBLET_STORAGE) + sizeof(DOUBLET_INFO) + sizeof(TrigAccel::DETECTOR_MODEL);
  
  p->h_size = 0;
  
  p->m_size = sizeof(TrigAccel::SEED_FINDER_SETTINGS) + sizeof(TrigAccel::SPACEPOINT_STORAGE) + sizeof(TrigAccel::OUTPUT_SEED_STORAGE);

  checkError(14);
  return p;
}

TrigAccel::Work* TrigInDetModuleCuda::createWork(int workType, std::shared_ptr<TrigAccel::OffloadBuffer> data){
  
  if(workType == TrigAccel::InDetJobControlCode::SIL_LAYERS_EXPORT){

    memcpy(m_h_detmodel, (unsigned char*)data->get(), sizeof(TrigAccel::DETECTOR_MODEL));

    return 0;
  }

  if(workType == TrigAccel::InDetJobControlCode::FIND_SEEDS){
 
    int deviceId = 0;//always using device 0 for the time being

    //TO-DO: to support mult-GPU load balancing get deviceId from a tbb_concurrent_queue

    SeedMakingDeviceContext* ctx = createSeedMakingContext(deviceId);

    hipMemcpy(ctx->d_detmodel, m_h_detmodel, sizeof(TrigAccel::DETECTOR_MODEL), hipMemcpyHostToDevice);
    checkError(21);
    TrigAccel::SEED_MAKING_JOB *pArray = reinterpret_cast<TrigAccel::SEED_MAKING_JOB*>(data->get());
    
    //1. copy settings to the context host array

    TrigAccel::SEED_FINDER_SETTINGS* p_settings = reinterpret_cast<TrigAccel::SEED_FINDER_SETTINGS*>(ctx->h_settings);
    memcpy(p_settings, &pArray->m_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS));

    //2. copy spacepoints to the context host array

    TrigAccel::SPACEPOINT_STORAGE* p_spacePoints = reinterpret_cast<TrigAccel::SPACEPOINT_STORAGE*>(ctx->h_spacepoints);
    memcpy(p_spacePoints, &pArray->m_data, sizeof(TrigAccel::SPACEPOINT_STORAGE));
    
    unsigned int workNum = m_workItemCounters[0]++;//seed making uses counter #0
    
    unsigned int workId = workNum*100;
    
    SeedMakingWorkCuda* w = new SeedMakingWorkCuda(workId, ctx, data, &m_timeLine);
    
    return w;
  }

  if(workType == TrigAccel::InDetJobControlCode::MAKE_SEEDS){
 
    int deviceId = 0;//always using device 0 for the time being

    //TO-DO: to support mult-GPU load balancing get deviceId from a tbb_concurrent_queue

    SeedMakingManagedDeviceContext* ctx = createManagedSeedMakingContext(deviceId);

    hipMemcpy(ctx->d_detmodel, m_h_detmodel, sizeof(TrigAccel::DETECTOR_MODEL), hipMemcpyHostToDevice);//TO-DO: try CoW here
    checkError(21);
    TrigAccel::SEED_MAKING_JOB *pArray = reinterpret_cast<TrigAccel::SEED_MAKING_JOB*>(data->get());
    
    //1. copy settings to the context host array

    TrigAccel::SEED_FINDER_SETTINGS* p_settings = reinterpret_cast<TrigAccel::SEED_FINDER_SETTINGS*>(ctx->m_settings);
    memcpy(p_settings, &pArray->m_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS));

    //2. copy spacepoints to the context host array

    TrigAccel::SPACEPOINT_STORAGE* p_spacePoints = reinterpret_cast<TrigAccel::SPACEPOINT_STORAGE*>(ctx->m_spacepoints);
    memcpy(p_spacePoints, &pArray->m_data, sizeof(TrigAccel::SPACEPOINT_STORAGE));
    
    unsigned int workNum = m_workItemCounters[0]++;//seed making uses counter #0
    
    unsigned int workId = workNum*100;
    
    SeedMakingWorkCudaManaged* w = new SeedMakingWorkCudaManaged(workId, ctx, data, &m_timeLine);
    
    return w;
  }

  return 0;
}

const std::vector<int> TrigInDetModuleCuda::getProvidedAlgs(){
  std::vector<int> v{
      TrigAccel::InDetJobControlCode::SIL_LAYERS_EXPORT,
      TrigAccel::InDetJobControlCode::MAKE_SEEDS,//the default
      TrigAccel::InDetJobControlCode::FIND_SEEDS //the alternative
  };
  return v;
}

