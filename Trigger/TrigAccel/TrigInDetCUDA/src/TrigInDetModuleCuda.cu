/*
  Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <atomic>

#include "TrigInDetModuleCuda.h"
#include "SeedMakingDataStructures.h"
#include "SeedMakingWorkCuda.h"

#include "TrigAccelEvent/TrigInDetAccelCodes.h"

#include <sstream>
#include "gpu_helpers.h"

TrigInDetModuleCuda::TrigInDetModuleCuda() : m_maxDevice(0), m_dumpTimeLine(false) {

  m_h_detmodel = 0;

  m_maxDevice = GPUHelpers::getNumberOfGPUs();

  for(unsigned int i=0;i<getProvidedAlgs().size();i++) {
    m_workItemCounters[i] = 0;
  }

  m_timeLine.clear();

}

TrigInDetModuleCuda::~TrigInDetModuleCuda() {
  if(m_dumpTimeLine) {
    if(m_timeLine.size() > 0) {
       tbb::tick_count t0 = m_timeLine[0].m_time;
       std::ostringstream fileName;
       fileName <<"timeLine_"<<getpid()<<".csv";
       std::ofstream tl(fileName.str());
       tl<<"workId,eventType,time"<<std::endl;
       tl<<m_timeLine[0].m_workId<<","<<m_timeLine[0].m_eventType<<",0"<<std::endl;
       for(unsigned int tIdx = 1;tIdx < m_timeLine.size();++tIdx) {
          tbb::tick_count t1 = m_timeLine[tIdx].m_time;
          auto duration = t1-t0;
          tl<<m_timeLine[tIdx].m_workId<<","<<m_timeLine[tIdx].m_eventType<<","<<1000*duration.seconds()<<std::endl;
       }
       tl.close();
      m_timeLine.clear();
    }
 }

 for(auto pair: m_d_detmodel_ptrs){
  hipFree(pair.second);
 }
}

bool TrigInDetModuleCuda::configure() {

  if(m_maxDevice == 0) {
     std::cout<<"No CUDA devices found"<<std::endl;
     return false;
  }
  return true;
}

SeedMakingDeviceContext* TrigInDetModuleCuda::createSeedMakingContext(int id) const {

  hipSetDevice(id);
  checkError(11);
  SeedMakingDeviceContext* p = new SeedMakingDeviceContext;

  p->m_deviceId = id;

  //set stream

  hipStreamCreate(&p->m_stream);
  checkError(12);
  //check device property

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, id);

  p->m_gpuParams.m_nSMX = deviceProp.multiProcessorCount;

  int ncores = GPUHelpers::getNumberOfCores(deviceProp.major, deviceProp.minor);
  
  p->m_gpuParams.m_nNUM_SMX_CORES = ncores;//_ConvertSMVer2Cores_local(deviceProp.major, deviceProp.minor);
  p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = NUM_TRIPLET_BLOCKS;
  if(deviceProp.maxThreadsPerBlock < p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS) 
    p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = deviceProp.maxThreadsPerBlock;

  //Allocate memory
  
  hipMalloc((void **)&p->d_settings,    sizeof(TrigAccel::SEED_FINDER_SETTINGS));
  hipMalloc((void **)&p->d_spacepoints, sizeof(TrigAccel::SPACEPOINT_STORAGE));
  checkError();
  hipMalloc((void **)&p->d_outputseeds, sizeof(TrigAccel::OUTPUT_SEED_STORAGE));
  hipMalloc((void **)&p->d_doubletstorage, sizeof(DOUBLET_STORAGE));
  hipMalloc((void **)&p->d_doubletinfo, sizeof(DOUBLET_INFO));
  checkError(13);

  p->d_size = sizeof(TrigAccel::SEED_FINDER_SETTINGS) +  
              sizeof(TrigAccel::SPACEPOINT_STORAGE) + sizeof(TrigAccel::OUTPUT_SEED_STORAGE) + sizeof(DOUBLET_STORAGE) + sizeof(DOUBLET_INFO) + 
              sizeof(TrigAccel::DETECTOR_MODEL);
  
  hipHostMalloc((void **)&p->h_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS));
  hipHostMalloc((void **)&p->h_spacepoints, sizeof(TrigAccel::SPACEPOINT_STORAGE));
  hipHostMalloc((void **)&p->h_outputseeds, sizeof(TrigAccel::OUTPUT_SEED_STORAGE));

  p->h_size = sizeof(TrigAccel::SEED_FINDER_SETTINGS) + sizeof(TrigAccel::SPACEPOINT_STORAGE) + sizeof(TrigAccel::OUTPUT_SEED_STORAGE);
  
  p->d_detmodel = m_d_detmodel_ptrs.at(id); //get the detmodel pointer for the selected device

  checkError(14);
  return p;
}

SeedMakingManagedDeviceContext* TrigInDetModuleCuda::createManagedSeedMakingContext(int id) const {

  hipSetDevice(id);
  checkError(11);
  SeedMakingManagedDeviceContext* p = new SeedMakingManagedDeviceContext;

  p->m_deviceId = id;

  //set stream

  hipStreamCreate(&p->m_stream);
  checkError(12);
  //check device property

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, id);

  p->m_gpuParams.m_nSMX = deviceProp.multiProcessorCount;

  int ncores = GPUHelpers::getNumberOfCores(deviceProp.major, deviceProp.minor);

  p->m_gpuParams.m_nNUM_SMX_CORES = ncores;//_ConvertSMVer2Cores_local(deviceProp.major, deviceProp.minor);
  p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = NUM_TRIPLET_BLOCKS;
  if(deviceProp.maxThreadsPerBlock < p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS) 
    p->m_gpuParams.m_nNUM_TRIPLET_BLOCKS = deviceProp.maxThreadsPerBlock;

  //Allocate memory
  
  hipMallocManaged((void **)&p->m_settings,    sizeof(TrigAccel::SEED_FINDER_SETTINGS));
  hipMallocManaged((void **)&p->m_spacepoints, sizeof(TrigAccel::SPACEPOINT_STORAGE));
  hipMallocManaged((void **)&p->m_outputseeds, sizeof(TrigAccel::OUTPUT_SEED_STORAGE));

  checkError();
  hipMalloc((void **)&p->d_doubletstorage, sizeof(DOUBLET_STORAGE));
  hipMalloc((void **)&p->d_doubletinfo, sizeof(DOUBLET_INFO));
  checkError(13);

  p->d_size = sizeof(DOUBLET_STORAGE) + sizeof(DOUBLET_INFO) + sizeof(TrigAccel::DETECTOR_MODEL);
  
  p->h_size = 0;
  
  p->m_size = sizeof(TrigAccel::SEED_FINDER_SETTINGS) + sizeof(TrigAccel::SPACEPOINT_STORAGE) + sizeof(TrigAccel::OUTPUT_SEED_STORAGE);

  p->d_detmodel = m_d_detmodel_ptrs.at(id); //get the detmodel pointer for the selected device

  checkError(14);
  return p;
}

TrigAccel::Work* TrigInDetModuleCuda::createWork(int workType, std::shared_ptr<TrigAccel::OffloadBuffer> data){
  if(workType == TrigAccel::InDetJobControlCode::SIL_LAYERS_EXPORT){
    unsigned char* d_detmodel;
    int deviceId = 0;//always using device 0 for the time being

    hipSetDevice(deviceId);
    hipMalloc(&d_detmodel, sizeof(TrigAccel::DETECTOR_MODEL));
    checkError();

    m_d_detmodel_ptrs[deviceId] = d_detmodel;

    hipMemcpy(d_detmodel, (unsigned char*)data->get(), sizeof(TrigAccel::DETECTOR_MODEL), hipMemcpyHostToDevice);
    checkError(21);

    return 0;
  }

  if(workType == TrigAccel::InDetJobControlCode::FIND_SEEDS){
 
    int deviceId = 0;//always using device 0 for the time being

    //TO-DO: to support mult-GPU load balancing get deviceId from a tbb_concurrent_queue

    SeedMakingDeviceContext* ctx = createSeedMakingContext(deviceId);

    TrigAccel::SEED_MAKING_JOB *pArray = reinterpret_cast<TrigAccel::SEED_MAKING_JOB*>(data->get());
    
    //1. copy settings to the context host array

    TrigAccel::SEED_FINDER_SETTINGS* p_settings = reinterpret_cast<TrigAccel::SEED_FINDER_SETTINGS*>(ctx->h_settings);
    memcpy(p_settings, &pArray->m_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS));

    //2. copy spacepoints to the context host array

    TrigAccel::SPACEPOINT_STORAGE* p_spacePoints = reinterpret_cast<TrigAccel::SPACEPOINT_STORAGE*>(ctx->h_spacepoints);
    memcpy(p_spacePoints, &pArray->m_data, sizeof(TrigAccel::SPACEPOINT_STORAGE));
    
    unsigned int workNum = m_workItemCounters[0]++;//seed making uses counter #0
    
    unsigned int workId = workNum*100;
    
    SeedMakingWorkCuda* w = new SeedMakingWorkCuda(workId, ctx, data, &m_timeLine);
    
    return w;
  }

  if(workType == TrigAccel::InDetJobControlCode::MAKE_SEEDS){
 
    int deviceId = 0;//always using device 0 for the time being

    //TO-DO: to support mult-GPU load balancing get deviceId from a tbb_concurrent_queue

    SeedMakingManagedDeviceContext* ctx = createManagedSeedMakingContext(deviceId);

    TrigAccel::SEED_MAKING_JOB *pArray = reinterpret_cast<TrigAccel::SEED_MAKING_JOB*>(data->get());
    
    //1. copy settings to the context host array

    TrigAccel::SEED_FINDER_SETTINGS* p_settings = reinterpret_cast<TrigAccel::SEED_FINDER_SETTINGS*>(ctx->m_settings);
    memcpy(p_settings, &pArray->m_settings, sizeof(TrigAccel::SEED_FINDER_SETTINGS));

    //2. copy spacepoints to the context host array

    TrigAccel::SPACEPOINT_STORAGE* p_spacePoints = reinterpret_cast<TrigAccel::SPACEPOINT_STORAGE*>(ctx->m_spacepoints);
    memcpy(p_spacePoints, &pArray->m_data, sizeof(TrigAccel::SPACEPOINT_STORAGE));
    
    unsigned int workNum = m_workItemCounters[0]++;//seed making uses counter #0
    
    unsigned int workId = workNum*100;
    
    SeedMakingWorkCudaManaged* w = new SeedMakingWorkCudaManaged(workId, ctx, data, &m_timeLine);
    
    return w;
  }

  return 0;
}

const std::vector<int> TrigInDetModuleCuda::getProvidedAlgs(){
  std::vector<int> v{
      TrigAccel::InDetJobControlCode::SIL_LAYERS_EXPORT,
      TrigAccel::InDetJobControlCode::MAKE_SEEDS,//the default
      TrigAccel::InDetJobControlCode::FIND_SEEDS //the alternative
  };
  return v;
}

