/*
  Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iterator>
#include <sstream>
#include <atomic>
#include <iostream>
#include "gpu_helpers.h"

#include "tbb/concurrent_queue.h"
#include "tbb/concurrent_vector.h"

int GPUHelpers::getNumberOfGPUs() {
  pid_t childpid;
  int fd[2];
  // create pipe descriptors
  pipe(fd);

  childpid = fork();
  if(childpid != 0) {  // parent
    close(fd[1]);
    // read the data (blocking operation)
    int maxDev;
    read(fd[0], &maxDev, sizeof(maxDev));
    // close the read-descriptor
    close(fd[0]);
    return maxDev;
  }
  else {  // child
    // writing only, no need for read-descriptor
    close(fd[0]);
    int maxDevice = 0;
    hipGetDeviceCount(&maxDevice);
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
      maxDevice = 0;
      std::cout << "ERROR: " << hipGetErrorString( error ) << std::endl;
    }
    // send the value on the write-descriptor
    write(fd[1], &maxDevice, sizeof(maxDevice)); 
    // close the write descriptor
    close(fd[1]);
    exit(0);
  }
}

int GPUHelpers::getNumberOfCores(int major, int minor) {
    
    int ncores = 0;
    
    if ((major == 7) && (minor == 5)) {
       ncores = 64;//Turing
    }
    if ((minor == 1) || (minor == 2)) ncores = 128;
    else if (minor == 0) ncores = 64;
    else if ((major == 8) && (minor == 6) ){
       ncores = 32;
    }

    if(ncores == 0) {
       std::cout<<"Cannot determine the number of cores: unknown device type, major="<<major<<" minor="<<minor<<std::endl;
    }
    return ncores;
}