/*
  Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "SeedMakingDataStructures_ITk.h"
#include "TrigAccelEvent/TrigITkAccelEDM.h"
#include "SeedMakingWorkCuda_ITk.h"

#include "device_context.h" //for SeedMakingDeviceContext
#include "tbb/tick_count.h"
#include <cstring>
#include <cmath>
#include <iostream>
#include <algorithm>

#include "DoubletCountingKernelCuda_ITk.cuh"
#include "DoubletMakingKernelCuda_ITk.cuh"
#include "DoubletMatchingKernelCuda_ITk.cuh"

SeedMakingWorkCudaITk::SeedMakingWorkCudaITk(unsigned int id, SeedMakingDeviceContext* ctx, std::shared_ptr<TrigAccel::OffloadBuffer> data, 
  tbb::concurrent_vector<WorkTimeStamp>* TL) : 
  m_workId(id),
  m_context(ctx), 
  m_input(data),
  m_timeLine(TL)
 {
  
  m_output = std::make_shared<TrigAccel::OffloadBuffer>(sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE));//output data
}

SeedMakingWorkCudaITk::~SeedMakingWorkCudaITk() {
  
  SeedMakingDeviceContext* p = m_context;

  int id = p->m_deviceId;

  hipSetDevice(id);

  hipStreamDestroy(p->m_stream);

  hipFree(p->d_settings);
  hipFree(p->d_spacepoints);
  
  hipFree(p->d_outputseeds);
  hipFree(p->d_doubletstorage);
  hipFree(p->d_doubletinfo);

  hipHostFree(p->h_settings);
  hipHostFree(p->h_spacepoints);
  hipHostFree(p->h_outputseeds);

  delete p;
  m_context = 0;
}

std::shared_ptr<TrigAccel::OffloadBuffer> SeedMakingWorkCudaITk::getOutput() {
  return m_output;
}

bool SeedMakingWorkCudaITk::run() {

  m_timeLine->push_back(WorkTimeStamp(m_workId, 0, tbb::tick_count::now()));

  const SeedMakingDeviceContext& p = *m_context;
  
  int id = p.m_deviceId;  
  
  TrigAccel::ITk::OUTPUT_SEED_STORAGE* ps = reinterpret_cast<TrigAccel::ITk::OUTPUT_SEED_STORAGE*>(p.h_outputseeds);
  
  hipSetDevice(id);

  checkError();
  
  hipMemcpyAsync(p.d_settings, p.h_settings, sizeof(TrigAccel::ITk::SEED_FINDER_SETTINGS), hipMemcpyHostToDevice, p.m_stream);

  checkError();

  hipMemcpyAsync(p.d_spacepoints, p.h_spacepoints, sizeof(TrigAccel::ITk::SPACEPOINT_STORAGE), hipMemcpyHostToDevice, p.m_stream);

  checkError();

  hipStreamSynchronize(p.m_stream);
    
  TrigAccel::ITk::SEED_FINDER_SETTINGS* dSettings  = reinterpret_cast<TrigAccel::ITk::SEED_FINDER_SETTINGS *>(p.d_settings);
  TrigAccel::ITk::SPACEPOINT_STORAGE* dSpacepoints = reinterpret_cast<TrigAccel::ITk::SPACEPOINT_STORAGE *>(p.d_spacepoints);
  TrigAccel::ITk::DETECTOR_MODEL* dDetModel        = reinterpret_cast<TrigAccel::ITk::DETECTOR_MODEL*>(p.d_detmodel);
  TrigAccel::ITk::OUTPUT_SEED_STORAGE* dOutput     = reinterpret_cast<TrigAccel::ITk::OUTPUT_SEED_STORAGE*>(p.d_outputseeds);

  DOUBLET_INFO_ITk* dInfo                         = reinterpret_cast<DOUBLET_INFO_ITk*>(p.d_doubletinfo);
  DOUBLET_STORAGE_ITk* dStorage                   = reinterpret_cast<DOUBLET_STORAGE_ITk*>(p.d_doubletstorage);

  hipMemset(p.d_outputseeds,0,10*sizeof(int));

  checkError();

  hipMemset(p.d_doubletstorage,0,3*sizeof(int));

  checkError();
  
  const TrigAccel::ITk::SPACEPOINT_STORAGE* pSPS = reinterpret_cast<const TrigAccel::ITk::SPACEPOINT_STORAGE *>(p.h_spacepoints);
  int nSlices = pSPS->m_nPhiSlices;
  int nLayers = pSPS->m_nLayers;
  
  int nMiddleSp = NUM_MIDDLE_THREADS_ITk;//determines size of the doublet/triplet buffers
  int nOtherSp = OUTER_THREADS_MULTIPLIER_ITk*p.m_gpuParams.m_nNUM_SMX_CORES/NUM_MIDDLE_THREADS_ITk;//the size of the spacepoint buffer

  dim3 gridDimensions(nSlices, nLayers);
  dim3 blockDimensions(nMiddleSp, nOtherSp);

  hipMemset(p.d_doubletinfo,0,sizeof(DOUBLET_INFO_ITk));

  checkError();

  hipStreamSynchronize(p.m_stream);

  checkError();

  doubletCountingKernel_ITk<<<gridDimensions, blockDimensions, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dInfo, nLayers, nSlices);

  hipStreamSynchronize(p.m_stream);

  checkError();

  doubletMakingKernel_ITk<<<gridDimensions, blockDimensions, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dOutput, 
    dInfo, dStorage, nLayers, nSlices);

  hipStreamSynchronize(p.m_stream);

  checkError();

  int nStats[3];

  hipMemcpy(&nStats[0], p.d_doubletstorage, 3*sizeof(int), hipMemcpyDeviceToHost);

  
  doubletMatchingKernel_ITk<<<p.m_gpuParams.m_nNUM_TRIPLET_BLOCKS, NUM_TRIPLET_THREADS_ITk, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dInfo, 
    dStorage,  dOutput, nStats[0]);

  hipStreamSynchronize(p.m_stream);

  checkError();

  TrigAccel::ITk::OUTPUT_SEED_STORAGE* pOutput = reinterpret_cast<TrigAccel::ITk::OUTPUT_SEED_STORAGE *>(m_output->m_rawBuffer);

  //Read back GPU results

  pOutput->m_nMiddleSps = 0;
  pOutput->m_nSeeds = 0;		
  pOutput->m_nI = 0;
  pOutput->m_nO = 0;

  hipMemcpyAsync(p.h_outputseeds, p.d_outputseeds, sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE), hipMemcpyDeviceToHost, p.m_stream);

  checkError();
  
  hipStreamSynchronize(p.m_stream);

  checkError();

  memcpy(pOutput, ps, sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE));
  
  m_timeLine->push_back(WorkTimeStamp(m_workId, 1, tbb::tick_count::now()));

  return true;
}

SeedMakingWorkCudaManagedITk::SeedMakingWorkCudaManagedITk(unsigned int id, SeedMakingManagedDeviceContext* ctx, std::shared_ptr<TrigAccel::OffloadBuffer> data, tbb::concurrent_vector<WorkTimeStamp>* TL) :
  m_workId(id),
  m_context(ctx), 
  m_input(data),
  m_timeLine(TL){
  
  m_output = std::make_shared<TrigAccel::OffloadBuffer>(sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE));//output data
}

SeedMakingWorkCudaManagedITk::~SeedMakingWorkCudaManagedITk() {
  
  SeedMakingManagedDeviceContext* p = m_context;

  int id = p->m_deviceId;

  hipSetDevice(id);

  hipStreamDestroy(p->m_stream);

  hipFree(p->m_settings);
  hipFree(p->m_spacepoints);
  
  hipFree(p->m_outputseeds);
  hipFree(p->d_doubletstorage);
  hipFree(p->d_doubletinfo);

  delete p;
  m_context = 0;
}

std::shared_ptr<TrigAccel::OffloadBuffer> SeedMakingWorkCudaManagedITk::getOutput() {
  return m_output;
}

bool SeedMakingWorkCudaManagedITk::run() {

  m_timeLine->push_back(WorkTimeStamp(m_workId, 0, tbb::tick_count::now()));

  const SeedMakingManagedDeviceContext& p = *m_context;
  
  int id = p.m_deviceId;  
  
  TrigAccel::ITk::OUTPUT_SEED_STORAGE* ps = reinterpret_cast<TrigAccel::ITk::OUTPUT_SEED_STORAGE*>(p.m_outputseeds);
  
  hipSetDevice(id);

  checkError();
  


  hipMemPrefetchAsync(p.m_settings, sizeof(TrigAccel::ITk::SEED_FINDER_SETTINGS), id, p.m_stream);

  checkError();



  hipMemPrefetchAsync(p.m_spacepoints, sizeof(TrigAccel::ITk::SPACEPOINT_STORAGE), id, p.m_stream);

  checkError();

  hipStreamSynchronize(p.m_stream);
    
  TrigAccel::ITk::SEED_FINDER_SETTINGS* dSettings  = reinterpret_cast<TrigAccel::ITk::SEED_FINDER_SETTINGS *>(p.m_settings);
  TrigAccel::ITk::SPACEPOINT_STORAGE* dSpacepoints = reinterpret_cast<TrigAccel::ITk::SPACEPOINT_STORAGE *>(p.m_spacepoints);
  TrigAccel::ITk::DETECTOR_MODEL* dDetModel        = reinterpret_cast<TrigAccel::ITk::DETECTOR_MODEL*>(p.d_detmodel);
  TrigAccel::ITk::OUTPUT_SEED_STORAGE* dOutput     = reinterpret_cast<TrigAccel::ITk::OUTPUT_SEED_STORAGE*>(p.m_outputseeds);

  DOUBLET_INFO_ITk* dInfo                         = reinterpret_cast<DOUBLET_INFO_ITk*>(p.d_doubletinfo);
  DOUBLET_STORAGE_ITk* dStorage                   = reinterpret_cast<DOUBLET_STORAGE_ITk*>(p.d_doubletstorage);

  hipMemset(p.m_outputseeds,0,10*sizeof(int));

  checkError();

  hipMemset(p.d_doubletstorage,0,3*sizeof(int));

  checkError();
  
  const TrigAccel::ITk::SPACEPOINT_STORAGE* pSPS = reinterpret_cast<const TrigAccel::ITk::SPACEPOINT_STORAGE *>(p.m_spacepoints);
  int nSlices = pSPS->m_nPhiSlices;
  int nLayers = pSPS->m_nLayers;
  
  int nMiddleSp = NUM_MIDDLE_THREADS_ITk;//determines size of the doublet/triplet buffers
  int nOtherSp = OUTER_THREADS_MULTIPLIER_ITk*p.m_gpuParams.m_nNUM_SMX_CORES/NUM_MIDDLE_THREADS_ITk;//the size of the spacepoint buffer

  dim3 gridDimensions(nSlices, nLayers);
  dim3 blockDimensions(nMiddleSp, nOtherSp);

  hipMemset(p.d_doubletinfo,0,sizeof(DOUBLET_INFO_ITk));

  checkError();

  hipStreamSynchronize(p.m_stream);

  checkError();

  doubletCountingKernel_ITk<<<gridDimensions, blockDimensions, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dInfo, nLayers, nSlices);

  hipStreamSynchronize(p.m_stream);

  checkError();

  doubletMakingKernel_ITk<<<gridDimensions, blockDimensions, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dOutput, 
    dInfo, dStorage, nLayers, nSlices);

  hipStreamSynchronize(p.m_stream);

  checkError();

  int nStats[3];

  hipMemcpy(&nStats[0], p.d_doubletstorage, 3*sizeof(int), hipMemcpyDeviceToHost);

  
  doubletMatchingKernel_ITk<<<p.m_gpuParams.m_nNUM_TRIPLET_BLOCKS, NUM_TRIPLET_THREADS_ITk, 0, p.m_stream>>>(dSettings, dSpacepoints, dDetModel, dInfo, 
    dStorage,  dOutput, nStats[0]);

  hipStreamSynchronize(p.m_stream);

  checkError();

  TrigAccel::ITk::OUTPUT_SEED_STORAGE* pOutput = reinterpret_cast<TrigAccel::ITk::OUTPUT_SEED_STORAGE *>(m_output->m_rawBuffer);

  //Read back GPU results

  pOutput->m_nMiddleSps = 0;
  pOutput->m_nSeeds = 0;		
  pOutput->m_nI = 0;
  pOutput->m_nO = 0;


  checkError();

  memcpy(pOutput, ps, sizeof(TrigAccel::ITk::OUTPUT_SEED_STORAGE));
  
  m_timeLine->push_back(WorkTimeStamp(m_workId, 1, tbb::tick_count::now()));

  return true;
}
