#include "hip/hip_runtime.h"
//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//
// Dear emacs, this is -*- c++ -*-
//

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "CaloTopoClusterSplitterGPUImpl.h"


#include <cstring>
#include <cmath>
#include <iostream>
#include <stdio.h>

using namespace CaloRecGPU;

void GPUSplitterOptionsHolder::allocate()
{
  m_options.allocate();
  m_options_dev.allocate();
}

void GPUSplitterOptionsHolder::sendToGPU(const bool clear_CPU)
{
  m_options_dev = m_options;
  if (clear_CPU)
    {
      m_options.clear();
    }
}


#define check_if_secondary(cell_id, d_meta, d_fullclusters) ((d_fullcalogeometry)->sampling((cell_id)) >= (d_meta)->m_minSecondarySampling &&\
                                                             (d_fullcalogeometry)->sampling((cell_id)) <= (d_meta)->m_maxSecondarySampling &&\
                                                             ((d_meta)->m_useSecondarySampling & (1 << ((d_fullcalogeometry)->sampling((cell_id)) - (d_meta)->m_minSecondarySampling))))

constexpr static int DefaultBlockSize = 512;


/************************************************************************/


void preProcessingPreparation(EventDataHolder & holder, Helpers::CUDA_kernel_object<GPUSplitterTemporaries> temps,
                              const ConstantDataHolder & instance_data, const GPUSplitterOptionsHolder & options, const bool synchronize)
{
  CUDA_ERRCHECK(hipMemsetAsync((void *) temps->max_cells, 0, NCaloCells * sizeof(temps->max_cells[0]), hipStreamPerThread));
  CUDA_ERRCHECK(hipMemsetAsync((void *) temps->secondary_max_cells, 0, NCaloCells * sizeof(temps->secondary_max_cells[0]), hipStreamPerThread));
  CUDA_ERRCHECK(hipMemsetAsync((void *) temps->splitter_seeds, 0, NMaxClusters * sizeof(temps->splitter_seeds[0]), hipStreamPerThread));
  CUDA_ERRCHECK(hipMemsetAsync((void *) temps->secondary_splitter_seeds, 0, NMaxClusters * sizeof(temps->secondary_splitter_seeds[0]), hipStreamPerThread));

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(hipStreamPerThread));
    }
}



/***********************************************************************/

static __global__ void find_local_maximums(const Helpers::CUDA_kernel_object<CaloTopoClusterSplitterMetadata> d_meta,
                                           const Helpers::CUDA_kernel_object<GeometryArr> d_fullcalogeometry,
                                           const Helpers::CUDA_kernel_object<CellInfoArr> d_cellsfulldata,
                                           const Helpers::CUDA_kernel_object<CellStateArr> d_cellstate,
                                           Helpers::CUDA_kernel_object<GPUSplitterTemporaries> temporaries)
{
  unsigned cell_id = blockIdx.x * blockDim.x + threadIdx.x;
  int num_neighbours, n_id;
  int cell_tag;
  int i;
  int count_neigh = 0;
  bool max = true;
  float energy, n_energy, n_eta, n_phi;
  bool is_secondary = false, is_primary = true;
  int calo_sample, n_sample;

  if (cell_id > NCaloCells)
    {
      return;
    }

  num_neighbours = d_fullcalogeometry->neighbours.get_total_number_of_neighbours(cell_id);
  calo_sample = d_fullcalogeometry->sampling(cell_id);
  energy = d_cellsfulldata->energy[cell_id];
  cell_tag = ClusterTag::cluster_index(d_cellstate->clusterTag[cell_id]);

  if (d_meta->m_absOpt)
    {
      energy = fabs(energy);
    }

  if (energy < d_meta->m_minEnergy)
    {
      return;
    }

  if (d_cellsfulldata->is_bad(*d_fullcalogeometry, cell_id, d_meta->m_treatL1PredictedCellsAsGood) && energy > 0)
    {

      /* check if cell can be used for local max */
      if (d_meta->uses_sampling(calo_sample))
        {
          is_primary = true;
        }
      else if (d_meta->uses_secondary_sampling(calo_sample))
        {
          is_secondary = true;
        }
    }

  for (i = 0; i < num_neighbours; i++)
    {
      n_id = d_fullcalogeometry->neighbours.get_neighbour(cell_id, i);
      //FUTURE TODO WARNING FIX ALERT ERROR PAY ATTENTION:
      //this is not taking into account limited neighbours!

      /* skip if both cells aren't in the same cluster */
      if (cell_tag != ClusterTag::cluster_index(d_cellstate->clusterTag[n_id]))
        {
          continue;
        }

      n_energy = d_meta->m_absOpt ? fabs(d_cellsfulldata->energy[n_id]) : d_cellsfulldata->energy[n_id];
      n_phi = d_fullcalogeometry->phi[n_id];
      n_eta = d_fullcalogeometry->eta[n_id];
      n_sample = d_fullcalogeometry->sampling(n_id);

      if (energy > n_energy)
        {
          count_neigh++;
        }
      else
        {
          max = false;
        }

      if (n_energy >= d_meta->m_minEnergy &&
          is_secondary &&
          d_meta->uses_sampling(n_sample))
        {
          if (fabs(n_eta - d_fullcalogeometry->eta[cell_id]) < 0.0025)
            {
              double diff_phi = n_phi - d_fullcalogeometry->phi[cell_id];
              if (diff_phi > M_PI)
                {
                  diff_phi = diff_phi - 2 * M_PI;
                }
              else if (diff_phi < -M_PI)
                {
                  diff_phi = diff_phi + 2 * M_PI;
                }

              if (fabs(diff_phi) < 0.02)
                {
                  max = false;
                }
            }
        }
    }

  if (count_neigh < d_meta->m_nCells)
    {
      max = false;
    }

  if (max)
    {
      if (is_primary)
        {
          temporaries->max_cells[cell_id] = 1;
          atomicAdd(&temporaries->splitter_seeds[cell_tag], 1);
        }
      else if (is_secondary)
        {
          temporaries->secondary_max_cells[cell_id] = 1;
          atomicAdd(&temporaries->secondary_splitter_seeds[cell_tag], 1);
        }
    }
}


void findLocalMaxima(EventDataHolder & holder, Helpers::CUDA_kernel_object<GPUSplitterTemporaries> temps,
                     const ConstantDataHolder & instance_data, const GPUSplitterOptionsHolder & options, const bool synchronize)
{
  const int block_size = DefaultBlockSize;
  const int num_blocks = Helpers::int_ceil_div(NCaloCells, block_size);

  dim3 bsize(block_size, 1, 1);
  dim3 gsize(num_blocks, 1, 1);

  find_local_maximums <<< gsize, bsize>>>(options.m_options_dev, instance_data.m_geometry_dev,
                                          holder.m_cell_info_dev, holder.m_cell_state_dev, temps);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(hipStreamPerThread));
    }
}


/***********************************************************************/

__global__ void splitter_tag_propagation(const Helpers::CUDA_kernel_object<GeometryArr> d_fullcalogeometry,
                                         const Helpers::CUDA_kernel_object<CellStateArr> d_cellstate,
                                         Helpers::CUDA_kernel_object<ClusterInfoArr> d_fullclusters,
                                         Helpers::CUDA_kernel_object<GPUSplitterTemporaries> temporaries)
{
  int tid = threadIdx.x;
  int cell_tag, n_cell_tag;
  int cell_id, n_cell_id;
  int old_num;
  /*int offset = d_fullclusters->number;*/
  int i, k, steps = 0;
  __shared__ int * aux, *first_q, *sec_q;


  __shared__ int current;
  __shared__ int Q_len, Q2_len;

  if (tid == 0)
    {
      current = 0;
      Q_len = 0;
      first_q = temporaries->queue1;
      sec_q = temporaries->queue2;

      //NSF: We need this so the new clusters can be properly processed...
      //(Obviously, since this is all on one block,
      // it's fine to zero it here...)
      d_fullclusters->number = 0;

    }

  __syncthreads();

  /* Iterate through all cells and:
   * - queue the local max and secondary maximums
   * - assign a new cluster tag = call_id + n_clusters
   * Compared to the CPU implementation, this doesn't start from the cells
   * with highest energy since we don't sort the array on GPU so we might
   * obtain slightly different results because of that.
   */
  for (cell_id = tid; cell_id < NCaloCells; cell_id += blockDim.x)
    {
      if (temporaries->max_cells[cell_id] || temporaries->secondary_max_cells[cell_id])
        {

          old_num = atomicAdd(&current, 1);
          temporaries->tags[cell_id] = old_num /*+ offset*/;

          old_num = atomicAdd(&Q_len, 1);
          temporaries->queue1[old_num] = cell_id;


          //NSF: We need this so the new clusters can be properly processed...
          const int old_size = atomicAdd(&(d_fullclusters->number), 1);
          d_fullclusters->seedCellID[old_size] = cell_id;

        }
      else
        {
          temporaries->tags[cell_id] = -1;
        }
    }

  __syncthreads();

  /* Start iterate through cells in queue built at previous step and:
   * - check what cell neighbour doesn't have a tag assigned in d_tags but
   *   has the same original cluster tag with current cell
   * - if the tag is -1 (unused yet), change its tag with current cell
   *   tag and add the neighbour cell in a secondary queue.
   * This algorithm will be repeated until the secondary queue is empty.
   */
  do
    {
      if (tid == 0)
        {
          Q2_len = 0;
        }

      __syncthreads();

      for (i = tid; i < Q_len; i += blockDim.x)
        {
          cell_id = first_q[i];
          cell_tag = ClusterTag::cluster_index(d_cellstate->clusterTag[cell_id]);
          int num_neighbours = d_fullcalogeometry->neighbours.get_total_number_of_neighbours(cell_id);
          //FUTURE TODO WARNING FIX ALERT ERROR PAY ATTENTION:
          //this is not taking into account limited neighbours!

          for (k = 0; k < num_neighbours; k++)
            {
              n_cell_id = d_fullcalogeometry->neighbours.get_neighbour(cell_id, k);
              n_cell_tag = ClusterTag::cluster_index(d_cellstate->clusterTag[n_cell_id]);

              if (cell_tag == n_cell_tag)
                {
                  old_num = atomicCAS(&temporaries->tags[n_cell_id], -1, temporaries->tags[cell_id]);
                  if (old_num == -1)
                    {
                      int j = atomicAdd(&Q2_len, 1);
                      sec_q[j] = n_cell_id;
                    }
                }
            }
        }

      __syncthreads();

      if (tid == 0)
        {
          steps ++;
          aux = sec_q;
          sec_q = first_q;
          first_q = aux;
          Q_len = Q2_len;
        }

      __syncthreads();
    }
  while (Q2_len > 0);
}


void propagateTags(EventDataHolder & holder, Helpers::CUDA_kernel_object<GPUSplitterTemporaries> temps,
                   const ConstantDataHolder & instance_data, const GPUSplitterOptionsHolder & options, const bool synchronize)
{
  splitter_tag_propagation <<< 1, 2 * DefaultBlockSize>>>(instance_data.m_geometry_dev,
                                                          holder.m_cell_state_dev,
                                                          holder.m_clusters_dev,
                                                          temps);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(hipStreamPerThread));
    }

}

/***********************************************************************/


static __global__ void refill_clusters(const Helpers::CUDA_kernel_object<GeometryArr> d_fullcalogeometry,
                                       Helpers::CUDA_kernel_object<ClusterInfoArr> d_fullclusters,
                                       Helpers::CUDA_kernel_object<CellStateArr> d_cellstate,
                                       Helpers::CUDA_kernel_object<GPUSplitterTemporaries> temporaries)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= NCaloCells)
    {
      return;
    }

  /* reset all clusters to 0 */
  /*
  if (i < NMaxClusters) {
      d_fullclusters->clusterSize[i] = 0;
      d_fullclusters->clusterEnergy[i] = 0;
      d_fullclusters->clusterEt[i] = 0;
      d_fullclusters->clusterEta[i] = 0;
  }
  */

  int tag = temporaries->tags[i];

  /* set the seed tags accordingly to what we found in splitter algorithm */
  if (tag >= 0)
    {
      d_cellstate->clusterTag[i] = ClusterTag::make_tag(tag);
    }
  else
    {
      tag = -1;
      d_cellstate->clusterTag[i] = ClusterTag::make_invalid_tag();
    }

  //NSF: As there's no synchronization, there might be trouble between the zeroing and the calculation...

  /*
  if (tag >= 0) {
      float energy = d_fullclusters->cellSN2[i];
      atomicAdd(&d_fullclusters->clusterEnergy[tag], energy);
      atomicAdd(&d_fullclusters->clusterEt[tag], abs(energy));
      atomicAdd(&d_fullclusters->clusterEta[tag], d_fullcalogeometry->eta[i] * abs(energy));

      // TODO: why?
      float phi0 = d_fullcalogeometry->phi[i];
      float phi = phi0;
      if (phi > phi0 + M_PI)
          phi = phi - 2 * M_PI;
      if (phi < phi0 - M_PI)
          phi = phi + 2 * M_PI;
      atomicAdd(&d_fullclusters->clusterPhi[tag], phi * abs(energy));
  }
  */
}

void refillClusters(EventDataHolder & holder, Helpers::CUDA_kernel_object<GPUSplitterTemporaries> temps,
                    const ConstantDataHolder & instance_data, const GPUSplitterOptionsHolder & options, const bool synchronize)
{
  const int block_size = DefaultBlockSize;
  const int num_blocks = Helpers::int_ceil_div(NCaloCells, block_size);

  dim3 bsize(block_size, 1, 1);
  dim3 gsize(num_blocks, 1, 1);

  refill_clusters <<< gsize, bsize>>>(instance_data.m_geometry_dev, holder.m_clusters_dev,
                                      holder.m_cell_state_dev, temps);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(hipStreamPerThread));
    }
}
