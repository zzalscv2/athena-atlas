#include "hip/hip_runtime.h"
//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//
// Dear emacs, this is -*- c++ -*-
//

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "BasicGPUClusterInfoCalculatorImpl.h"

#include <cstring>
#include <cmath>
#include <iostream>
#include <stdio.h>

#include "CaloRecGPU/IGPUKernelSizeOptimizer.h"

using namespace CaloRecGPU;
using namespace BasicClusterInfoCalculator;

/**********************************************************************************/
constexpr static int SeedCellPropertiesBlockSize = 512;

constexpr static int CalculateClusterInfoBlockSize = 320;
constexpr static int FinalizeClusterInfoBlockSize = 256;
constexpr static int ClearInvalidCellsBlockSize = 512;

/**********************************************************************************/

__global__ static
void seedCellPropertiesKernel(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                              Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temporaries,
                              const Helpers::CUDA_kernel_object<GeometryArr> geometry)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  const int cluster_number = clusters_arr->number;
  for (int cluster = index; cluster < cluster_number; cluster += grid_size)
    {
      clusters_arr->clusterEnergy[cluster] = 0.f;
      clusters_arr->clusterEt[cluster] = 0.f;
      clusters_arr->clusterEta[cluster] = 0.f;
      clusters_arr->clusterPhi[cluster] = 0.f;
      const int seed_cell = clusters_arr->seedCellID[cluster];
      if (seed_cell >= 0)
        {
          temporaries->seedCellPhi[cluster] = geometry->phi[seed_cell];
        }
      else
        {
          temporaries->seedCellPhi[cluster] = 0.f;
        }
    }
}

__global__ static
void seedCellPropertiesDeferKernel(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                   Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temporaries,
                                   const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                                   const int i_dimBlock)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    {
      const int cluster_number = clusters_arr->number;

      //const int i_dimBlock = SeedCellPropertiesBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(cluster_number, i_dimBlock);

#if CUDA_CAN_USE_TAIL_LAUNCH
      seedCellPropertiesKernel <<< i_dimGrid, i_dimBlock, 0, cudaStreamTailLaunch>>>(clusters_arr, temporaries, geometry);
#else
      seedCellPropertiesKernel <<< i_dimGrid, i_dimBlock>>>(clusters_arr, temporaries, geometry);
#endif
    }
}


void BasicClusterInfoCalculator::updateSeedCellProperties(CaloRecGPU::EventDataHolder & holder,
                                                          CaloRecGPU::Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temps,
                                                          const ConstantDataHolder & instance_data,
                                                          const IGPUKernelSizeOptimizer & optimizer,
                                                          const bool synchronize,
                                                          CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  const CUDAKernelLaunchConfiguration launch_config = optimizer.get_launch_configuration("BasicClusterInfoCalculator", 0);

  if (optimizer.use_minimal_kernel_sizes() && optimizer.can_use_dynamic_parallelism())
    {

      seedCellPropertiesDeferKernel <<< 1, 1, 0, stream_to_use>>>(holder.m_clusters_dev, temps, instance_data.m_geometry_dev, launch_config.block_x);
    }
  else
    {
      seedCellPropertiesKernel <<< launch_config.grid_x, launch_config.block_x, 0, stream_to_use>>>(holder.m_clusters_dev, temps, instance_data.m_geometry_dev);

    }

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}


/**********************************************************************************/

__global__ static
void calculateClusterInfoKernel(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                                const Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temporaries)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const ClusterTag tag = cell_state_arr->clusterTag[cell];
      if (tag.is_part_of_cluster())
        //By this point they all have the terminals anyway, so...
        {
          if (tag.is_shared_between_clusters())
            {
              const int primary_cluster = tag.cluster_index();
              const int secondary_cluster = tag.secondary_cluster_index();

              const float secondary_weight = __int_as_float(tag.secondary_cluster_weight());
              const float weight = 1.0f - secondary_weight;

              const float energy = cell_info_arr->energy[cell];
              const float abs_energy = fabsf(energy);
              const float phi_raw = geometry->phi[cell];

              atomicAdd(&(clusters_arr->clusterEnergy[primary_cluster]), energy * weight);
              atomicAdd(&(clusters_arr->clusterEt[primary_cluster]), abs_energy * weight);
              atomicAdd(&(clusters_arr->clusterEta[primary_cluster]), abs_energy * geometry->eta[cell] * weight);

              const float primary_phi_0 = temporaries->seedCellPhi[primary_cluster];
              const float primary_phi_real = Helpers::regularize_angle(phi_raw, primary_phi_0);
              atomicAdd(&(clusters_arr->clusterPhi[primary_cluster]), primary_phi_real * abs_energy * weight);

              atomicAdd(&(clusters_arr->clusterEnergy[secondary_cluster]), energy * secondary_weight);
              atomicAdd(&(clusters_arr->clusterEt[secondary_cluster]), abs_energy * secondary_weight);
              atomicAdd(&(clusters_arr->clusterEta[secondary_cluster]), abs_energy * geometry->eta[cell] * secondary_weight);

              const float secondary_phi_0 = temporaries->seedCellPhi[secondary_cluster];
              const float secondary_phi_real = Helpers::regularize_angle(phi_raw, secondary_phi_0);
              atomicAdd(&(clusters_arr->clusterPhi[secondary_cluster]), secondary_phi_real * abs_energy * secondary_weight);
            }
          else
            {
              const int cluster_index = tag.cluster_index();
              const float energy = cell_info_arr->energy[cell];
              const float abs_energy = fabsf(energy);
              const float phi_raw = geometry->phi[cell];

              atomicAdd(&(clusters_arr->clusterEnergy[cluster_index]), energy);
              atomicAdd(&(clusters_arr->clusterEt[cluster_index]), abs_energy);
              atomicAdd(&(clusters_arr->clusterEta[cluster_index]), abs_energy * geometry->eta[cell]);

              const float phi_0 = temporaries->seedCellPhi[cluster_index];
              const float phi_real = Helpers::regularize_angle(phi_raw, phi_0);
              atomicAdd(&(clusters_arr->clusterPhi[cluster_index]), phi_real * abs_energy);
            }
        }
    }
}


__global__ static
void finalizeClusterInfoKernel(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                               const bool cut_in_absolute_ET, const float ET_threshold   )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  const int cluster_number = clusters_arr->number;
  for (int cluster = index; cluster < cluster_number; cluster += grid_size)
    {
      const float abs_energy = clusters_arr->clusterEt[cluster];

      if (abs_energy > 0)
        {
          const float tempeta = clusters_arr->clusterEta[cluster] / abs_energy;

          clusters_arr->clusterEta[cluster] = tempeta;

          const float temp_ET = clusters_arr->clusterEnergy[cluster] / coshf(abs(tempeta));

          clusters_arr->clusterEt[cluster] = temp_ET;

          clusters_arr->clusterPhi[cluster] = Helpers::regularize_angle(clusters_arr->clusterPhi[cluster] / abs_energy, 0.f);

          if ( !(temp_ET > ET_threshold || (cut_in_absolute_ET && fabsf(temp_ET) > ET_threshold) ) )
            {
              clusters_arr->seedCellID[cluster] = -1;
            }
        }
      else
        {
          clusters_arr->seedCellID[cluster] = -1;
          //This is just a way to signal that this is an invalid cluster.
        }
    }
}

__global__ static
void finalizeClustersDeferKernel(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                 const bool cut_in_absolute_ET, const float ET_threshold,
                                 const int i_dimBlock)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    {
      const int cluster_number = clusters_arr->number;

      //const int i_dimBlock = FinalizeClusterInfoBlockSize;
      const int i_dimGrid = Helpers::int_ceil_div(cluster_number, i_dimBlock);
      const dim3 dimBlock(i_dimBlock, 1, 1);
      const dim3 dimGrid(i_dimGrid, 1, 1);
#if CUDA_CAN_USE_TAIL_LAUNCH
      finalizeClusterInfoKernel <<< dimGrid, dimBlock, 0, cudaStreamTailLaunch>>>(clusters_arr, cut_in_absolute_ET, ET_threshold);
#else
      finalizeClusterInfoKernel <<< dimGrid, dimBlock>>>(clusters_arr, cut_in_absolute_ET, ET_threshold);
#endif
    }
}

__global__ static
void clearInvalidCells(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                       const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const ClusterTag tag = cell_state_arr->clusterTag[cell];
      if (tag.is_part_of_cluster())
        //By this point they all have the terminals anyway, so...
        {
          if (tag.is_shared_between_clusters())
            {
              const int first_cluster = tag.cluster_index();
              const int second_cluster = tag.secondary_cluster_index();

              const int first_seed = clusters_arr->seedCellID[first_cluster];
              const int second_seed = clusters_arr->seedCellID[second_cluster];

              if (first_seed < 0 && second_seed < 0)
                {
                  cell_state_arr->clusterTag[cell] = ClusterTag:: make_invalid_tag();
                }
              else if (first_seed < 0)
                {
                  cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(second_cluster);
                }
              else if (second_seed < 0)
                {
                  cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(first_cluster);
                }
              else /*if (first_seed >= 0 && second_seed >= 0)*/
                {
                  //Do nothing: the tag's already OK.
                }
            }
          else
            {
              if (clusters_arr->seedCellID[tag.cluster_index()] < 0)
                {
                  cell_state_arr->clusterTag[cell] = ClusterTag:: make_invalid_tag();
                }
            }
        }
    }
}

void BasicClusterInfoCalculator::calculateClusterProperties(CaloRecGPU::EventDataHolder & holder,
                                                            CaloRecGPU::Helpers::CUDA_kernel_object<ClusterInfoCalculatorTemporaries> temps,
                                                            const ConstantDataHolder & instance_data,
                                                            const IGPUKernelSizeOptimizer & optimizer,
                                                            const bool synchronize,
                                                            const bool cut_in_absolute_ET, const float ET_threshold,
                                                            CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  const CUDAKernelLaunchConfiguration cfg_calculate = optimizer.get_launch_configuration("BasicClusterInfoCalculator", 1);
  const CUDAKernelLaunchConfiguration cfg_finalize  = optimizer.get_launch_configuration("BasicClusterInfoCalculator", 2);
  const CUDAKernelLaunchConfiguration cfg_clear     = optimizer.get_launch_configuration("BasicClusterInfoCalculator", 3);

  calculateClusterInfoKernel <<< cfg_calculate.grid_x, cfg_calculate.block_x, 0, stream_to_use>>>(holder.m_clusters_dev, holder.m_cell_state_dev,
                                                                                                  holder.m_cell_info_dev, instance_data.m_geometry_dev, temps);

  if (optimizer.use_minimal_kernel_sizes() && optimizer.can_use_dynamic_parallelism())
    {
      finalizeClustersDeferKernel <<< 1, 1, 0, stream_to_use>>>(holder.m_clusters_dev, cut_in_absolute_ET, ET_threshold, cfg_finalize.block_x);
    }
  else
    {
      finalizeClusterInfoKernel <<< cfg_finalize.grid_x, cfg_finalize.block_x, 0, stream_to_use>>>(holder.m_clusters_dev, cut_in_absolute_ET, ET_threshold);
    }
  clearInvalidCells <<< cfg_clear.block_x, cfg_clear.block_x, 0, stream_to_use>>>(holder.m_cell_state_dev, holder.m_clusters_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}

/*******************************************************************************************************************************/

void BasicClusterInfoCalculator::register_kernels(IGPUKernelSizeOptimizer & optimizer)
{
  void * kernels[] = { (void *) seedCellPropertiesKernel,
                       (void *) calculateClusterInfoKernel,
                       (void *) finalizeClusterInfoKernel,
                       (void *) clearInvalidCells
                     };

  int blocksizes[] = { SeedCellPropertiesBlockSize,
                       CalculateClusterInfoBlockSize,
                       FinalizeClusterInfoBlockSize,
                       ClearInvalidCellsBlockSize
                     };

  int  gridsizes[] = { Helpers::int_ceil_div(NMaxClusters, SeedCellPropertiesBlockSize),
                       Helpers::int_ceil_div(NCaloCells, CalculateClusterInfoBlockSize),
                       Helpers::int_ceil_div(NMaxClusters, FinalizeClusterInfoBlockSize),
                       Helpers::int_ceil_div(NCaloCells, ClearInvalidCellsBlockSize)
                     };

  optimizer.register_kernels("BasicClusterInfoCalculator", 4, kernels, blocksizes, gridsizes);
}