#include "hip/hip_runtime.h"
//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//
// Dear emacs, this is -*- c++ -*-
//

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "TopoAutomatonClusteringImpl.h"

#include "CaloIdentifier/LArNeighbours.h"
//It's just a struct.

#include <cstring>
#include <cmath>
#include <iostream>
#include <stdio.h>
#include <cstddef>
#include <stdexcept>

#include "FPHelpers.h"


#include <hip/hip_cooperative_groups.h>

#include "CaloRecGPU/IGPUKernelSizeOptimizer.h"

using namespace CaloRecGPU;
using namespace TAGrowing;

void TAGrowing::TACOptionsHolder::allocate()
{
  m_options.allocate();
  m_options_dev.allocate();
}

void TAGrowing::TACOptionsHolder::sendToGPU(const bool clear_CPU)
{
  m_options_dev = m_options;
  if (clear_CPU)
    {
      m_options.clear();
    }
}

namespace TACHacks
{
  TopoAutomatonGrowingTemporaries * get_temporaries(EventDataHolder & edh)
  {
    return (TopoAutomatonGrowingTemporaries *) ((void *) ((ClusterMomentsArr *) edh.m_moments_dev));
  }
}


//constexpr static int DefaultBlockSize = 256;

constexpr static int SignalToNoiseBlockSize = 512;

constexpr static int CellPairsBlockSize = 256;

constexpr static int ClusterGrowingMainPropagationBlockSize = 1024;

constexpr static int ClusterGrowingPropagationBlockSize = 256;
constexpr static int ClusterGrowingCopyAndCheckBlockSize = 512;
constexpr static int ClusterGrowingTerminalPropagationBlockSize = 256;

constexpr static int ClusterGrowingSeedCellAssignmentBlockSize = 256;
constexpr static int ClusterGrowingFinalizationBlockSize = 512;


/******************************************************************************
 * Kernel to calculate the signal-to-noise ratio of cell energy deposition,
 * classify seed, growing, terminal cells and create the clusters for the seeds.
 ******************************************************************************/

static __global__
void signalToNoiseKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                         Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                         Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries,
                         const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                         const Helpers::CUDA_kernel_object<CellNoiseArr> noise_arr,
                         const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                         const Helpers::CUDA_kernel_object<TopoAutomatonOptions> opts)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  const int grid_size = gridDim.x * blockDim.x;

  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const int cell_sampling = geometry->sampling(index);
      const float cellEnergy = cell_info_arr->energy[cell];

      if (!cell_info_arr->is_valid(cell) || !opts->uses_calorimeter_by_sampling(cell_sampling))
        {
          cell_state_arr->clusterTag[cell] = TACTag::make_invalid_tag();
          temporaries->secondary_array[cell] = TACTag::make_invalid_tag();
          return;
        }

      float sigNoiseRatio = 0.00001f;
      //It's what's done in the CPU implementation...
      if (!cell_info_arr->is_bad(cell, opts->treat_L1_predicted_as_good))
        {
          const int gain = cell_info_arr->gain[cell];

          float cellNoise = 0.f;

          if (opts->use_two_gaussian && geometry->is_tile(cell))
            {
              cellNoise = noise_arr->get_double_gaussian_noise(cell, gain, cellEnergy);
            }
          else
            {
              cellNoise = noise_arr->get_noise(cell, gain);
            }

          if (isfinite(cellNoise) && cellNoise > 0.0f)
            {
              sigNoiseRatio = cellEnergy / cellNoise;
            }
        }

      const float absRatio = fabsf(sigNoiseRatio);

      bool can_be_seed = (opts->abs_seed ? absRatio : sigNoiseRatio) > opts->seed_threshold;
      bool can_be_grow = (opts->abs_grow ? absRatio : sigNoiseRatio) > opts->grow_threshold;
      bool can_be_term = (opts->abs_terminal ? absRatio : sigNoiseRatio) > opts->terminal_threshold;

      if (can_be_seed && opts->use_time_cut && (!opts->keep_significant_cells || sigNoiseRatio <= opts->snr_threshold_for_keeping_cells))
        {
          if (!cell_info_arr->passes_time_cut(*geometry, cell, opts->time_threshold, opts->use_crosstalk, opts->crosstalk_delta))
            {
              can_be_seed = false;
              if (opts->completely_exclude_cut_seeds)
                {
                  can_be_grow = false;
                  can_be_term = false;
                }
            }
        }


      if (can_be_seed && opts->uses_seed_sampling(cell_sampling))
        {
          const int n = atomicAdd(&(clusters_arr->number), 1);

          const unsigned int SNR_pattern = __float_as_int(opts->abs_seed ? absRatio : sigNoiseRatio);
          //In principle, we would expect
          //the seed threshold to always be positive,
          //so we could use absRatio by default.
          //However, since we can support
          //also the most general case
          //with total-ordered floating points,
          //why not do it?

          const unsigned int ordered_SNR_pattern = FloatingPointHelpers::StandardFloat::template to_total_ordering<uint32_t>(SNR_pattern);

          const TACTag tag = TACTag::make_seed_tag(cell, ordered_SNR_pattern, can_be_grow);
          //As per the CPU algorithm,
          //if a cell does not pass the grow threshold
          //(which can happen if seeds are being evaluated
          // as absolute value while growing cells are not),
          //the clusters cannot be merged. Somehow.

          cell_state_arr->clusterTag[cell] = tag;
          temporaries->secondary_array[cell] = tag;

          temporaries->cell_to_cluster_map[cell] = n;

          unsigned long long int snr_and_cell = ordered_SNR_pattern;

          snr_and_cell = (snr_and_cell << 32) | cell;

          temporaries->seed_cell_table[n] = snr_and_cell;
        }
      else if (can_be_grow)
        {
          cell_state_arr->clusterTag[cell] = TACTag::make_grow_tag();
          temporaries->secondary_array[cell] = TACTag::make_grow_tag();
        }
      else if (can_be_term)
        {
          cell_state_arr->clusterTag[cell] = TACTag::make_terminal_tag();
          temporaries->secondary_array[cell] = TACTag::make_terminal_tag();
        }
      else //is invalid for propagation
        {
          cell_state_arr->clusterTag[cell] = TACTag::make_invalid_tag();
          temporaries->secondary_array[cell] = TACTag::make_invalid_tag();
        }
    }
}


void TAGrowing::signalToNoise(EventDataHolder & holder,
                              const ConstantDataHolder & instance_data,
                              const TACOptionsHolder & options,
                              const IGPUKernelSizeOptimizer & optimizer,
                              const bool synchronize,
                              CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  hipMemsetAsync(&(holder.m_clusters_dev->number), 0, sizeof(holder.m_clusters_dev->number), stream_to_use);

  const CUDAKernelLaunchConfiguration config = optimizer.get_launch_configuration("TopoAutomatonGrowing", 0);

  signalToNoiseKernel <<< config.grid_x, config.block_x, 0, stream_to_use>>>(holder.m_cell_state_dev,
                                                                             holder.m_clusters_dev,
                                                                             TACHacks::get_temporaries(holder),
                                                                             holder.m_cell_info_dev,
                                                                             instance_data.m_cell_noise_dev,
                                                                             instance_data.m_geometry_dev,
                                                                             options.m_options_dev);
  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}


/******************************************************************************
 * Kernel to generate the cell pairs for the growing algorithm.
 ******************************************************************************/

static __global__
void cellPairsKernel(Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries,
                     const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                     const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                     const Helpers::CUDA_kernel_object<TopoAutomatonOptions> opts)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  const int grid_size = gridDim.x * blockDim.x;

  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const TACTag this_tag = cell_state_arr->clusterTag[cell];

      if (this_tag.is_grow_or_seed())
        {
          int neighbours[NMaxNeighbours];

          int num_grow_neighs = 0, num_term_neighs = 0;

          const bool is_limited = ( opts->limit_HECIW_and_FCal_neighs && geometry->is_HECIW_or_FCal(cell) ) ||
                                  ( opts->limit_PS_neighs             && geometry->is_PS(cell)            );

          const unsigned int limited_flags  = LArNeighbours::neighbourOption::nextInSamp & opts->neighbour_options;

          const unsigned int neighbour_option = (is_limited ? limited_flags : opts->neighbour_options);

          const int num_neighs = geometry->get_neighbours(neighbour_option, cell, neighbours);

          constexpr int grow_seed_neighbour_mark = 0x100000;
          constexpr int term_neighbour_mark      = 0x200000;
          //Mark growing or terminal neighbours.

          for (int i = 0; i < num_neighs; ++i)
            {
              const int neigh_ID = neighbours[i];
              const TACTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];
              if (neigh_tag.is_grow_or_seed())
                {
                  neighbours[i] |= grow_seed_neighbour_mark;
                  ++num_grow_neighs;
                }
              else if (neigh_tag.is_non_assigned_terminal())
                {
                  neighbours[i] |= term_neighbour_mark;
                  ++num_term_neighs;
                }
            }

          int seedgrow_pair_index = atomicAdd(&(temporaries->seedgrow_pairs.number), num_grow_neighs);
          int term_pair_index = atomicAdd(&(temporaries->term_pairs.number), num_term_neighs);
          constexpr int clear_flags_mask = ~(grow_seed_neighbour_mark | term_neighbour_mark);

          for (int i = 0; i < num_neighs; ++i)
            {
              const int neigh = neighbours[i];
              if (neigh & grow_seed_neighbour_mark)
                {
                  temporaries->seedgrow_pairs.cellID[seedgrow_pair_index] = neigh & clear_flags_mask;
                  temporaries->seedgrow_pairs.neighbourID[seedgrow_pair_index] = cell;
                  ++seedgrow_pair_index;
                }
              else if (neigh & term_neighbour_mark)
                {
                  temporaries->term_pairs.cellID[term_pair_index] = neigh & clear_flags_mask;
                  temporaries->term_pairs.neighbourID[term_pair_index] = cell;
                  ++term_pair_index;
                }
            }
        }
    }
}


void TAGrowing::cellPairs(EventDataHolder & holder,
                          const ConstantDataHolder & instance_data,
                          const TACOptionsHolder & options,
                          const IGPUKernelSizeOptimizer & optimizer,
                          const bool synchronize,
                          CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  TopoAutomatonGrowingTemporaries * temps = TACHacks::get_temporaries(holder);

  hipMemsetAsync(&(temps->seedgrow_pairs.number), 0, sizeof(int), stream_to_use);
  hipMemsetAsync(&(temps->term_pairs.number), 0, sizeof(int), stream_to_use);

  const CUDAKernelLaunchConfiguration config = optimizer.get_launch_configuration("TopoAutomatonGrowing", 1);

  cellPairsKernel <<< config.grid_x, config.block_x, 0, stream_to_use>>>(temps,
                                                                         holder.m_cell_state_dev,
                                                                         instance_data.m_geometry_dev,
                                                                         options.m_options_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}



/******************************************************************************
 * Series of kernels for the growing algorithm!
 ******************************************************************************/

__device__ static
void propagate_through_pair_main(const int pair,
                                 Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                 Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries)
{
  //Maybe we should go back to the old idea of per-cell iteration
  //since each thread is anyway evaluating multiple things?

  const int this_ID = temporaries->seedgrow_pairs.cellID[pair];
  const int neigh_ID = temporaries->seedgrow_pairs.neighbourID[pair];

  const TACTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];

  const TACTag prop_tag = neigh_tag.propagate();

  const TACTag this_old_tag = cell_state_arr->clusterTag[this_ID];

  if (this_old_tag.is_part_of_cluster() && neigh_tag.is_part_of_cluster() && this_old_tag.can_merge())
    //If the cell was already part of a cluster,
    //we must merge the two of them.
    //Else, we keep growing.
    {
      const int this_seed_idx = this_old_tag.index();
      const int neigh_seed_idx = neigh_tag.index();
      if (this_seed_idx != neigh_seed_idx)
        {
          const int this_cluster_index = temporaries->cell_to_cluster_map[this_seed_idx];
          const int neigh_cluster_index = temporaries->cell_to_cluster_map[neigh_seed_idx];
          if (this_cluster_index != neigh_cluster_index)
            {
              if (this_cluster_index > neigh_cluster_index)
                {
                  atomicMax(&(temporaries->cell_to_cluster_map[neigh_seed_idx]), this_cluster_index);
                  atomicMax(&(temporaries->seed_cell_table[this_cluster_index]),
                            temporaries->seed_cell_table[neigh_cluster_index]    );
                }
              else /* if (neigh_cluster_index > this_cluster_index) */
                {
                  atomicMax(&(temporaries->cell_to_cluster_map[this_seed_idx]), neigh_cluster_index);
                  atomicMax(&(temporaries->seed_cell_table[neigh_cluster_index]),
                            temporaries->seed_cell_table[this_cluster_index]    );
                }
              temporaries->continue_flag = 1;
            }
        }
    }
  else if (!this_old_tag.is_part_of_cluster() && neigh_tag.is_part_of_cluster())
    {
      temporaries->continue_flag = 1;
      atomicMax(&(temporaries->secondary_array[this_ID]), prop_tag);
    }
}


__device__ static
void propagate_through_pair_terminal(const int pair,
                                     Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                     Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries)
{
  const int this_ID = temporaries->term_pairs.cellID[pair];
  const int neigh_ID = temporaries->term_pairs.neighbourID[pair];

  const TACTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];

  atomicMax(&(temporaries->secondary_array[this_ID]), neigh_tag.propagate());
}


__global__ static
void clusterGrowingMainCooperativeKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                         Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries)
{
  cooperative_groups::grid_group grid = cooperative_groups::this_grid();

  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  const int num_pairs_main = temporaries->seedgrow_pairs.number;
  const int num_pairs_term = temporaries->term_pairs.number;

  const int grid_size = grid.size();
  //Using the "legacy" version for the time being
  //due to CUDA toolkit version availability...

  //int counter = 0;

  while (!temporaries->stop_flag)
    {
      for (int pair = index; pair < num_pairs_main; pair += grid_size)
        {
          propagate_through_pair_main(pair, cell_state_arr, temporaries);
        }

      grid.sync();

      for (int cell = index; cell < NCaloCells; cell += grid_size)
        {
          cell_state_arr->clusterTag[cell] = temporaries->secondary_array[cell];
        }

      if (index == 0)
        {
          if (!temporaries->continue_flag)
            {
              temporaries->stop_flag = 1;
            }
          else
            {
              temporaries->continue_flag = 0;
            }
        }

      //++counter;

      grid.sync();
    }

  //printf("COUNTS: %16d\n", counter);

  for (int pair = index; pair < num_pairs_term; pair += grid_size)
    {
      propagate_through_pair_terminal(pair, cell_state_arr, temporaries);
    }

}

__global__ static
void assignSeedCellsKernel(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                     const Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int clusters_number = clusters_arr->number;
  const int grid_size = gridDim.x * blockDim.x;

  for (int cluster = index; cluster < clusters_number; cluster += grid_size)
    {
      const unsigned long long int SNR_and_cell = temporaries->seed_cell_table[index];
      const int cell = SNR_and_cell & 0xFFFFFU;
      clusters_arr->seedCellID[index] = cell;
    }
}

__global__ static
void finalizeClusterAttributionKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                const Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  const int grid_size = gridDim.x * blockDim.x;

  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const TACTag old_tag = temporaries->secondary_array[cell];

      if (old_tag.is_part_of_cluster())
        {
          cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(temporaries->cell_to_cluster_map[old_tag.index()]);
        }
      else
        {
          cell_state_arr->clusterTag[cell] = ClusterTag::make_invalid_tag();
        }
    }
}


namespace
{
  struct kernel_sizes
  {
    int neigh_prop, copy_and_check, term_prop, seed_assign;
  };
}

__global__ static
void propagateNeighboursKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                         Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries,
                         Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const kernel_sizes blocks, const kernel_sizes grids
#endif
                        );

__global__ static
void propagateTerminalsKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                        Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries,
                        Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const kernel_sizes blocks, const kernel_sizes grids
#endif
                       );

__global__ static
void copyTagsAndCheckTerminationKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                 Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries,
                                 Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const kernel_sizes blocks, const kernel_sizes grids
#endif
                                );

__global__ static
void propagateNeighboursKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                         Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries,
                         Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const kernel_sizes blocks, const kernel_sizes grids
#endif
                        )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  const int num_pairs = temporaries->seedgrow_pairs.number;

  for (int pair = index; pair < num_pairs; pair += grid_size)
    {
      propagate_through_pair_main(pair, cell_state_arr, temporaries);
    }
#if CUDA_CAN_USE_TAIL_LAUNCH
  if (index == grid_size - 1)
    {
      copyTagsAndCheckTerminationKernel <<< grids.copy_and_check, blocks.copy_and_check, 0, cudaStreamTailLaunch>>>(cell_state_arr,
                                                                                                              temporaries,
                                                                                                              clusters_arr,
                                                                                                              blocks, grids);
    }
#endif
}


__global__ static
void copyTagsAndCheckTerminationKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                 Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries,
                                 Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const kernel_sizes blocks, const kernel_sizes grids
#endif
                                )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      cell_state_arr->clusterTag[cell] = temporaries->secondary_array[cell];
    }

  if (index == grid_size - 1)
    {
#if CUDA_CAN_USE_TAIL_LAUNCH
      if (temporaries->continue_flag)
        {
          temporaries->continue_flag = 0;

          propagateNeighboursKernel <<< grid.neigh_prop, block.neigh_prop, 0, cudaStreamTailLaunch>>>(cell_state_arr,
                                                                                                temporaries,
                                                                                                clusters_arr,
                                                                                                blocks, grids);
        }
      else
        {

          propagateTerminalsKernel <<< grid.term_prop, block.term_prop, 0, cudaStreamTailLaunch>>>(cell_state_arr,
                                                                                             temporaries,
                                                                                             clusters_arr,
                                                                                             blocks, grids);

        }
#else
      if (!(temporaries->continue_flag))
        {
          temporaries->stop_flag = 1;
        }
      else
        {
          temporaries->continue_flag = 0;
        }
#endif
    }

}

__global__ static
void propagateTerminalsKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                        Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries,
                        Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const kernel_sizes blocks, const kernel_sizes grids
#endif
                       )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  const int num_pairs = temporaries->term_pairs.number;
  for (int pair = index; pair < num_pairs; pair += grid_size)
    {
      propagate_through_pair_terminal(pair, cell_state_arr, temporaries);
    }
#if CUDA_CAN_USE_TAIL_LAUNCH
  if (index == grid_size - 1)
    {
      assignSeedCellsKernel <<< grids.seed_assign, blocks.seed_assign, 0, cudaStreamTailLaunch>>>(clusters_arr, temporaries);
    }
#endif
}

__global__ static
void clusterGrowingMainDefer(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                         Helpers::CUDA_kernel_object<TopoAutomatonGrowingTemporaries> temporaries,
                         Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                         const kernel_sizes blocks, kernel_sizes grids)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      if (grids.neigh_prop < 0)
        //We want the minimum grid size.
        {
          grids.neigh_prop = Helpers::int_ceil_div(temporaries->seedgrow_pairs.number, blocks.neigh_prop);
          grids.term_prop = Helpers::int_ceil_div(temporaries->term_pairs.number, blocks.term_prop);
          grids.seed_assign = Helpers::int_ceil_div(clusters_arr->number, blocks.seed_assign);
        }

#if CUDA_CAN_USE_TAIL_LAUNCH

      propagateNeighboursKernel <<< grids.neigh_prop, blocks.neigh_prop, 0, cudaStreamTailLaunch>>>(cell_state_arr,
                                                                                              temporaries,
                                                                                              clusters_arr,
                                                                                              blocks, grids);

#else

      //int counter = 0;

      while (!temporaries->stop_flag)
        {
          propagateNeighboursKernel <<< grids.neigh_prop, blocks.neigh_prop>>>(cell_state_arr,
                                                                         temporaries,
                                                                         clusters_arr);

          copyTagsAndCheckTerminationKernel <<< grids.copy_and_check, blocks.copy_and_check>>>(cell_state_arr,
                                                                                         temporaries,
                                                                                         clusters_arr);


          //++counter;

        }

      //printf("COUNTS: %16d\n", counter);

      propagateTerminalsKernel <<< grids.term_prop, blocks.term_prop>>>(cell_state_arr,
                                                                  temporaries,
                                                                  clusters_arr);
      assignSeedCellsKernel <<< grids.seed_assign, blocks.seed_assign>>>(clusters_arr, temporaries);

#endif
    }
}

void TAGrowing::clusterGrowing(EventDataHolder & holder,
                               const ConstantDataHolder & instance_data,
                               const TACOptionsHolder & options,
                               const IGPUKernelSizeOptimizer & optimizer,
                               const bool synchronize,
                               CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{

  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  TopoAutomatonGrowingTemporaries * temps = TACHacks::get_temporaries(holder);

  hipMemsetAsync(&(temps->continue_flag), 0, sizeof(int), stream_to_use);
  hipMemsetAsync(&(temps->stop_flag), 0, sizeof(int), stream_to_use);


  if (optimizer.can_use_cooperative_groups())
    {
      const CUDAKernelLaunchConfiguration cfg_iter = optimizer.get_launch_configuration("TopoAutomatonGrowing", 2);
      const CUDAKernelLaunchConfiguration cfg_seed = optimizer.get_launch_configuration("TopoAutomatonGrowing", 3);

      void * main_args[] = { &holder.m_cell_state_dev, &temps };

      hipLaunchCooperativeKernel((void *) clusterGrowingMainCooperativeKernel,
                                  cfg_iter.grid_x, cfg_iter.block_x,
                                  main_args, 0, stream_to_use);
                                  
      assignSeedCellsKernel <<< cfg_seed.grid_x, cfg_seed.block_x, 0, stream_to_use>>>(holder.m_clusters_dev, temps);

    }
  else if (optimizer.can_use_dynamic_parallelism())
    {
      const CUDAKernelLaunchConfiguration cfg_seed  = optimizer.get_launch_configuration("TopoAutomatonGrowing", 3);
      const CUDAKernelLaunchConfiguration cfg_neigh = optimizer.get_launch_configuration("TopoAutomatonGrowing", 4);
      const CUDAKernelLaunchConfiguration cfg_check = optimizer.get_launch_configuration("TopoAutomatonGrowing", 5);
      const CUDAKernelLaunchConfiguration cfg_term  = optimizer.get_launch_configuration("TopoAutomatonGrowing", 6);

      kernel_sizes grids, blocks;

      if (optimizer.use_minimal_kernel_sizes())
        {
          grids.neigh_prop     = -1;
          grids.term_prop      = -1;
          grids.seed_assign    = -1;
        }
      else
        {
          grids.neigh_prop     = cfg_neigh.grid_x;
          grids.term_prop      = cfg_term.grid_x;
          grids.seed_assign    = cfg_seed.grid_x;
        }

      grids.copy_and_check = cfg_check.grid_x;

      blocks.neigh_prop     = cfg_neigh.block_x;
      blocks.copy_and_check = cfg_check.block_x;
      blocks.term_prop      = cfg_term.block_x;
      blocks.seed_assign    = cfg_seed.block_x;

      clusterGrowingMainDefer <<< 1, 1, 0, stream_to_use>>>(holder.m_cell_state_dev, temps, holder.m_clusters_dev, blocks, grids);
    }
  else
    {
      throw std::runtime_error("The GPU must support either cooperative grid launches "
                               "or dynamic parallelism for the algorithm to work properly.");
      //Well, technically, we could code something up
      //(launch, say, 64 iterations, since the maximum
      // I have seen for ttbar is around ~30
      // and iterations are idempotent after the
      // stopping criterion has been reached),
      //but the added complexity and the fact that
      //architectures without dynamic parallelism
      //either are or soon will be deprecated
      //mean that it's not worth the effort.
    }

  const CUDAKernelLaunchConfiguration cfg_final = optimizer.get_launch_configuration("TopoAutomatonGrowing", 7);

  finalizeClusterAttributionKernel <<< cfg_final.grid_x, cfg_final.block_x, 0, stream_to_use>>>(holder.m_cell_state_dev, temps);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}

/*******************************************************************************************************************************/

void TAGrowing::register_kernels(IGPUKernelSizeOptimizer & optimizer)
{
  void * kernels[] = { (void *) signalToNoiseKernel,
                       (void *) cellPairsKernel,
                       (void *) clusterGrowingMainCooperativeKernel,
                       (void *) assignSeedCellsKernel,
                       (void *) propagateNeighboursKernel,
                       (void *) copyTagsAndCheckTerminationKernel,
                       (void *) propagateTerminalsKernel,
                       (void *) finalizeClusterAttributionKernel
                     };

  int blocksizes[] = { SignalToNoiseBlockSize,
                       CellPairsBlockSize,
                       ClusterGrowingMainPropagationBlockSize,
                       ClusterGrowingSeedCellAssignmentBlockSize,
                       ClusterGrowingPropagationBlockSize,
                       ClusterGrowingCopyAndCheckBlockSize,
                       ClusterGrowingTerminalPropagationBlockSize,
                       ClusterGrowingFinalizationBlockSize
                     };

  int  gridsizes[] = { Helpers::int_ceil_div(NCaloCells, SignalToNoiseBlockSize),
                       Helpers::int_ceil_div(NCaloCells, CellPairsBlockSize),
                       IGPUKernelSizeOptimizer::SpecialSizeHints::CooperativeLaunch,
                       Helpers::int_ceil_div(NMaxClusters, ClusterGrowingSeedCellAssignmentBlockSize),
                       Helpers::int_ceil_div(NMaxPairs, ClusterGrowingPropagationBlockSize),
                       Helpers::int_ceil_div(NCaloCells, ClusterGrowingCopyAndCheckBlockSize),
                       Helpers::int_ceil_div(NMaxPairs, ClusterGrowingTerminalPropagationBlockSize),
                       Helpers::int_ceil_div(NCaloCells, ClusterGrowingFinalizationBlockSize)
                     };

  optimizer.register_kernels("TopoAutomatonGrowing", 8, kernels, blocksizes, gridsizes);
}