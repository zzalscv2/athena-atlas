#include "hip/hip_runtime.h"
//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//
// Dear emacs, this is -*- c++ -*-
//

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "TopoAutomatonSplittingImpl.h"

#include "CaloIdentifier/LArNeighbours.h"
//It's just a struct.


#include "CLHEP/Units/SystemOfUnits.h"
//Probably will also work, given that it's just constexpr stuff.

#include <cstring>
#include <cmath>
#include <iostream>
#include <stdio.h>

#include <hip/hip_cooperative_groups.h>


#include "CaloRecGPU/IGPUKernelSizeOptimizer.h"

using namespace CaloRecGPU;
using namespace TASplitting;

void TASplitting::TASOptionsHolder::allocate()
{
  m_options.allocate();
  m_options_dev.allocate();
}

void TASplitting::TASOptionsHolder::sendToGPU(const bool clear_CPU)
{
  m_options_dev = m_options;
  if (clear_CPU)
    {
      m_options.clear();
    }
}

constexpr static int FillNeighboursBlockSize = 128;

constexpr static int CountInferiorNeighsBlockSize = 256;
constexpr static int FindLocalMaximaBlockSize = 512;
constexpr static int FixClustersWithoutMaximaBlockSize = 512;

constexpr static int ExcludeMaximaPropagationBlockSize = 512;
constexpr static int ExcludeMaximaOnePropagationBlockSize = 512;
constexpr static int CleanUpSecondariesBlockSize = 512;


constexpr static int ClusterSplittingMainPropagationBlockSize = 1024;
constexpr static int PropagateSplitTagsBlockSize = 256;
constexpr static int HandleSplitTagChangesBlockSize = 256;

constexpr static int SumCellsBlockSize = 320;
constexpr static int CalculateCentroidsBlockSize = 512;
constexpr static int FinalCellsBlockSize = 512;

//These numbers are not at all optimized,
//just going from rough similarity to TAC operations
//(which themselves are not that optimised
// since they were last tested on a previous version...)

namespace TASHacks
{
  TopoAutomatonSplittingTemporaries * get_temporaries(EventDataHolder & edh)
  {
    return (TopoAutomatonSplittingTemporaries *) ((void *) ((ClusterMomentsArr *) edh.m_moments_dev));
  }
}

/******************************************************************************************
 * Determine the same-cluster neighbours of the cells and fill the pairs list accordingly.
 ******************************************************************************************/

namespace
{
  constexpr int LocalMaximaDetection = 0;
}

//Note: we might benefit from using shared memory
//      to hold the neighbours for each thread.
//      See how is register pressure...
//      (Probably not too bad, max. 255 per thread...)

static __global__
void fillNeighboursKernel(Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                          const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                          const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                          const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  const int grid_size = gridDim.x * blockDim.x;

  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const ClusterTag this_tag = cell_state_arr->clusterTag[cell];

      int neighbours[NMaxNeighbours];
      int num_normal = 0, num_extra = 0, num_next = 0, num_prev = 0, num_limited = 0, total_neighs = 0;

      constexpr int non_neighbour_mark = 0x100000;
      //Will mark neighbours that are not part of the same cluster;

      const bool is_limited = ( opts->limit_HECIW_and_FCal_neighs && geometry->is_HECIW_or_FCal(cell) )  ||
                              ( opts->limit_PS_neighs             && geometry->is_PS(cell)            );
      //The cells that have limited neighbours, for the split cluster growing part.
      //WARNING: the CPU version of the code does not limit PS neighbours ever, but we give additional freedom
      //         (even if it is disabled by default).

      if (this_tag.is_part_of_cluster())
        {
          const unsigned int limited_flags   = LArNeighbours::neighbourOption::nextInSamp & opts->neighbour_options;

          const unsigned int pre_next_flags  = LArNeighbours::neighbourOption::nextSuperCalo & opts->neighbour_options;

          const unsigned int prev_flags      = ( LArNeighbours::neighbourOption::prevSuperCalo |
                                                 LArNeighbours::neighbourOption::prevInSamp      ) & opts->neighbour_options;

          const unsigned int covered_flags   = limited_flags | pre_next_flags | prev_flags;

          const unsigned int remaining_flags = (~covered_flags) & opts->neighbour_options;

          num_limited = geometry->get_neighbours(limited_flags, cell, neighbours);

          num_next = num_limited + geometry->get_neighbours(pre_next_flags, cell, neighbours + num_limited);

          num_prev = geometry->get_neighbours(prev_flags, cell, neighbours + num_next);

          const int num_rest = geometry->get_neighbours(remaining_flags, cell, neighbours + num_next + num_prev);

          total_neighs = num_next + num_prev + num_rest;

          for (int i = 0; i < total_neighs; ++i)
            {
              const int neigh_ID = neighbours[i];
              const ClusterTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];
              if (neigh_tag.is_part_of_cluster() && this_tag.cluster_index() == neigh_tag.cluster_index())
                {
                  if (is_limited && i >= num_limited)
                    {
                      ++num_extra;
                    }
                  else
                    {
                      ++num_normal;
                    }
                }
              else
                {
                  neighbours[i] |= non_neighbour_mark;
                }
            }
        }
      else
        {
          const unsigned int next_flags = ( LArNeighbours::neighbourOption::nextSuperCalo |
                                            LArNeighbours::neighbourOption::nextInSamp      ) & opts->neighbour_options;

          const unsigned int prev_flags = ( LArNeighbours::neighbourOption::prevSuperCalo |
                                            LArNeighbours::neighbourOption::prevInSamp      ) & opts->neighbour_options;

          num_next = geometry->get_neighbours(next_flags, cell, neighbours);
          num_prev = geometry->get_neighbours(prev_flags, cell, neighbours + num_next);
          total_neighs = num_next + num_prev;

          for (int i = 0; i < total_neighs; ++i)
            {
              neighbours[i] |= non_neighbour_mark;
            }
        }

      constexpr int WarpSize = 32;
      constexpr unsigned int full_mask = 0xFFFFFFFFU;
      const int intra_warp_index = threadIdx.x % WarpSize;

      int normal_prefix = num_normal;
      int next_prefix   = num_next;
      int prev_prefix   = num_prev;
      int extra_prefix  = num_extra;

      for (int i = 1; i < WarpSize; i *= 2)
        {
          const int other_normal = __shfl_down_sync (full_mask, normal_prefix, i) * (intra_warp_index + i < WarpSize);
          const int other_next   = __shfl_down_sync (full_mask, next_prefix,   i) * (intra_warp_index + i < WarpSize);
          const int other_prev   = __shfl_up_sync   (full_mask, prev_prefix,   i) * (intra_warp_index >= i);
          const int other_extra  = __shfl_up_sync   (full_mask, extra_prefix,  i) * (intra_warp_index >= i);

          normal_prefix += other_normal;
          next_prefix   += other_next;
          prev_prefix   += other_prev;
          extra_prefix  += other_extra;
        }


      int real_next_prefix = 0;
      int real_prev_prefix = 0;

      if (intra_warp_index <= 1)
        {
          real_next_prefix = __shfl_sync(0x00000003U, next_prefix, 0);
        }
      else if (intra_warp_index >= WarpSize - 2)
        {
          real_prev_prefix = __shfl_sync(0xC0000000U, prev_prefix, WarpSize - 1);
        }

      int base_normal_offset = 0;
      int base_next_offset   = 0;
      int base_prev_offset   = 0;
      int base_extra_offset  = 0;

      switch (intra_warp_index)
        {
          case 0:
            base_normal_offset = atomicAdd(&(temporaries->pairs.number_normal), normal_prefix);
            break;
          case 1:
            base_next_offset   = atomicAdd(&(temporaries->pairs.number_next),   real_next_prefix);
            break;
          case WarpSize - 2:
            base_prev_offset   = atomicAdd(&(temporaries->pairs.number_prev),   real_prev_prefix);
            break;
          case WarpSize - 1:
            base_extra_offset  = atomicAdd(&(temporaries->pairs.number_extra),  extra_prefix);
            break;
          default:
            break;

        }

      const int normal_offset = __shfl_sync(full_mask, base_normal_offset,            0) + normal_prefix - num_normal;
      const int next_offset   = __shfl_sync(full_mask, base_next_offset,              1) + next_prefix   - num_next;
      const int prev_offset   = __shfl_sync(full_mask, base_prev_offset,   WarpSize - 2) + prev_prefix   - num_prev;
      const int extra_offset  = __shfl_sync(full_mask, base_extra_offset,  WarpSize - 1) + extra_prefix  - num_extra;

      int normal_pair_index   = normal_offset;
      int next_pair_index     = TopoAutomatonSplittingTemporaries::PairsArr::s_intermediate_mark - next_offset - num_next;
      int prev_pair_index     = TopoAutomatonSplittingTemporaries::PairsArr::s_intermediate_mark + prev_offset;
      int extra_pair_index    = TopoAutomatonSplittingTemporaries::PairsArr::s_size - extra_offset - num_extra;

      for (int i = 0; i < num_limited; ++i)
        {
          const int neigh = neighbours[i];
          if (neigh < non_neighbour_mark)
            {
              temporaries->pairs.cellID[normal_pair_index] = neigh;
              temporaries->pairs.neighbourID[normal_pair_index] = cell;
              ++normal_pair_index;
            }
          temporaries->pairs.cellID[next_pair_index] = neigh & (~non_neighbour_mark);
          temporaries->pairs.neighbourID[next_pair_index] = cell;
          ++next_pair_index;
        }
      for (int i = num_limited; i < num_next; ++i)
        {
          const int neigh = neighbours[i];
          if (neigh < non_neighbour_mark)
            {
              int & pair_index = (is_limited ? extra_pair_index : normal_pair_index);
              temporaries->pairs.cellID[pair_index] = neigh;
              temporaries->pairs.neighbourID[pair_index] = cell;
              ++pair_index;
            }
          temporaries->pairs.cellID[next_pair_index] = neigh & (~non_neighbour_mark);
          temporaries->pairs.neighbourID[next_pair_index] = cell;
          ++next_pair_index;
        }
      for (int i = num_next; i < num_next + num_prev; ++i)
        {
          const int neigh = neighbours[i];
          if (neigh < non_neighbour_mark)
            {
              int & pair_index = (is_limited ? extra_pair_index : normal_pair_index);
              temporaries->pairs.cellID[pair_index] = neigh;
              temporaries->pairs.neighbourID[pair_index] = cell;
              ++pair_index;
            }
          temporaries->pairs.cellID[prev_pair_index] = neigh & (~non_neighbour_mark);
          temporaries->pairs.neighbourID[prev_pair_index] = cell;
          ++prev_pair_index;
        }
      for (int i = num_next + num_prev; i < total_neighs; ++i)
        {
          const int neigh = neighbours[i];
          if (neigh < non_neighbour_mark)
            {
              int & pair_index = (is_limited ? extra_pair_index : normal_pair_index);
              temporaries->pairs.cellID[pair_index] = neigh;
              temporaries->pairs.neighbourID[pair_index] = cell;
              ++pair_index;
            }
        }

      if (this_tag.is_part_of_cluster())
        {

          temporaries->get_cells_extra_array<LocalMaximaDetection, int>(cell) = num_normal + num_extra;
        }
      else
        {

          temporaries->get_cells_extra_array<LocalMaximaDetection, int>(cell) = -NMaxNeighbours;
        }
    }
}

void TASplitting::fillNeighbours(EventDataHolder & holder,
                                 const ConstantDataHolder & instance_data,
                                 const TASOptionsHolder & options,
                                 const IGPUKernelSizeOptimizer & optimizer,
                                 const bool synchronize,
                                 CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  TopoAutomatonSplittingTemporaries * temps = TASHacks::get_temporaries(holder);

  hipMemsetAsync(&(temps->pairs.number_normal), 0, sizeof(int), stream_to_use);
  hipMemsetAsync(&(temps->pairs.number_next),   0, sizeof(int), stream_to_use);
  hipMemsetAsync(&(temps->pairs.number_prev),   0, sizeof(int), stream_to_use);
  hipMemsetAsync(&(temps->pairs.number_extra),  0, sizeof(int), stream_to_use);

  const CUDAKernelLaunchConfiguration config = optimizer.get_launch_configuration("TopoAutomatonSplitting", 0);

  fillNeighboursKernel <<< config.grid_x, config.block_x, 0, stream_to_use>>>(temps,
                                                                              holder.m_cell_state_dev,
                                                                              instance_data.m_geometry_dev,
                                                                              options.m_options_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}


/******************************************************************************************
 * Determine the local maxima and initialize the cell arrays appropriately.
 ******************************************************************************************/

static __global__
void countInferiorNeighsKernel(Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                               const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                               const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                               const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts)
{
  const int num_normal_pairs = temporaries->pairs.number_normal;
  const int num_extra_pairs = temporaries->pairs.number_extra;
  const int start_extra_pairs = TopoAutomatonSplittingTemporaries::PairsArr::s_size - num_extra_pairs;
  const int num_total_pairs = num_normal_pairs + num_extra_pairs;

  const int thread_index = blockIdx.x * blockDim.x + threadIdx.x;

  const int grid_size = gridDim.x * blockDim.x;

  for (int pair = thread_index; pair < num_total_pairs; pair += grid_size)
    {
      const int real_pair = ( pair >= num_normal_pairs ?
                              start_extra_pairs + pair - num_normal_pairs : pair);

      const int this_ID = temporaries->pairs.cellID[real_pair];
      const int neigh_ID = temporaries->pairs.neighbourID[real_pair];

      const int this_sampling = geometry->sampling(this_ID);
      const int neigh_sampling = geometry->sampling(neigh_ID);

      float this_energy = 0.f, neigh_energy = 0.f;

      if (!cell_info_arr->is_bad(this_ID, opts->treat_L1_predicted_as_good) && opts->uses_sampling(this_sampling))
        {
          this_energy = cell_info_arr->energy[this_ID];
          if (opts->use_absolute_energy)
            {
              this_energy = fabsf(this_energy);
            }
          else if (this_energy <= 0.f)
            {
              this_energy = 0.f;
            }
        }

      if (!cell_info_arr->is_bad(neigh_ID, opts->treat_L1_predicted_as_good) && opts->uses_sampling(neigh_sampling))
        {
          neigh_energy = cell_info_arr->energy[neigh_ID];
          if (opts->use_absolute_energy)
            {
              neigh_energy = fabsf(neigh_energy);
            }
          else if (neigh_energy <= 0.f)
            {
              neigh_energy = 0.f;
            }
        }

      bool is_max_neig = neigh_energy > this_energy;

      if (opts->uses_primary_sampling(neigh_sampling))
        {
          if (!opts->uses_primary_sampling(this_sampling) && opts->uses_secondary_sampling(this_sampling))
            {
              is_max_neig = true;
            }
        }

      if (!is_max_neig)
        {
          temporaries->get_cells_extra_array<LocalMaximaDetection, int>(neigh_ID) = -NCaloCells;
          //No need to count, just to invalidate!
        }
    }
}

static __global__
void countNeighsDeferrerKernel(Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                               const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                               const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                               const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts,
                               const int i_dimBlock)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      const int normal_pairs = temporaries->pairs.number_normal;
      const int extra_pairs  = temporaries->pairs.number_extra;

      const int num_total_pairs = normal_pairs + extra_pairs;

      const int i_dimGrid = Helpers::int_ceil_div(num_total_pairs, i_dimBlock);

#if CUDA_CAN_USE_TAIL_LAUNCH
      countInferiorNeighsKernel <<< i_dimGrid, i_dimBlock, 0, cudaStreamTailLaunch>>>(temporaries,
                                                                                      cell_info_arr,
                                                                                      geometry,
                                                                                      opts);
#else
      countInferiorNeighsKernel <<< i_dimGrid, i_dimBlock>>>(temporaries,
                                                             cell_info_arr,
                                                             geometry,
                                                             opts);
#endif
    }
}

static __global__
void findLocalMaximaKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                           Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                           Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                           const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                           const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                           const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;

  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const ClusterTag this_tag = cell_state_arr->clusterTag[cell];

      if (this_tag.is_part_of_cluster())
        {
          const int this_sampling = geometry->sampling(cell);

          float cell_energy = 0.f;
          const float raw_cell_energy = cell_info_arr->energy[cell];

          if (!cell_info_arr->is_bad(cell, opts->treat_L1_predicted_as_good) && opts->uses_sampling(this_sampling))
            {
              cell_energy = raw_cell_energy;
              if (opts->use_absolute_energy)
                {
                  cell_energy = fabsf(cell_energy);
                }
              else if (cell_energy <= 0.f)
                {
                  cell_energy = 0.f;
                }
            }

          const int num_neighs = temporaries->get_cells_extra_array<LocalMaximaDetection, int>(cell);

          bool is_primary = false, is_maximum = false;

          if (/*num_neighs >= 0 && */ num_neighs >= opts->min_num_cells && cell_energy >= opts->min_maximum_energy)
            {
              if (opts->uses_primary_sampling(this_sampling))
                {
                  is_maximum = true;
                  is_primary = true;
                }
              else if (opts->uses_secondary_sampling(this_sampling))
                {
                  is_maximum = true;
                  is_primary = false;
                }
            }

          if (is_maximum)
            {
              const int original_cluster = this_tag.cluster_index();

              const int new_cluster = atomicAdd(&(clusters_arr->number), 1);

              const TASTag new_tag = TASTag::make_maximum_tag(new_cluster, __float_as_uint(raw_cell_energy), is_primary);

              cell_state_arr->clusterTag[cell] = new_tag;

              clusters_arr->seedCellID[new_cluster] = cell;

              clusters_arr->seedCellID[original_cluster] = -1;

              temporaries->original_cluster_map[new_cluster] = original_cluster;

            }
          else
            {
              const int this_cluster = this_tag.cluster_index();

              cell_state_arr->clusterTag[cell] = TASTag::make_cluster_cell_tag(this_cluster);
            }
        }
      else
        {
          cell_state_arr->clusterTag[cell] = TASTag::make_invalid_tag();
        }
    }
}

static __global__
void fixClustersWithoutMaximaKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                    Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                    const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  const int grid_size = gridDim.x * blockDim.x;

  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      TASTag this_tag = cell_state_arr->clusterTag[cell];

      if (this_tag.is_part_of_splitter_cluster())
        {
          const int cluster_index = this_tag.index();

          if (this_tag.is_secondary())
            {
              temporaries->secondary_array[cell] = this_tag.update_index(cell);
              temporaries->tertiary_array[cell] = this_tag.update_index(cell);
              //So that we get the correct secondary ordering.
            }
          else
            {
              temporaries->secondary_array[cell] = TASTag::secondary_maxima_eliminator();
              temporaries->tertiary_array[cell] = TASTag::secondary_maxima_eliminator();
            }

          temporaries->cell_to_cluster_map[cell] = cluster_index;
        }
      else if (this_tag.is_valid())
        //It'll be part of an original cluster, given how we've assigned the tags.
        {
          const int cluster_index = this_tag.index();

          if (clusters_arr->seedCellID[cluster_index] >= 0)
            //This means the cluster is not split.
            {
              TASTag new_tag = TASTag::make_original_cluster_tag(cluster_index);
              cell_state_arr->clusterTag[cell] = new_tag;

              temporaries->secondary_array[cell] = TASTag::make_invalid_tag();
              temporaries->tertiary_array[cell] = TASTag::make_invalid_tag();
            }
          else
            //This means the tag belongs to a cluster to be split
            {
              temporaries->secondary_array[cell] = TASTag::make_invalid_tag();
              temporaries->tertiary_array[cell]  = TASTag::make_invalid_tag();
            }

          temporaries->cell_to_cluster_map[cell] = cluster_index;
          temporaries->original_cluster_map[cluster_index] = cluster_index;
        }
      else
        {
          temporaries->secondary_array[cell] = TASTag::make_invalid_tag();
          temporaries->tertiary_array[cell]  = TASTag::make_invalid_tag();
          temporaries->cell_to_cluster_map[cell] = -1;
        }
    }
}

void TASplitting::findLocalMaxima(EventDataHolder & holder,
                                  const ConstantDataHolder & instance_data,
                                  const TASOptionsHolder & options,
                                  const IGPUKernelSizeOptimizer & optimizer,
                                  const bool synchronize,
                                  CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  TopoAutomatonSplittingTemporaries * temps = TASHacks::get_temporaries(holder);


  const CUDAKernelLaunchConfiguration cfg_neigh_count = optimizer.get_launch_configuration("TopoAutomatonSplitting", 1);
  const CUDAKernelLaunchConfiguration cfg_find_maxima = optimizer.get_launch_configuration("TopoAutomatonSplitting", 2);
  const CUDAKernelLaunchConfiguration cfg_fix_non_max = optimizer.get_launch_configuration("TopoAutomatonSplitting", 3);

  if (optimizer.use_minimal_kernel_sizes() && optimizer.can_use_dynamic_parallelism())
    {

      countNeighsDeferrerKernel <<< 1, 1, 0, stream_to_use>>>(temps,
                                                              holder.m_cell_info_dev,
                                                              instance_data.m_geometry_dev,
                                                              options.m_options_dev,
                                                              cfg_neigh_count.block_x);
    }
  else
    {
      countInferiorNeighsKernel <<< cfg_neigh_count.grid_x, cfg_neigh_count.block_x, 0, stream_to_use>>>(temps,
                                                                                                         holder.m_cell_info_dev,
                                                                                                         instance_data.m_geometry_dev,
                                                                                                         options.m_options_dev);
    }


  findLocalMaximaKernel <<< cfg_find_maxima.grid_x, cfg_find_maxima.block_x, 0, stream_to_use>>>(holder.m_cell_state_dev,
                                                                                                 holder.m_clusters_dev,
                                                                                                 temps,
                                                                                                 holder.m_cell_info_dev,
                                                                                                 instance_data.m_geometry_dev,
                                                                                                 options.m_options_dev);

  fixClustersWithoutMaximaKernel <<< cfg_fix_non_max.grid_x, cfg_fix_non_max.block_x, 0, stream_to_use>>>(holder.m_cell_state_dev,
                                                                                                          temps,
                                                                                                          holder.m_clusters_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}

/*****************************************************************************
 * Delete secondary maxima according to the criteria on the CPU version.
 ******************************************************************************/

__device__ static
void propagate_secondary_maxima_pair(const int pair,
                                     Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries)
{
  const int this_ID = temporaries->pairs.cellID[pair];
  const int neigh_ID = temporaries->pairs.neighbourID[pair];

  tag_type * array = ( pair < TopoAutomatonSplittingTemporaries::PairsArr::s_intermediate_mark ?
                       temporaries->secondary_array :
                       temporaries->tertiary_array    );

  const TASTag this_tag = array[this_ID];
  const TASTag neigh_tag = array[neigh_ID];

  if (this_tag.is_secondary_maxima_eliminator() || this_tag.is_secondary_maximum_seed())
    {
      if (atomicMax(&(array[neigh_ID]), this_tag) < this_tag)
        {
          temporaries->continue_flag = 1;
        }
    }
}


__global__ static
void secondaryMaximaCooperativeKernel(Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries)
{
  cooperative_groups::grid_group grid = cooperative_groups::this_grid();

  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  const int grid_size = grid.size();
  //Using the "legacy" version for the time being
  //due to CUDA toolkit version availability...

  const int next_number = temporaries->pairs.number_next;
  const int pair_start  = TopoAutomatonSplittingTemporaries::PairsArr::s_intermediate_mark - next_number;
  const int pair_number = temporaries->pairs.number_prev + next_number;

  //int counter = 0;

  while (!temporaries->stop_flag)
    {
      for (int pair = index; pair < pair_number; pair += grid_size)
        {
          const int pair_index = pair_start + pair;
          propagate_secondary_maxima_pair(pair_index, temporaries);
        }

      grid.sync();

      if (index == 0)
        {
          if (!temporaries->continue_flag)
            {
              temporaries->stop_flag = 1;
            }
          else
            {
              temporaries->continue_flag = 0;
            }
        }

      grid.sync();

      //++counter;
    }

  //if (index == 0)
  //  {
  //    printf("SECONDARY SPLITTING: %16d\n", counter);
  //  }

}

__global__ static
void checkForMaximaExclusionTermination(Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const int i_dimBlock, const int i_dimGrid
#endif
                                       );

__global__ static
void propagateForMaximaExclusionKernel( Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const int i_dimBlock, const int i_dimGrid
#endif
                                      )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;

  const int next_number = temporaries->pairs.number_next;
  const int pair_start  = TopoAutomatonSplittingTemporaries::PairsArr::s_intermediate_mark - next_number;
  const int pair_number = temporaries->pairs.number_prev + next_number;

  for (int pair = index; pair < pair_number; pair += grid_size)
    {
      const int pair_index = pair_start + pair;

      propagate_secondary_maxima_pair(pair_index, temporaries);

    }
#if CUDA_CAN_USE_TAIL_LAUNCH
  if (index == grid_size - 1)
    {
      checkForMaximaExclusionTermination <<< 1, 1, 0, cudaStreamTailLaunch>>>(temporaries, i_dimBlock, i_dimGrid);
    }
#endif
}

__global__ static
void checkForMaximaExclusionTermination(Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const int i_dimBlock, const int i_dimGrid
#endif
                                       )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      if (temporaries->continue_flag)
        {
          temporaries->continue_flag = 0;

#if CUDA_CAN_USE_TAIL_LAUNCH
          secondaryMaximaPropagationKernel <<< i_dimGrid, i_dimBlock, 0, cudaStreamTailLaunch>>>(temporaries, i_dimBlock, i_dimGrid);

#endif
        }
#if !CUDA_CAN_USE_TAIL_LAUNCH
      else /*if (!temporaries->continue_flag)*/
        {
          temporaries->stop_flag = 1;
        }
#endif
    }
}

__global__ static
void excludeSecondaryMaximaDefer(Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                 const int i_dimBlock, int i_dimGrid)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      if (i_dimGrid < 0)
        //We want the minimum grid size.
        {
          const int pair_number = temporaries->pairs.number_prev + temporaries->pairs.number_next;

          i_dimGrid = Helpers::int_ceil_div(pair_number, i_dimBlock);

        }

#if CUDA_CAN_USE_TAIL_LAUNCH
      propagateForMaximaExclusionKernel <<< i_dimGrid, i_dimBlock, 0, cudaStreamTailLaunch>>>(temporaries, i_dimBlock, i_dimGrid);
#else
      //int counter = 0;

      while (!temporaries->stop_flag)
        {
          propagateForMaximaExclusionKernel <<< i_dimGrid, i_dimBlock>>>(temporaries);
          checkForMaximaExclusionTermination <<< 1, 1>>>(temporaries);

          //++counter;
        }

      //printf("SECONDARY SPLITTING: %16d\n", counter);
#endif
    }
}

__global__ static
void cleanUpSecondariesKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                              Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                              Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const TASTag original_tag  = cell_state_arr->clusterTag[cell];
      const TASTag tag_from_next = temporaries->secondary_array[cell];
      const TASTag tag_from_prev = temporaries->tertiary_array[cell];

      TASTag final_tag = original_tag;

      if (original_tag.is_part_of_splitter_cluster())
        {
          if (original_tag.is_secondary())
            {
              if (tag_from_next.index() == cell && tag_from_prev.index() == cell)
                {
                  final_tag = final_tag.set_primary();
                  final_tag = final_tag.update_index(cell);
                }
              else
                {
                  const int original_index = original_tag.index();
                  const int original_cluster = temporaries->original_cluster_map[original_index];
                  final_tag = TASTag::make_cluster_cell_tag(original_cluster);
                  clusters_arr->seedCellID[original_index] = -1;
                  temporaries->cell_to_cluster_map[cell] = original_cluster;
                }
            }
          else
            {
              final_tag = final_tag.update_index(cell);
            }
        }

      cell_state_arr->clusterTag[cell] = final_tag;
      temporaries->secondary_array[cell] = final_tag;
    }
}

void TASplitting::excludeSecondaryMaxima(EventDataHolder & holder,
                                         const ConstantDataHolder & instance_data,
                                         const TASOptionsHolder & options,
                                         const IGPUKernelSizeOptimizer & optimizer,
                                         const bool synchronize,
                                         CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  TopoAutomatonSplittingTemporaries * temps = TASHacks::get_temporaries(holder);

  if (options.m_options->valid_sampling_secondary != 0)
    {
      hipMemsetAsync((&temps->continue_flag), 0, sizeof(int), stream_to_use);
      hipMemsetAsync((&temps->stop_flag), 0, sizeof(int), stream_to_use);

      if (optimizer.can_use_cooperative_groups())
        {

          const CUDAKernelLaunchConfiguration cfg_secprop = optimizer.get_launch_configuration("TopoAutomatonSplitting", 4);
          void * propagate_args[] = { &temps };

          hipLaunchCooperativeKernel((void *) secondaryMaximaCooperativeKernel,
                                      cfg_secprop.grid_x, cfg_secprop.block_x,
                                      propagate_args, 0, stream_to_use);

        }
      else if (optimizer.can_use_dynamic_parallelism())
        {
          const CUDAKernelLaunchConfiguration cfg_secprop = optimizer.get_launch_configuration("TopoAutomatonSplitting", 5);

          excludeSecondaryMaximaDefer <<< 1, 1, 0, stream_to_use>>>(temps,
                                                                    cfg_secprop.block_x,
                                                                    (optimizer.use_minimal_kernel_sizes() ? -1 : cfg_secprop.grid_x));
        }
      else
        {
          throw std::runtime_error("The GPU must support either cooperative grid launches "
                                   "or dynamic parallelism for the algorithm to work properly.");
          //Well, technically, we could code something up
          //(launch, say, 64 iterations, since the maximum
          // I have seen for ttbar is around ~9
          // and iterations are idempotent after the
          // stopping criterion has been reached),
          //but the added complexity and the fact that
          //architectures without dynamic parallelism
          //either are or soon will be deprecated
          //mean that it's not worth the effort.
        }
    }

  const CUDAKernelLaunchConfiguration cfg_cleanup = optimizer.get_launch_configuration("TopoAutomatonSplitting", 6);

  cleanUpSecondariesKernel <<< cfg_cleanup.grid_x, cfg_cleanup.block_x, 0, stream_to_use>>>(holder.m_cell_state_dev, holder.m_clusters_dev, temps);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}


/******************************************************************************************
 * Propagate the new tags and create the final clusters.
 ******************************************************************************************/

__device__ static
void propagate_main_pair(const int pair,
                         Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                         const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                         const bool counter_select,
                         const bool share_cells)
{
  const int this_ID = temporaries->pairs.cellID[pair];
  const int neigh_ID = temporaries->pairs.neighbourID[pair];

  const TASTag neigh_tag = cell_state_arr->clusterTag[neigh_ID];

  if (!neigh_tag.is_part_of_splitter_cluster())
    {
      return;
    }

  TASTag prop_tag = neigh_tag.propagate();

  const TASTag old_tag = cell_state_arr->clusterTag[this_ID];

  if ( share_cells                           &&
       old_tag.is_part_of_splitter_cluster() &&
       !old_tag.is_shared()                  &&
       !old_tag.is_primary()                 &&
       !neigh_tag.is_shared()                    )
    {
      const int old_cluster = temporaries->cell_to_cluster_map[this_ID];
      const int new_cluster = temporaries->cell_to_cluster_map[neigh_ID];

      const int old_counter = old_tag.counter();
      const int new_counter = prop_tag.counter();

      if (old_counter == new_counter && old_cluster != new_cluster)
        {
          prop_tag = old_tag.prepare_for_sharing(prop_tag);
          atomicMax(&(temporaries->reset_counters[counter_select]), new_counter);
          temporaries->continue_flag = 1;
        }
    }
  else if (neigh_tag.is_shared() && !neigh_tag.is_primary() && neigh_tag.counter() > 0x7FF)
    {
      prop_tag = prop_tag.update_counter(0x7FF);
      //Shared cells after the original ones
      //are not ordered by the propagation step
      //of the original shared cell.
      //Assuming less than 2^11 = 2048 propagation steps
      //before making a shared cell seems safe-ish?
    }

  if (old_tag < prop_tag && (!old_tag.is_part_of_splitter_cluster()          ||
                             prop_tag.counter() > old_tag.counter()          ||
                             prop_tag.is_primary()                           ||
                             (!prop_tag.is_shared() && old_tag.is_shared())     ))
    {
      atomicMax(&(temporaries->secondary_array[this_ID]), prop_tag);
      temporaries->continue_flag = 1;
    }
}

__device__ static
void update_cell_tag(const int cell,
                     Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                     Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                     const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                     const bool counter_select)
{
  TASTag new_tag = temporaries->secondary_array[cell];

  if (!new_tag.is_part_of_splitter_cluster())
    {
      return;
    }

  const int desired_counter = temporaries->reset_counters[counter_select];

  const TASTag old_tag = cell_state_arr->clusterTag[cell];

  const int cell_from = new_tag.index();
  const int new_cluster = temporaries->cell_to_cluster_map[cell_from];

  if (new_tag.counter() < desired_counter ||
      (old_tag.is_part_of_splitter_cluster() && old_tag.counter() < desired_counter) )
    {
      const int original_cluster = temporaries->original_cluster_map[new_cluster & 0xFFFFU];

      new_tag = TASTag::make_cluster_cell_tag(original_cluster);
      temporaries->cell_to_cluster_map[cell] = original_cluster;

      cell_state_arr->clusterTag[cell] = new_tag;
      temporaries->secondary_array[cell] = new_tag;
      return;
    }

  if (new_tag == old_tag)
    {
      return;
    }


  if ( new_tag.is_shared() &&  new_tag.is_primary() &&
       !old_tag.is_shared() && !old_tag.is_primary()    )
    {
      new_tag = new_tag.update_counter(old_tag.counter());
      const int old_cluster = temporaries->cell_to_cluster_map[cell];
      const unsigned int min_index = min(new_cluster, old_cluster);
      const unsigned int max_index = max(new_cluster, old_cluster);
      temporaries->cell_to_cluster_map[cell] = (max_index << 16) | min_index;
    }
  else
    {
      temporaries->cell_to_cluster_map[cell] = new_cluster;
    }

  const float cell_energy = cell_info_arr->energy[cell];
  new_tag = new_tag.update_cell(cell, __float_as_uint(cell_energy));

  cell_state_arr->clusterTag[cell] = new_tag;
  temporaries->secondary_array[cell] = new_tag;
}

__global__ static
void clusterSplittingMainCooperativeKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                           Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                           const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                           const bool share_cells)
{
  cooperative_groups::grid_group grid = cooperative_groups::this_grid();

  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  const int num_pairs = temporaries->pairs.number_normal;

  const int grid_size = grid.size();
  //Using the "legacy" version for the time being
  //due to CUDA toolkit version availability...

  bool counter_select = false;

  //int counter = 0;

  while (!temporaries->stop_flag)
    {
      for (int pair = index; pair < num_pairs; pair += grid_size)
        {
          propagate_main_pair(pair, temporaries, cell_state_arr, counter_select, share_cells);
        }

      grid.sync();

      for (int cell = index; cell < NCaloCells; cell += grid_size)
        {
          update_cell_tag(cell, cell_state_arr, temporaries, cell_info_arr, counter_select);
        }

      if (index == 0)
        {
          if (!temporaries->continue_flag)
            {
              temporaries->stop_flag = 1;
            }
          else
            {
              temporaries->reset_counters[!counter_select] = 0;
              temporaries->continue_flag = 0;
            }
        }

      //++counter;
      
      counter_select = !counter_select;

      grid.sync();

    }

  //if (index == 0)
  //  {
  //    printf("SPLITTING: %16d\n", counter);
  //  }

}

namespace
{
  struct kernel_sizes
  {
    int main_prop, tag_change;
  };
}


__global__ static
void handleSplitterTagChangesAndTerminationKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                                  Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                                  const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                                  const bool counter_select,
                                                  const bool share_cells
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const kernel_sizes blocks, const kernel_sizes grids
#endif
                                                 );

__global__ static
void propagateSplitterTagsKernel(const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                 Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                 const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                 const bool counter_select,
                                 const bool share_cells
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const kernel_sizes blocks, const kernel_sizes grids
#endif
                                )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  const int num_pairs = temporaries->pairs.number_normal;

  for (int pair = index; pair < num_pairs; pair += grid_size)
    {
      propagate_main_pair(pair, temporaries, cell_state_arr, counter_select, share_cells);
    }

#if CUDA_CAN_USE_TAIL_LAUNCH
  if (index == grid_size - 1)
    {

      handleSplitterTagChangesAndTerminationKernel <<< grids.tag_change, block.tag_change, 0, cudaStreamTailLaunch>>>(cell_state_arr,
                                                                                                                      temporaries,
                                                                                                                      cell_info_arr,
                                                                                                                      counter_select,
                                                                                                                      share_cells,
                                                                                                                      blocks, grids);
    }
#endif
}

__global__ static
void handleSplitterTagChangesAndTerminationKernel(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                                  Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                                  const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                                  const bool counter_select,
                                                  const bool share_cells
#if CUDA_CAN_USE_TAIL_LAUNCH
  , const kernel_sizes blocks, const kernel_sizes grids
#endif
                                                 )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      update_cell_tag(cell, cell_state_arr, temporaries, cell_info_arr, counter_select);
    }

  if (index == grid_size - 1)
    //Will be called with just 1 thread, but...
    {
      temporaries->reset_counters[!counter_select] = 0;
      if (temporaries->continue_flag)
        {
          temporaries->continue_flag = 0;
#if CUDA_CAN_USE_TAIL_LAUNCH

          propagateSplitterTagsKernel <<< grids.main_prop, blocks.main_prop, 0, cudaStreamTailLaunch>>>(cell_state_arr,
                                                                                                        temporaries,
                                                                                                        cell_info_arr,
                                                                                                        !counter_select,
                                                                                                        share_cells,
                                                                                                        blocks, grids);
#endif
        }
#if !CUDA_CAN_USE_TAIL_LAUNCH
      else /*if (!temporaries->continue_flag)*/
        {
          temporaries->stop_flag = 1;
        }
#endif
    }
}



__global__ static
void clusterSplittingMainDefer(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                               Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                               const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                               const bool share_cells,
                               const kernel_sizes blocks, kernel_sizes grids)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      if (grids.main_prop < 0)
        //We want the minimum grid size.
        {
          grids.main_prop =  Helpers::int_ceil_div(temporaries->pairs.number_normal, blocks.main_prop);
        }

#if CUDA_CAN_USE_TAIL_LAUNCH

      propagateSplitterTagsKernel <<< grids.main_prop, blocks.main_prop, 0, cudaStreamTailLaunch>>>(cell_state_arr,
                                                                                                    temporaries,
                                                                                                    cell_info_arr,
                                                                                                    0,
                                                                                                    share_cells,
                                                                                                    blocks, grids);
#else

      //int counter = 0;

      bool counter_select = false;

      while (!temporaries->stop_flag)
        {
          propagateSplitterTagsKernel <<< grids.main_prop, blocks.main_prop>>>(cell_state_arr,
                                                                               temporaries,
                                                                               cell_info_arr,
                                                                               counter_select,
                                                                               share_cells);

          handleSplitterTagChangesAndTerminationKernel <<< grids.tag_change, blocks.tag_change>>>(cell_state_arr,
                                                                                                  temporaries,
                                                                                                  cell_info_arr,
                                                                                                  counter_select,
                                                                                                  share_cells);

          counter_select = !counter_select;
          //++counter;
        }
      //printf("SPLITTING: %16d\n", counter);
#endif
    }
}


void TASplitting::splitClusterGrowing(EventDataHolder & holder,
                                      const ConstantDataHolder & instance_data,
                                      const TASOptionsHolder & options,
                                      const IGPUKernelSizeOptimizer & optimizer,
                                      const bool synchronize,
                                      CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  TopoAutomatonSplittingTemporaries * temps = TASHacks::get_temporaries(holder);

  hipMemsetAsync(&(temps->continue_flag),  0, sizeof(int),     stream_to_use);
  hipMemsetAsync(&(temps->reset_counters), 0, sizeof(int) * 2, stream_to_use);
  hipMemsetAsync(&(temps->stop_flag),      0, sizeof(int),     stream_to_use);

  if (optimizer.can_use_cooperative_groups())
    {
      const CUDAKernelLaunchConfiguration cfg_mainprop = optimizer.get_launch_configuration("TopoAutomatonSplitting", 7);

      void * main_args[] = { &holder.m_cell_state_dev, &temps,
                             &holder.m_cell_info_dev,
                             (void *) & (options.m_options->share_border_cells)
                           };

      hipLaunchCooperativeKernel((void *) clusterSplittingMainCooperativeKernel,
                                  cfg_mainprop.grid_x, cfg_mainprop.block_x,
                                  main_args, 0, stream_to_use);

    }
  else if (optimizer.can_use_dynamic_parallelism())
    {
      const CUDAKernelLaunchConfiguration cfg_mainprop    = optimizer.get_launch_configuration("TopoAutomatonSplitting", 8);
      const CUDAKernelLaunchConfiguration cfg_tagchange   = optimizer.get_launch_configuration("TopoAutomatonSplitting", 9);

      kernel_sizes blocks, grids;

      blocks.main_prop    = cfg_mainprop.block_x;
      blocks.tag_change   = cfg_tagchange.block_x;

      grids.main_prop    = optimizer.use_minimal_kernel_sizes() ? -1 : cfg_mainprop.grid_x;
      grids.tag_change   = cfg_tagchange.grid_x;

      clusterSplittingMainDefer <<< 1, 1, 0, stream_to_use>>>(holder.m_cell_state_dev,
                                                              temps,
                                                              holder.m_cell_info_dev,
                                                              options.m_options->share_border_cells,
                                                              blocks, grids);
    }
  else
    {
      throw std::runtime_error("The GPU must support either cooperative grid launches "
                               "or dynamic parallelism for the algorithm to work properly.");
      //Well, technically, we could code something up
      //(launch, say, 64 iterations, since the maximum
      // I have seen for ttbar is around ~40
      // and iterations are idempotent after the
      // stopping criterion has been reached),
      //but the added complexity and the fact that
      //architectures without dynamic parallelism
      //either are or soon will be deprecated
      //mean that it's not worth the effort.
    }

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}




/******************************************************************************************
 * Calculate the cell weights (only if indeed using shared_cells).
 ******************************************************************************************/

namespace
{
  constexpr int clusterprop_abs_E = 0;
  constexpr int clusterprop_E = 1;
  constexpr int clusterprop_x = 2;
  constexpr int clusterprop_y = 3;
  constexpr int clusterprop_z = 4;
}


__global__ static
void sumCellsForCentroidKernel( Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                const Helpers::CUDA_kernel_object<GeometryArr> geometry)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const TASTag tag = cell_state_arr->clusterTag[cell];
      if (tag.is_part_of_splitter_cluster() && !tag.is_shared())
        {
          const int cluster = temporaries->cell_to_cluster_map[cell];

          const float energy = cell_info_arr->energy[cell];
          const float abs_energy = fabsf(energy);
          const float x = geometry->x[cell];
          const float y = geometry->y[cell];
          const float z = geometry->z[cell];

          atomicAdd( &( temporaries->get_cluster_extra_array<clusterprop_abs_E, float>(cluster) ), abs_energy    );
          atomicAdd( &( temporaries->get_cluster_extra_array<clusterprop_E, float>(cluster) ), energy        );

          atomicAdd( &( temporaries->get_cluster_extra_array<clusterprop_x, float>(cluster) ), x * abs_energy );
          atomicAdd( &( temporaries->get_cluster_extra_array<clusterprop_y, float>(cluster) ), y * abs_energy );
          atomicAdd( &( temporaries->get_cluster_extra_array<clusterprop_z, float>(cluster) ), z * abs_energy );

        }
    }
}



__global__ static
void calculateCentroidsKernel(Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                              const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
{

  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;

  const int cluster_number = clusters_arr->number;

  for (int cluster = index; cluster < cluster_number; cluster += grid_size)
    {

      const float abs_energy = temporaries->get_cluster_extra_array<clusterprop_abs_E, float>(cluster);

      if (abs_energy > 0)
        {
          const float rev_abs_E = 1.0f / abs_energy;

          temporaries->get_cluster_extra_array<clusterprop_x, float>(cluster) *= rev_abs_E; // x

          temporaries->get_cluster_extra_array<clusterprop_y, float>(cluster) *= rev_abs_E; // y

          temporaries->get_cluster_extra_array<clusterprop_z, float>(cluster) *= rev_abs_E; // z
        }
    }
}

__global__ static
void calculateCentroidsKernelDeferKernel(Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                                         const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                         const int i_dimBlock)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    //Will be called with just 1 thread, but...
    {
      const int i_dimGrid = Helpers::int_ceil_div(clusters_arr->number, i_dimBlock);

#if CUDA_CAN_USE_TAIL_LAUNCH
      calculateCentroidsKernel <<< i_dimGrid, i_dimBlock, 0, cudaStreamTailLaunch>>>(temporaries, clusters_arr);
#else
      calculateCentroidsKernel <<< i_dimGrid, i_dimBlock>>>(temporaries, clusters_arr);
#endif
    }
}

__global__ static
void assignFinalCellsKernel( Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                             Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                             const Helpers::CUDA_kernel_object<TopoAutomatonSplittingTemporaries> temporaries,
                             const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                             const Helpers::CUDA_kernel_object<TopoAutomatonSplittingOptions> opts )
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;

  for (int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const TASTag tag = cell_state_arr->clusterTag[cell];
      if (tag.is_part_of_splitter_cluster())
        {
          if (opts->share_border_cells && tag.is_shared())
            {
              const uint32_t shared_clusters_packed = temporaries->cell_to_cluster_map[cell];
              const int cluster_1 = shared_clusters_packed & 0xFFFFU;
              const int cluster_2 = (shared_clusters_packed >> 16) & 0xFFFFU;

              const float cell_x = geometry->x[cell];
              const float cell_y = geometry->y[cell];
              const float cell_z = geometry->z[cell];


              const float delta_x_1 = cell_x - temporaries->get_cluster_extra_array<clusterprop_x, float>(cluster_1);
              const float delta_x_2 = cell_x - temporaries->get_cluster_extra_array<clusterprop_x, float>(cluster_2);

              const float delta_y_1 = cell_y - temporaries->get_cluster_extra_array<clusterprop_y, float>(cluster_1);
              const float delta_y_2 = cell_y - temporaries->get_cluster_extra_array<clusterprop_y, float>(cluster_2);

              const float delta_z_1 = cell_z - temporaries->get_cluster_extra_array<clusterprop_z, float>(cluster_1);
              const float delta_z_2 = cell_z - temporaries->get_cluster_extra_array<clusterprop_z, float>(cluster_2);


              const float d_1 = sqrtf(delta_x_1 * delta_x_1 + delta_y_1 * delta_y_1 + delta_z_1 * delta_z_1);

              const float d_2 = sqrtf(delta_x_2 * delta_x_2 + delta_y_2 * delta_y_2 + delta_z_2 * delta_z_2);

              float r_exp = (d_1 - d_2) / opts->EM_shower_scale;

              if (r_exp > 10)
                {
                  r_exp = 10;
                }
              else if (r_exp < -10)
                {
                  r_exp = -10;
                }

              const float r = expf(r_exp);
              const float r_reverse = expf(-r_exp);

              float E_1 = temporaries->get_cluster_extra_array<clusterprop_E, float>(cluster_1);

              float E_2 = temporaries->get_cluster_extra_array<clusterprop_E, float>(cluster_2);

              if (opts->use_absolute_energy)
                {
                  E_1 = fabsf(E_1);
                  E_2 = fabsf(E_2);
                }

              if (E_1 <= 0)
                {
                  E_1 = 1.0f * CLHEP::MeV;
                }
              if (E_2 <= 0)
                {
                  E_2 = 1.0f * CLHEP::MeV;
                }

              float weight = E_1 / (E_1 + E_2 * r);
              float rev_weight = E_2 / (E_2 + E_1 * r_reverse);

              //Optimization opportunity:
              //I think w_1 > w_2 is satisfied by 0 < r < E1/E2,
              //so we could save some of the computation
              //at the cost of slightly complicating the logic
              //(since we need to deal with the reverse weight
              // and ensure we always use the most accurate value).

              if (__float_as_uint(weight) == 0)
                {
                  weight == __uint_as_float(1);
                }

              if (__float_as_uint(rev_weight) == 0)
                {
                  rev_weight == __uint_as_float(1);
                }

              //This is just so that shared clusters
              //always show up as shared clusters.
              //A denormal weight is... negligible for physics.

              if (weight > 0.5f)
                {
                  cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(cluster_1, __float_as_uint(rev_weight), cluster_2);
                }
              else if (weight == 0.5f)
                {
                  const int max_cluster = cluster_1 > cluster_2 ? cluster_1 : cluster_2;
                  const int min_cluster = cluster_1 > cluster_2 ? cluster_2 : cluster_1;
                  cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(max_cluster, __float_as_uint(weight), min_cluster);
                }
              else /*if (weight < 0.5f)*/
                {
                  cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(cluster_2, __float_as_uint(weight), cluster_1);
                }
            }
          else
            {
              const int this_cluster = temporaries->cell_to_cluster_map[cell];
              cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(this_cluster);
            }
        }
      else if (tag.is_non_assigned_part_of_split_cluster())
        {
          const int this_cluster = tag.index();
          cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(this_cluster);
          //Cells that are part of a pre-splitter cluster get added to the "same"?

          atomicMax(&(clusters_arr->seedCellID[this_cluster]), cell);
          //Not the seed cell, but just a consistent way of marking this cluster as still valid...
        }
      else if (tag.is_part_of_original_cluster())
        {
          const int this_cluster = tag.index();
          cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(this_cluster);
        }
      else
        {
          cell_state_arr->clusterTag[cell] = ClusterTag::make_invalid_tag();
        }
    }
}


void TASplitting::cellWeightingAndFinalization(EventDataHolder & holder,
                                               const ConstantDataHolder & instance_data,
                                               const TASOptionsHolder & options,
                                               const IGPUKernelSizeOptimizer & optimizer,
                                               const bool synchronize,
                                               CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  TopoAutomatonSplittingTemporaries * temps = TASHacks::get_temporaries(holder);

  if (options.m_options->share_border_cells)
    {
      hipMemsetAsync(temps->secondary_array, 0, sizeof(tag_type) * NCaloCells, hipStreamPerThread);

      const CUDAKernelLaunchConfiguration cfg_sumcells = optimizer.get_launch_configuration("TopoAutomatonSplitting", 10);
      const CUDAKernelLaunchConfiguration cfg_centroid = optimizer.get_launch_configuration("TopoAutomatonSplitting", 11);

      sumCellsForCentroidKernel <<< cfg_sumcells.grid_x, cfg_sumcells.block_x, 0, stream_to_use>>>(temps,
                                                                                                   holder.m_cell_state_dev,
                                                                                                   holder.m_cell_info_dev,
                                                                                                   instance_data.m_geometry_dev);
      if (optimizer.use_minimal_kernel_sizes())
        {
          calculateCentroidsKernelDeferKernel <<< 1, 1, 0, stream_to_use>>>(temps, holder.m_clusters_dev, cfg_centroid.block_x);
        }
      else
        {
          calculateCentroidsKernel <<< cfg_centroid.grid_x, cfg_centroid.block_x, 0, stream_to_use>>>(temps, holder.m_clusters_dev);
        }

    }

  const CUDAKernelLaunchConfiguration cfg_finalize = optimizer.get_launch_configuration("TopoAutomatonSplitting", 12);

  assignFinalCellsKernel <<< cfg_finalize.grid_x, cfg_finalize.block_x, 0, stream_to_use>>>(holder.m_cell_state_dev,
                                                                                            holder.m_clusters_dev,
                                                                                            temps,
                                                                                            instance_data.m_geometry_dev,
                                                                                            options.m_options_dev);

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}

/*******************************************************************************************************************************/

void TASplitting::register_kernels(IGPUKernelSizeOptimizer & optimizer)
{

  void * kernels[] = { (void *) fillNeighboursKernel,
                       (void *) countInferiorNeighsKernel,
                       (void *) findLocalMaximaKernel,
                       (void *) fixClustersWithoutMaximaKernel,
                       (void *) secondaryMaximaCooperativeKernel,
                       (void *) propagateForMaximaExclusionKernel,
                       (void *) cleanUpSecondariesKernel,
                       (void *) clusterSplittingMainCooperativeKernel,
                       (void *) propagateSplitterTagsKernel,
                       (void *) handleSplitterTagChangesAndTerminationKernel,
                       (void *) sumCellsForCentroidKernel,
                       (void *) calculateCentroidsKernel,
                       (void *) assignFinalCellsKernel
                     };

  int blocksizes[] = { FillNeighboursBlockSize,
                       CountInferiorNeighsBlockSize,
                       FindLocalMaximaBlockSize,
                       FixClustersWithoutMaximaBlockSize,
                       ExcludeMaximaPropagationBlockSize,
                       ExcludeMaximaOnePropagationBlockSize,
                       CleanUpSecondariesBlockSize,
                       ClusterSplittingMainPropagationBlockSize,
                       PropagateSplitTagsBlockSize,
                       HandleSplitTagChangesBlockSize,
                       SumCellsBlockSize,
                       CalculateCentroidsBlockSize,
                       FinalCellsBlockSize
                     };

  int  gridsizes[] = { Helpers::int_ceil_div(NCaloCells, FillNeighboursBlockSize),
                       Helpers::int_ceil_div(NMaxPairs + NMaxPairs / 2, CountInferiorNeighsBlockSize),
                       Helpers::int_ceil_div(NCaloCells, FindLocalMaximaBlockSize),
                       Helpers::int_ceil_div(NCaloCells, FixClustersWithoutMaximaBlockSize),
                       IGPUKernelSizeOptimizer::SpecialSizeHints::CooperativeLaunch,
                       Helpers::int_ceil_div(NMaxPairs, ExcludeMaximaOnePropagationBlockSize),
                       Helpers::int_ceil_div(NCaloCells, CleanUpSecondariesBlockSize),
                       IGPUKernelSizeOptimizer::SpecialSizeHints::CooperativeLaunch,
                       Helpers::int_ceil_div(NMaxPairs, PropagateSplitTagsBlockSize),
                       Helpers::int_ceil_div(NCaloCells, HandleSplitTagChangesBlockSize),
                       Helpers::int_ceil_div(NCaloCells, SumCellsBlockSize),
                       Helpers::int_ceil_div(NMaxClusters, CalculateCentroidsBlockSize),
                       Helpers::int_ceil_div(NCaloCells, FinalCellsBlockSize)
                     };

  int   maxsizes[] = { NCaloCells,
                       NMaxPairs + NMaxPairs / 2,
                       NCaloCells,
                       NCaloCells,
                       NMaxPairs,
                       NMaxPairs,
                       NCaloCells,
                       std::max(NMaxPairs, NCaloCells),
                       NMaxPairs,
                       NCaloCells,
                       NCaloCells,
                       NMaxClusters,
                       NCaloCells
                     };

  optimizer.register_kernels("TopoAutomatonSplitting", 13, kernels, blocksizes, gridsizes, maxsizes);

}