//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//
// Dear emacs, this is -*- c++ -*-
//

#include "CaloRecGPU/DataHolders.h"

#include "MacroHelpers.h"

void CaloRecGPU::ConstantDataHolder::sendToGPU(const bool clear_CPU)
{
  m_cell_noise_dev = m_cell_noise;
  m_geometry_dev = m_geometry;
  if (clear_CPU)
    {
      m_cell_noise.clear();
      m_geometry.clear();
    }
}

void CaloRecGPU::EventDataHolder::sendToGPU(const bool clear_CPU,
                                            const bool has_state,
                                            const bool has_clusters,
                                            const bool has_moments)
{
  m_cell_info_dev = m_cell_info;
  if (has_state)
    {
      m_cell_state_dev = m_cell_state;
    }
  else
    {
      m_cell_state_dev.allocate();
    }
  if (has_clusters)
    {
      m_clusters_dev = m_clusters;
    }
  else
    {
      m_clusters_dev.allocate();
    }
  if (has_moments)
    {
      m_moments_dev = m_moments;
    }
  else
    {
      m_moments_dev.allocate();
    }

  if (!has_clusters)
    {
      CUDA_ERRCHECK(hipMemset(&(m_clusters_dev->number), 0, sizeof(m_clusters_dev->number)));
    }
  //We're not doing this through hipMemsetAsync because it is reasonable to expect
  //the clusters to be fully sent before doing any more operations.

  if (clear_CPU)
    {
      m_cell_info.clear();
      m_cell_state.clear();
      m_moments.clear();
    }
}

void CaloRecGPU::EventDataHolder::returnToCPU(const bool clear_GPU,
                                              const bool return_cells,
                                              const bool return_clusters,
                                              const bool return_moments)
{
  if (return_cells)
    {
      m_cell_state = m_cell_state_dev;
    }
  if (return_clusters)
    {
      m_clusters = m_clusters_dev;
    }
  if (return_moments)
    {
      m_moments = m_moments_dev;
    }
  if (clear_GPU)
    {
      m_cell_state_dev.clear();
      m_clusters_dev.clear();
      m_cell_info_dev.clear();
      m_moments_dev.clear();
    }
}



void CaloRecGPU::EventDataHolder::returnCellsToCPU(CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  CUDA_ERRCHECK( hipMemcpyAsync((CaloRecGPU::CellStateArr *) m_cell_state,
                                 (CaloRecGPU::CellStateArr *) m_cell_state_dev,
                                 sizeof(CaloRecGPU::CellStateArr),
                                 hipMemcpyDeviceToHost, stream_to_use) );
}

void CaloRecGPU::EventDataHolder::returnClustersToCPU(CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  CUDA_ERRCHECK( hipMemcpyAsync((CaloRecGPU::ClusterInfoArr *) m_clusters,
                                 (CaloRecGPU::ClusterInfoArr *) m_clusters_dev,
                                 sizeof(CaloRecGPU::ClusterInfoArr),
                                 hipMemcpyDeviceToHost, stream_to_use) );
}

void CaloRecGPU::EventDataHolder::returnMomentsToCPU(CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  CUDA_ERRCHECK(hipMemcpyAsync((CaloRecGPU::ClusterMomentsArr *) m_moments,
                                (CaloRecGPU::ClusterMomentsArr *) m_moments_dev,
                                sizeof(CaloRecGPU::ClusterMomentsArr),
                                hipMemcpyDeviceToHost, stream_to_use) );
}

void CaloRecGPU::EventDataHolder::returnClusterNumberToCPU(CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  CUDA_ERRCHECK(hipMemcpyAsync(&(m_clusters->number),
                                &(m_clusters_dev->number),
                                sizeof(int),
                                hipMemcpyDeviceToHost, stream_to_use) );
}


#define CALORECGPU_ASYNC_TRANSFER_HELPER(OBJ, MEMBER, TYPE, NUM, STREAM)            \
  CUDA_ERRCHECK( hipMemcpyAsync ( &( OBJ -> MEMBER [0]),                           \
                                   &( OBJ ## _dev -> MEMBER [0]),                   \
                                   sizeof(TYPE) * (NUM),                            \
                                   hipMemcpyDeviceToHost,                          \
                                   STREAM                           ) );

#define CALORECGPU_ASYNC_TRANSFER_PER_SAMPLE_HELPER(OBJ, MEMBER, TYPE, NUM, STREAM) \
  for (int sampling = 0; sampling < NumSamplings; ++sampling)                       \
    {                                                                               \
      CUDA_ERRCHECK( hipMemcpyAsync ( &( OBJ -> MEMBER [sampling][0]),             \
                                       &( OBJ ## _dev -> MEMBER [sampling][0]),     \
                                       sizeof(TYPE) * (NUM),                        \
                                       hipMemcpyDeviceToHost,                      \
                                       STREAM                       ) );            \
    }

//I almost felt tempted to do some
//variadic structured binding magic
//and compile time stuff
//to allow us to convert on-the-fly
//like this per-struct-member
//directly from the helpers.
//Emphasis on almost.
//Let's just hope for C++26...

void CaloRecGPU::EventDataHolder::returnSomeClustersToCPU(const size_t num_clusters, CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  //We assume the cluster number we take is the known number of clusters,
  //so we skip copying that.

  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_clusters, clusterEnergy,      float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_clusters, clusterEt,          float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_clusters, clusterEta,         float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_clusters, clusterPhi,         float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_clusters, seedCellID,         int,   num_clusters, stream_to_use);
}

void CaloRecGPU::EventDataHolder::returnSomeMomentsToCPU(const size_t num_clusters, CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  CALORECGPU_ASYNC_TRANSFER_PER_SAMPLE_HELPER(m_moments, energyPerSample,     float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_PER_SAMPLE_HELPER(m_moments, maxEPerSample,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_PER_SAMPLE_HELPER(m_moments, maxPhiPerSample,     float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_PER_SAMPLE_HELPER(m_moments, maxEtaPerSample,     float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_PER_SAMPLE_HELPER(m_moments, etaPerSample,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_PER_SAMPLE_HELPER(m_moments, phiPerSample,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, time,                float, num_clusters, stream_to_use);

  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, firstPhi,            float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, firstEta,            float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, secondR,             float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, secondLambda,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, deltaPhi,            float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, deltaTheta,          float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, deltaAlpha,          float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, centerX,             float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, centerY,             float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, centerZ,             float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, centerMag,           float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, centerLambda,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, lateral,             float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, longitudinal,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engFracEM,           float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engFracMax,          float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engFracCore,         float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, firstEngDens,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, secondEngDens,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, isolation,           float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engBadCells,         float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, nBadCells,           int,   num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, nBadCellsCorr,       int,   num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, badCellsCorrE,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, badLArQFrac,         float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engPos,              float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, significance,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, cellSignificance,    float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, cellSigSampling,     int,   num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, avgLArQ,             float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, avgTileQ,            float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engBadHVCells,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, nBadHVCells,         float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, PTD,                 float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, mass,                float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, EMProbability,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, hadWeight,           float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, OOCweight,           float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, DMweight,            float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, tileConfidenceLevel, float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, secondTime,          float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, nBadHVCells,         float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_PER_SAMPLE_HELPER(m_moments, nCellSampling,       int,   num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, nExtraCellSampling,  int,   num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, numCells,            int,   num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, vertexFraction,      float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, nVertexFraction,     float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, etaCaloFrame,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, etaCaloFrame,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, phiCaloFrame,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, eta1CaloFrame,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, phi1CaloFrame,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, eta2CaloFrame,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, phi2CaloFrame,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibTot,         float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibOutL,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibOutM,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibOutT,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadL,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadM,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadT,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadT,       float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibEMB0,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibEME0,        float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibTileG3,      float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadTot,     float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadEMB0,    float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadTile0,   float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadTileG3,  float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadEME0,    float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadHEC0,    float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadFCAL,    float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadLeakage, float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibDeadUnclass, float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibFracEM,      float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibFracHad,     float, num_clusters, stream_to_use);
  CALORECGPU_ASYNC_TRANSFER_HELPER(           m_moments, engCalibFracRest,    float, num_clusters, stream_to_use);
}

void CaloRecGPU::EventDataHolder::allocate(const bool also_GPU)
{
  m_cell_info.allocate();
  m_cell_state.allocate();
  m_clusters.allocate();
  m_moments.allocate();

  if (also_GPU)
    {
      m_cell_info_dev.allocate();
      m_cell_state_dev.allocate();
      m_clusters_dev.allocate();
      m_moments_dev.allocate();
    }
}

void CaloRecGPU::EventDataHolder::clear_GPU()
{
  m_cell_info_dev.clear();
  m_cell_state_dev.clear();
  m_clusters_dev.clear();
  m_moments_dev.clear();
}