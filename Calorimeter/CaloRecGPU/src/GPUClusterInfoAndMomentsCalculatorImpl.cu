#include "hip/hip_runtime.h"
//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//
// Dear emacs, this is -*- c++ -*-
//

//NOTE: at several points of this implementation file,
//      some commented out appears here and there
//      to invalidate some clusters (seedCellID[clusters] = -1)
//      and clean them up. This is useful for debugging
//      the moments, by excluding clusters that may take
//      different choices than the CPU when there is a cutoff.

#include "CaloRecGPU/Helpers.h"
#include "CaloRecGPU/CUDAFriendlyClasses.h"
#include "GPUClusterInfoAndMomentsCalculatorImpl.h"
#include "FPHelpers.h"


#include "CaloGeoHelpers/CaloSampling.h"
//Just enums and stuff, CUDA compatible.

#include <cmath>




using namespace CaloRecGPU;
using namespace ClusterMomentsCalculator;

void ClusterMomentsCalculator::CMCOptionsHolder::allocate()
{
  m_options.allocate();
  m_options_dev.allocate();
}

void ClusterMomentsCalculator::CMCOptionsHolder::sendToGPU(const bool clear_CPU)
{
  m_options_dev = m_options;
  if (clear_CPU)
    {
      m_options.clear();
    }
}

constexpr static int ClusterPassBlockSize = 1024;
constexpr static int CellPassBlockSize = 1024;
//Maximize throughput?
//Needs measurements, perhaps...
//Also we could split all the sub-kernels
//to have different block sizes.

namespace
{

  namespace CMCHack
  {
    //A generic TempSpecifier has typedef called "type" to specify the type of the return array
    //(e. g. should be float for most things), a static constexpr unsigned int number
    //to count the number of the array (or, more accurately, the number of other 4-byte
    //arrays before this one), and a static constexpr bool reverse to mark temporaries
    //that we being storing from the end of the moments array...
    //In practice, for our use cases, we will specialize this
    //to be sure there is no overlap...
    template <class TempSpecifier>
    __host__ __device__ typename TempSpecifier::type * get_temporary_array(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      char * c_ptr = (char *) ((ClusterMomentsArr *) moments_arr);
      if (TempSpecifier::reverse)
        {
          return (typename TempSpecifier::type * ) ( c_ptr + (NMaxClusters * TempSpecifier::number * sizeof(float)) );
          //I know. sizeof(char) == 1. Clearer this way, still...
        }
      else
        {
          return (typename TempSpecifier::type * ) ( c_ptr + sizeof(ClusterMomentsArr) - (NMaxClusters * (TempSpecifier::number + 1) * sizeof(float)) );
          //I know. sizeof(char) == 1. Clearer this way, still...
        }
    }
  }

  struct SeedCellPhi
  {
    using type = float;
  };

  struct EnergyDensityNormalization
  {
    using type = float;
  };

  struct SumAbsEnergyNonMoments
  {
    using type = float;
  };

  struct MaxCellEnergyAndCell
  {
    using type = unsigned long long int;
  };

  struct SecondMaxCellEnergyAndCell
  {
    using type = unsigned long long int;
  };

  //For later:

  struct MaxCells
  {
    using type = int;
  };

  struct SecondMaxCells
  {
    using type = int;
  };

  //-----------------

  struct NumberEmptySamplings
  {
    using type = int[NMaxClusters];
  };

  struct NumberNonEmptySamplings
  {
    using type = int[NMaxClusters];
  };

  struct MX
  {
    using type = float;
  };
  struct MY
  {
    using type = float;
  };
  struct MZ
  {
    using type = float;
  };

  struct MaxMomentsEnergyPerSample
  {
    using type = unsigned int[NMaxClusters];
    //Stored as unsigned ints because there's no atomicMax for floats in CUDA...
  };

  struct MaxSignificanceAndSampling
  {
    using type = unsigned long long int;
  };

  struct Matrix00
  {
    using type = float;
  };
  struct Matrix10
  {
    using type = float;
  };
  struct Matrix20
  {
    using type = float;
  };
  struct Matrix11
  {
    using type = float;
  };
  struct Matrix21
  {
    using type = float;
  };
  struct Matrix22
  {
    using type = float;
  };
  struct SumSquareEnergies
  {
    using type = float;
  };
  struct TimeNormalization
  {
    using type = float;
  };
  struct AverageLArQNorm
  {
    using type = float;
  };
  struct AverageTileQNorm
  {
    using type = float;
  };

  struct ShowerAxisX
  {
    using type = float;
  };

  struct ShowerAxisY
  {
    using type = float;
  };

  struct ShowerAxisZ
  {
    using type = float;
  };

  struct NumPositiveEnergyCells
  {
    using type = int;
  };

  struct AbsoluteEnergyPerSample
  {
    using type = float[NMaxClusters];
  };

  struct LateralNormalization
  {
    using type = float;
  };
  struct LongitudinalNormalization
  {
    using type = float;
  };
  struct MaxEnergyAndCellPerSample
  {
    using type = unsigned long long int[NMaxClusters];
  };

  struct MaxECellPerSample
  {
    using type = int[NMaxClusters];
  };


  namespace CMCHack
//Special cases, to prevent overlap.
  {
    template <>
    __host__ __device__ typename SeedCellPhi::type * get_temporary_array<SeedCellPhi>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (SeedCellPhi::type *) moments_arr->engCalibFracRest;
    }

    template <>
    __host__ __device__ typename EnergyDensityNormalization::type * get_temporary_array<EnergyDensityNormalization>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (EnergyDensityNormalization::type *) moments_arr->etaCaloFrame;
    }

    template <>
    __host__ __device__ typename SumAbsEnergyNonMoments::type * get_temporary_array<SumAbsEnergyNonMoments>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (SumAbsEnergyNonMoments::type *) moments_arr->engCalibFracHad;
    }

    template <>
    __host__ __device__ typename MaxCellEnergyAndCell::type * get_temporary_array<MaxCellEnergyAndCell>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (MaxCellEnergyAndCell::type *) moments_arr->engCalibDeadUnclass;
    }

    template <>
    __host__ __device__ typename SecondMaxCellEnergyAndCell::type * get_temporary_array<SecondMaxCellEnergyAndCell>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (SecondMaxCellEnergyAndCell::type *) moments_arr->engCalibDeadFCAL;
    }

    template <>
    __host__ __device__ typename MaxCells::type * get_temporary_array<MaxCells>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (MaxCells::type *) moments_arr->engBadHVCells;
    }
    template <>
    __host__ __device__ typename SecondMaxCells::type * get_temporary_array<SecondMaxCells>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (SecondMaxCells::type *) moments_arr->nBadHVCells;
    }

    template <>
    __host__ __device__ typename NumberEmptySamplings::type * get_temporary_array<NumberEmptySamplings>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (NumberEmptySamplings::type *) moments_arr->maxEPerSample;
    }

    template <>
    __host__ __device__ typename NumberNonEmptySamplings::type * get_temporary_array<NumberNonEmptySamplings>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (NumberNonEmptySamplings::type *) moments_arr->maxPhiPerSample;
    }

    template <>
    __host__ __device__ typename MaxMomentsEnergyPerSample::type * get_temporary_array<MaxMomentsEnergyPerSample>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (MaxMomentsEnergyPerSample::type *) moments_arr->maxEtaPerSample;
    }

    template <>
    __host__ __device__ typename MaxSignificanceAndSampling::type * get_temporary_array<MaxSignificanceAndSampling>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (MaxSignificanceAndSampling::type *) moments_arr->eta1CaloFrame;
    }


    template <>
    __host__ __device__ typename MX::type * get_temporary_array<MX>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (MX::type *) moments_arr->engCalibOutL;
    }

    template <>
    __host__ __device__ typename MY::type * get_temporary_array<MY>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (MY::type *) moments_arr->engCalibOutM;
    }

    template <>
    __host__ __device__ typename MZ::type * get_temporary_array<MZ>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (MZ::type *) moments_arr->engCalibOutT;
    }

    template <>
    __host__ __device__ typename Matrix00::type * get_temporary_array<Matrix00>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (Matrix00::type *) moments_arr->engCalibDeadT;
    }
    template <>
    __host__ __device__ typename Matrix10::type * get_temporary_array<Matrix10>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (Matrix10::type *) moments_arr->engCalibEMB0;
    }
    template <>
    __host__ __device__ typename Matrix20::type * get_temporary_array<Matrix20>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (Matrix20::type *) moments_arr->engCalibEME0;
    }
    template <>
    __host__ __device__ typename Matrix11::type * get_temporary_array<Matrix11>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (Matrix11::type *) moments_arr->engCalibTileG3;
    }
    template <>
    __host__ __device__ typename Matrix21::type * get_temporary_array<Matrix21>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (Matrix21::type *) moments_arr->engCalibDeadTot;
    }
    template <>
    __host__ __device__ typename Matrix22::type * get_temporary_array<Matrix22>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (Matrix22::type *) moments_arr->engCalibDeadEMB0;
    }
    template <>
    __host__ __device__ typename SumSquareEnergies::type * get_temporary_array<SumSquareEnergies>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (SumSquareEnergies::type *) moments_arr->engCalibDeadTile0;
    }
    template <>
    __host__ __device__ typename TimeNormalization::type * get_temporary_array<TimeNormalization>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (TimeNormalization::type *) moments_arr->engCalibDeadTileG3;
    }
    template <>
    __host__ __device__ typename AverageLArQNorm::type * get_temporary_array<AverageLArQNorm>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (AverageLArQNorm::type *) moments_arr->engCalibDeadEME0;
    }
    template <>
    __host__ __device__ typename AverageTileQNorm::type * get_temporary_array<AverageTileQNorm>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (AverageTileQNorm::type *) moments_arr->engCalibDeadHEC0;
    }


    template <>
    __host__ __device__ ShowerAxisX::type * get_temporary_array<ShowerAxisX>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (ShowerAxisX::type *) moments_arr->EMProbability;
    }
    template <>
    __host__ __device__ ShowerAxisY::type * get_temporary_array<ShowerAxisY>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (ShowerAxisY::type *) moments_arr->hadWeight;
    }
    template <>
    __host__ __device__ ShowerAxisZ::type * get_temporary_array<ShowerAxisZ>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (ShowerAxisZ::type *) moments_arr->OOCweight;
    }

    template <>
    __host__ __device__ typename NumPositiveEnergyCells::type * get_temporary_array<NumPositiveEnergyCells>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (NumPositiveEnergyCells::type *) moments_arr->vertexFraction;
    }

    template <>
    __host__ __device__ typename AbsoluteEnergyPerSample::type * get_temporary_array<AbsoluteEnergyPerSample>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (AbsoluteEnergyPerSample::type *) moments_arr->vertexFraction;
    }


    template <>
    __host__ __device__ LateralNormalization::type * get_temporary_array<LateralNormalization>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return moments_arr->DMweight;
    }
    template <>
    __host__ __device__ LongitudinalNormalization::type * get_temporary_array<LongitudinalNormalization>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return moments_arr->tileConfidenceLevel;
    }
    template <>
    __host__ __device__ MaxEnergyAndCellPerSample::type * get_temporary_array<MaxEnergyAndCellPerSample>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (MaxEnergyAndCellPerSample::type *) moments_arr->maxPhiPerSample;
    }

    template <>
    __host__ __device__ typename MaxECellPerSample::type * get_temporary_array<MaxECellPerSample>(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr)
    {
      return (MaxECellPerSample::type *) moments_arr->vertexFraction;
    }
  }

}

//The per-sample things might very well benefit from being
//struct-of-arrays instead of pure array-of-structs
//since we do the samplings in parallel.
//However, it'd be less clean when it came
//to the temporary array thing...

constexpr int WarpSize = 32;
//Let's do this per warp...
//In sufficiently new hardware,
//independent thread scheduling
//might very well ensure all the threads
//that are updating the same moments
//get executed together,
//while the warp-wide broadcasting
//will possibly improve memory accesses
//(and, in general, doing the moments
// in parallel will be a net performance gain
// when compared to have all of them being updated in a single thread...))

static_assert(NumSamplings <= 28, "We wrote the code under the assumption of 28 samplings at most.");

/******************************************************************************
 * Clear invalid cells first. (The algorithm doesn't invalidate clusters.)    *
 ******************************************************************************/
/*
__global__ static
void clearInvalidCells(Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                       const Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  for (const int cell = index; cell < NCaloCells; cell += grid_size)
    {
      const ClusterTag tag = cell_state_arr->clusterTag[cell];
      if (tag.is_part_of_cluster())
        {
          if (tag.is_shared_between_clusters())
            {
              const int first_cluster = tag.cluster_index();
              const int second_cluster = tag.secondary_cluster_index();

              const int first_seed = clusters_arr->seedCellID[first_cluster];
              const int second_seed = clusters_arr->seedCellID[second_cluster];

              if (first_seed < 0 && second_seed < 0)
                {
                  cell_state_arr->clusterTag[cell] = ClusterTag:: make_invalid_tag();
                }
              else if (first_seed < 0)
                {
                  cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(second_cluster);
                }
              else if (second_seed < 0)
                {
                  cell_state_arr->clusterTag[cell] = ClusterTag::make_tag(first_cluster);
                }
              else / * if (first_seed >= 0 && second_seed >= 0) * /
                {
                  //Do nothing: the tag's already OK.
                }
            }
          else
            {
              if (clusters_arr->seedCellID[tag.cluster_index()] < 0)
                {
                  cell_state_arr->clusterTag[cell] = ClusterTag:: make_invalid_tag();
                }
            }
        }
    }
}
*/

/******************************************************************************
 * First Pass                                                                 *
 ******************************************************************************/

__global__ static
void zerothClusterPassKernel(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr,
                             Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                             const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                             const bool skip_invalid)
{
  const int cluster_number = clusters_arr->number;

  const int index   = blockIdx.x * blockDim.x + threadIdx.x;
  const int moment  = threadIdx.x % WarpSize;
  const int grid_size = gridDim.x * blockDim.x;

  for (int cluster = index / WarpSize; cluster < cluster_number; cluster += grid_size / WarpSize)
    {
      if (skip_invalid && clusters_arr->seedCellID[cluster] < 0)
        {
          continue;
        }

      if (moment < NumSamplings)
        {
          const int sampling = moment;
          moments_arr->energyPerSample[sampling][cluster] = 0.f;
          moments_arr->nCellSampling[sampling][cluster] = 0;
          CMCHack::get_temporary_array< NumberEmptySamplings      >(moments_arr)[sampling][cluster] = 0;
          CMCHack::get_temporary_array< NumberNonEmptySamplings   >(moments_arr)[sampling][cluster] = 0;
          CMCHack::get_temporary_array< MaxMomentsEnergyPerSample >(moments_arr)[sampling][cluster] = 0;
        }
      else
        {
          switch (moment - NumSamplings)
            //NumSamplings == 28
            {
              case 0:
                moments_arr->centerX[cluster] = 0.f;
                moments_arr->centerY[cluster] = 0.f;
                moments_arr->centerZ[cluster] = 0.f;
                moments_arr->firstEngDens[cluster] = 0.f;
                moments_arr->secondEngDens[cluster] = 0.f;
                break;
              case 1:
                moments_arr->engFracEM[cluster] = 0.f;
                moments_arr->engPos[cluster] = 0.f;
                clusters_arr->clusterEnergy[cluster] = 0.f;
                CMCHack::get_temporary_array<SumAbsEnergyNonMoments>(moments_arr)[cluster] = 0.f;
                CMCHack::get_temporary_array<EnergyDensityNormalization>(moments_arr)[cluster] = 0.f;
                break;
              case 2:
                CMCHack::get_temporary_array<MX>(moments_arr)[cluster] = 0.f;
                CMCHack::get_temporary_array<MY>(moments_arr)[cluster] = 0.f;
                CMCHack::get_temporary_array<MaxCellEnergyAndCell>(moments_arr)[cluster] = 0ULL;
                CMCHack::get_temporary_array<SecondMaxCellEnergyAndCell>(moments_arr)[cluster] = 0ULL;
                clusters_arr->clusterEta[cluster] = 0.f;
                break;
              case 3:
                CMCHack::get_temporary_array<MZ>(moments_arr)[cluster] = 0.f;
                clusters_arr->clusterPhi[cluster] = 0.f;
                {
                  const int seed_cell = clusters_arr->seedCellID[cluster];
                  if (seed_cell >= 0 && seed_cell < NCaloCells)
                    {
                      CMCHack::get_temporary_array<SeedCellPhi>(moments_arr)[cluster] = geometry->phi[seed_cell];
                    }
                }
                break;
              default:
                break;
            }
        }
    }
}


__global__ static
void firstCellPassKernel(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr,
                         Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                         const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                         const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                         const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                         const bool use_abs_energy)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  const int in_warp_index = threadIdx.x % WarpSize;

  for (int cell = index / WarpSize; cell < NCaloCells; cell += grid_size / WarpSize)
    {
      const ClusterTag tag        = cell_state_arr->clusterTag[cell];
      const int        sampling   = geometry->sampling(cell);

      if (tag.is_part_of_cluster())
        {
          const float energy         = cell_info_arr->energy[cell];
          const float abs_energy     = fabsf(energy);
          const float moments_energy = ((use_abs_energy || energy > 0.f) ? abs_energy : 0.f);
          const float x              = geometry->x[cell];
          const float y              = geometry->y[cell];
          const float z              = geometry->z[cell];
          const float eta            = geometry->eta[cell];
          const float phi            = geometry->phi[cell];
          const float volume         = geometry->volume[cell];

          auto accumulateForCluster = [&](const int cluster, const float weight, const int this_moment)
          {
            const float weighted_energy = moments_energy * weight;
            switch (this_moment)
              {
                case 0:
                  atomicAdd(&(moments_arr->energyPerSample[sampling][cluster]), energy * weight);
                  break;
                case 1:
                  atomicAdd(&(moments_arr->nCellSampling[sampling][cluster]), 1);
                  break;
                case 2:
                  atomicMax(&(CMCHack::get_temporary_array<MaxMomentsEnergyPerSample>(moments_arr)[sampling][cluster]), __float_as_uint(weighted_energy));
                  break;
                case 3:
                  atomicAdd(&(moments_arr->centerX[cluster]), x * weighted_energy);
                  break;
                case 4:
                  atomicAdd(&(moments_arr->centerY[cluster]), y * weighted_energy);
                  break;
                case 5:
                  atomicAdd(&(moments_arr->centerZ[cluster]), z * weighted_energy);
                  break;
                case 6:
                  if ( sampling == CaloSampling::EMB1   ||
                       sampling == CaloSampling::EMB2   ||
                       sampling == CaloSampling::EMB3   ||
                       sampling == CaloSampling::EME1   ||
                       sampling == CaloSampling::EME2   ||
                       sampling == CaloSampling::EME3   ||
                       sampling == CaloSampling::FCAL0     )
                    {
                      atomicAdd(&(moments_arr->engFracEM[cluster]), weighted_energy);
                    }
                  break;
                case 7:
                  if (volume > 0)
                    {
                      const float w_E_over_V = weighted_energy / volume;
                      atomicAdd(&(moments_arr->firstEngDens[cluster]), weighted_energy * w_E_over_V);
                      atomicAdd(&(moments_arr->secondEngDens[cluster]), weighted_energy * w_E_over_V * w_E_over_V);
                      atomicAdd(&(CMCHack::get_temporary_array<EnergyDensityNormalization>(moments_arr)[cluster]), weighted_energy);
                    }
                  break;
                case 8:
                case 9:
                case 10:
                  {
                    const float dir = x * x + y * y + z * z;
                    const float r_dir = (dir > 0.f ? rsqrtf(dir) : dir);
                    const float w_E_r_dir = weighted_energy * r_dir;
                    if (this_moment == 8)
                      {
                        const float mx = w_E_r_dir * x;
                        atomicAdd(&(CMCHack::get_temporary_array<MX>(moments_arr)[cluster]), mx);
                      }
                    else if (this_moment == 9)
                      {
                        const float my = w_E_r_dir * y;
                        atomicAdd(&(CMCHack::get_temporary_array<MY>(moments_arr)[cluster]), my);
                      }
                    else /*if (this_moment == 10)*/
                      {
                        const float mz = w_E_r_dir * z;
                        atomicAdd(&(CMCHack::get_temporary_array<MZ>(moments_arr)[cluster]), mz);
                      }
                  }
                  break;
                case 11:
                  atomicAdd(&(moments_arr->engPos[cluster]), weighted_energy);
                  break;
                case 12:
                  if (weighted_energy > 0)
                    {
                      unsigned long long int energy_and_cell = __float_as_uint(weighted_energy);
                      //Energy is positive, so no need to switch to total ordering...
                      energy_and_cell = (energy_and_cell << 32) | (cell + 1);

                      const unsigned long long int old_enc = atomicMax(&(CMCHack::get_temporary_array<MaxCellEnergyAndCell>(moments_arr)[cluster]), energy_and_cell);
                      atomicMax(&(CMCHack::get_temporary_array<SecondMaxCellEnergyAndCell>(moments_arr)[cluster]), min(old_enc, energy_and_cell));
                    }
                  break;
                case 13:
                  atomicAdd(&(clusters_arr->clusterEnergy[cluster]), energy * weight);
                  atomicAdd(&(CMCHack::get_temporary_array<SumAbsEnergyNonMoments>(moments_arr)[cluster]), abs_energy * weight);
                  break;
                case 14:
                  atomicAdd(&(clusters_arr->clusterEta[cluster]), abs_energy * weight * eta);
                  break;
                case 15:
                  {
                    const float phi_0 = CMCHack::get_temporary_array<SeedCellPhi>(moments_arr)[cluster];
                    const float phi_real = Helpers::regularize_angle(phi, phi_0);
                    atomicAdd(&(clusters_arr->clusterPhi[cluster]), phi_real * abs_energy * weight);
                  }
                  break;
                default:
                  break;
              }
          };

          if (tag.is_shared_between_clusters())
            {
              const float secondary_weight = __uint_as_float(tag.secondary_cluster_weight());
              if (in_warp_index >= WarpSize / 2)
                {
                  accumulateForCluster(tag.secondary_cluster_index(), secondary_weight, in_warp_index - WarpSize / 2);
                }
              else
                {
                  accumulateForCluster(tag.cluster_index(), 1.0f - secondary_weight, in_warp_index);
                }
            }
          else
            {
              accumulateForCluster(tag.cluster_index(), 1.0f, in_warp_index);
            }
        }

      //Optimization for our neighbour handling:
      //since we are using all2D,
      //which assures max neighbours < 32,
      //and which also encompasses
      //all of its smaller options,
      //we can have each thread knowing where to check
      //instead of building an actual neighbour list.
      //Also, this is symmetric,
      //that is, A being in the list of neighbours of B
      //implies B is in the list of neighbours of A.

      constexpr int neighbour_option_num = 4;

      const int num_relevant_neighbours = NeighOffset(geometry->neighbours.offsets[cell]).get_end_cell(neighbour_option_num);

      int cluster_to_check = 0;

      if (in_warp_index < num_relevant_neighbours)
        {
          const unsigned int mask = (1U << num_relevant_neighbours) - 1;

          const int neigh = geometry->neighbours.get_neighbour(cell, in_warp_index);
          const ClusterTag neigh_tag = cell_state_arr->clusterTag[neigh];

          if (neigh_tag.is_part_of_cluster())
            {
              cluster_to_check = neigh_tag.cluster_index() + 1;
              //We add 1 because 0 means no cluster to check here.
              //Also, since the cluster indices are 16 bit,
              //of course there's no issue here.
            }

          for (int i = 1; i < num_relevant_neighbours; ++i)
            {
              const int to_check = in_warp_index + i;
              const int warp_to_check = to_check % num_relevant_neighbours;
              const int other = __shfl_sync(mask, cluster_to_check, warp_to_check);
              if (warp_to_check < to_check && abs(other) == cluster_to_check)
                {
                  cluster_to_check = -cluster_to_check;
                  //Mark this cluster as already considered.
                }
            }

          //Maybe there is a solution that uses sorting instead?
          //Best choice here would be a sorting network
          //(bitonic sorting or Batcher's odd-even),
          //but then we'd need to eliminate the equal clusters too...
          //All in all, probably something like C ln(n)(ln(n) + 1) + 2 operations
          //per thread, with C somewhere around 2 rather than 0.5.
          //And, with n = 32, does the added complexity
          //really justify this? I would need a good reason
          //before considering to implement that sort of thing,
          //and I strongly suspect the performance benefits
          //not to be that significant. Still...

          if (cluster_to_check > 0)
            //Valid and non-repeated.
            {
              const int neigh_cluster = cluster_to_check - 1;
              if (tag.is_part_of_cluster())
                {
                  if (tag.cluster_index() != neigh_cluster)
                    {
                      atomicAdd(&(CMCHack::get_temporary_array<NumberNonEmptySamplings>(moments_arr)[sampling][neigh_cluster]), 1);
                    }
                }
              else
                {
                  atomicAdd(&(CMCHack::get_temporary_array<NumberEmptySamplings>(moments_arr)[sampling][neigh_cluster]), 1);
                }
            }
        }
    }
}

//Finalize/normalize the first pass moments,
//zero out what is needed for the second pass...

__global__ static
void firstClusterPassKernel(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr,
                            Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                            const bool skip_invalid)
{
  const int cluster_number = clusters_arr->number;

  const int index   = blockIdx.x * blockDim.x + threadIdx.x;
  const int moment  = threadIdx.x % WarpSize;
  const int grid_size = gridDim.x * blockDim.x;

  for (int cluster = index / WarpSize; cluster < cluster_number; cluster += grid_size / WarpSize)
    {
      if (skip_invalid && clusters_arr->seedCellID[cluster] < 0)
        {
          continue;
        }
      const float sum_energies = moments_arr->engPos[cluster];
      if (moment < NumSamplings)
        {
          const int sampling = moment;
          const float sampling_energy = moments_arr->energyPerSample[sampling][cluster];
          const unsigned int max_energy_pattern = CMCHack::get_temporary_array<MaxMomentsEnergyPerSample>(moments_arr)[sampling][cluster];
          const float sampling_max_energy = __uint_as_float(max_energy_pattern);
          const int sampling_empty = CMCHack::get_temporary_array<NumberEmptySamplings>(moments_arr)[sampling][cluster];
          const int sampling_non_empty = CMCHack::get_temporary_array<NumberNonEmptySamplings>(moments_arr)[sampling][cluster];

          int total = sampling_empty + sampling_non_empty;

          float isolation = 0.f, isolation_norm = 0.f, eng_frac_core = sampling_max_energy;

          if (total > 0 && sampling_energy > 0.f)
            {
              isolation = (sampling_energy * sampling_empty) / total;
              isolation_norm = sampling_energy;
            }

          const unsigned int mask = 0x0FFFFFFFU;
          //28 samplings, so without the last 4 threads.

          for (int i = 1; i < WarpSize; i *= 2)
            {
              const int origin = sampling ^ i;
              const float other_isol = __shfl_xor_sync(mask, isolation, i);
              const float other_isol_norm = __shfl_xor_sync(mask, isolation_norm, i);
              const float other_efc = __shfl_xor_sync(mask, eng_frac_core, i);
              if (origin < NumSamplings)
                {
                  isolation += other_isol;
                  isolation_norm += other_isol_norm;
                  eng_frac_core += other_efc;
                }
            }

          switch (moment)
            {
              case 0:
                moments_arr->isolation[cluster] = (isolation_norm != 0.f ? isolation / isolation_norm : 0.f);
                break;
              case NumSamplings - 1:
                moments_arr->engFracCore[cluster] = (sum_energies > 0.f) * eng_frac_core / sum_energies;
                break;
              default:
                break;
            }
        }
      else
        {
          switch (moment - NumSamplings)
            {
              case 0:
                {
                  const float rev_sum_energies = 1.0f / (sum_energies > 0.f ? sum_energies : 1.f);
                  moments_arr->centerX[cluster] *= rev_sum_energies;
                  moments_arr->centerY[cluster] *= rev_sum_energies;
                  moments_arr->centerZ[cluster] *= rev_sum_energies;
                  const float energy_density_norm = CMCHack::get_temporary_array<EnergyDensityNormalization>(moments_arr)[cluster];
                  const float rev_energy_density_norm = 1.f / (energy_density_norm > 0.f ? energy_density_norm : 1.f);
                  moments_arr->firstEngDens[cluster]  *= rev_energy_density_norm;
                  moments_arr->secondEngDens[cluster] *= rev_energy_density_norm;
                }
                break;
              case 1:
                {
                  moments_arr->engFracEM[cluster] /= (sum_energies > 0.f ? sum_energies : 1.f);

                  const float mx = CMCHack::get_temporary_array<MX>(moments_arr)[cluster];
                  const float my = CMCHack::get_temporary_array<MY>(moments_arr)[cluster];
                  const float mz = CMCHack::get_temporary_array<MZ>(moments_arr)[cluster];
                  const float sqrd_mass = sum_energies * sum_energies - mx * mx - my * my - mz * mz;

                  moments_arr->mass[cluster] = sqrtf(fabsf(sqrd_mass)) * ((sqrd_mass > 0.f) - (sqrd_mass < 0.f));
                }
                break;
              case 2:
                {
                  unsigned long long int max_E_and_cell = CMCHack::get_temporary_array<MaxCellEnergyAndCell>(moments_arr)[cluster];
                  const int max_cell = (max_E_and_cell & 0x7FFFFFFF) - 1;
                  const float max_E = __uint_as_float(max_E_and_cell >> 32);
                  moments_arr->engFracMax[cluster] = max_E / (sum_energies > 0.f ? sum_energies : 1.f);
                  CMCHack::get_temporary_array<MaxCells>(moments_arr)[cluster] = max_cell;

                  const unsigned long long int second_max_E_and_cell = CMCHack::get_temporary_array<SecondMaxCellEnergyAndCell>(moments_arr)[cluster];
                  const int second_max_cell = (second_max_E_and_cell & 0x7FFFFFFF) - 1;
                  CMCHack::get_temporary_array<SecondMaxCells>(moments_arr)[cluster] = second_max_cell;
                }
                break;
              case 3:
                {
                  const float abs_energy = CMCHack::get_temporary_array<SumAbsEnergyNonMoments>(moments_arr)[cluster];
                  if (abs_energy > 0)
                    {
                      const float tempeta = clusters_arr->clusterEta[cluster] / abs_energy;

                      clusters_arr->clusterEta[cluster] = tempeta;

                      const float temp_ET = clusters_arr->clusterEnergy[cluster] / coshf(abs(tempeta));

                      clusters_arr->clusterEt[cluster] = temp_ET;
                      clusters_arr->clusterPhi[cluster] = Helpers::regularize_angle(clusters_arr->clusterPhi[cluster] / abs_energy, 0.f);
                    }
                  /*
                  else
                    {
                      clusters_arr->seedCellID[cluster] = -1;
                    }
                  // */
                }
                break;
              default:
                break;
            }
        }

      switch (moment)
        {
          //Avoid 0 and NumSamplings - 1
          //since they set the sampling things...
          case 1:
            moments_arr->engBadCells[cluster] = 0.f;
            break;
          case 2:
            moments_arr->nBadCells[cluster] = 0;
            break;
          case 3:
            moments_arr->nBadCellsCorr[cluster] = 0;
            break;
          case 4:
            moments_arr->badCellsCorrE[cluster] = 0.f;
            break;
          case 5:
            moments_arr->badLArQFrac[cluster] = 0.f;
            break;
          case 6:
            moments_arr->avgLArQ[cluster] = 0.f;
            break;
          case 7:
            moments_arr->avgTileQ[cluster] = 0.f;
            break;
          case 8:
            moments_arr->PTD[cluster] = 0.f;
            break;
          case 9:
            moments_arr->numCells[cluster] = 0;
            break;
          case 10:
            CMCHack::get_temporary_array<NumPositiveEnergyCells>(moments_arr)[cluster] = 0;
            break;
          case 11:
            CMCHack::get_temporary_array<SumSquareEnergies>(moments_arr)[cluster] = 0.f;
            break;
          case 12:
            CMCHack::get_temporary_array<Matrix22>(moments_arr)[cluster] = 0.f;
            break;
          case 13:
            CMCHack::get_temporary_array<Matrix21>(moments_arr)[cluster] = 0.f;
            break;
          case 14:
            CMCHack::get_temporary_array<Matrix11>(moments_arr)[cluster] = 0.f;
            break;
          case 15:
            CMCHack::get_temporary_array<Matrix20>(moments_arr)[cluster] = 0.f;
            break;
          case 16:
            CMCHack::get_temporary_array<Matrix10>(moments_arr)[cluster] = 0.f;
            break;
          case 17:
            CMCHack::get_temporary_array<Matrix00>(moments_arr)[cluster] = 0.f;
            break;
          case 18:
            moments_arr->time[cluster] = 0.f;
            break;
          case 19:
            moments_arr->secondTime[cluster] = 0.f;
            break;
          case 20:
            CMCHack::get_temporary_array<TimeNormalization>(moments_arr)[cluster] = 0.f;
            break;
          case 21:
            CMCHack::get_temporary_array<AverageLArQNorm>(moments_arr)[cluster] = 0.f;
            break;
          case 22:
            CMCHack::get_temporary_array<AverageTileQNorm>(moments_arr)[cluster] = 0.f;
            break;
          case 23:
            CMCHack::get_temporary_array<MaxSignificanceAndSampling>(moments_arr)[cluster] = 0ULL;
            break;
          case 24:
            moments_arr->significance[cluster] = 0.f;
            break;
          default:
            break;
        }
    }
}

/******************************************************************************
 * Second pass.                                                               *
 ******************************************************************************/

__global__ static
void secondCellPassKernel(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr,
                          const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                          const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                          const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                          const Helpers::CUDA_kernel_object<CellNoiseArr> noise_arr,
                          const bool use_abs_energy, const bool use_two_gaussian_noise,
                          const float min_LAr_quality)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  const int in_warp_index = threadIdx.x % WarpSize;

  for (int cell = index / WarpSize; cell < NCaloCells; cell += grid_size / WarpSize)
    {
      const ClusterTag tag = cell_state_arr->clusterTag[cell];
      if (tag.is_part_of_cluster())
        {
          const float energy              = cell_info_arr->energy[cell];
          const float abs_energy          = fabsf(energy);
          const float moments_energy      = ((use_abs_energy || energy > 0.f) ? abs_energy : 0.f);
          const float time                = cell_info_arr->time[cell];
          const float x                   = geometry->x[cell];
          const float y                   = geometry->y[cell];
          const float z                   = geometry->z[cell];
          const bool  is_tile             = geometry->is_tile(cell);
          const int   sampling            = geometry->sampling(cell);
          const int   gain                = cell_info_arr->gain[cell];
          //No need to check for invalid cells as they won't be part of clusters...
          const float noise               = ( is_tile && use_two_gaussian_noise                        ?
                                              noise_arr->get_double_gaussian_noise(cell, gain, energy) :
                                              noise_arr->get_noise(cell, gain)                            );

          const QualityProvenance qp      = cell_info_arr->qualityProvenance[cell];
          const bool              is_bad  = cell_info_arr->is_bad(is_tile, qp, false);

          auto accumulateForCluster = [&](const int cluster, const float weight, const int this_moment)
          {
            const float weighted_energy = moments_energy * weight;
            const float weighted_energy_or_negative = (use_abs_energy ? fabsf(energy) : energy) * weight;
            const float square_w_E      = weighted_energy * weighted_energy;
            const float center_x        = moments_arr->centerX[cluster];
            const float center_y        = moments_arr->centerY[cluster];
            const float center_z        = moments_arr->centerZ[cluster];
            switch (this_moment)
              {
                case 0:
                  if (is_bad)
                    {
                      atomicAdd(&(moments_arr->engBadCells[cluster]), weighted_energy_or_negative);
                      atomicAdd(&(moments_arr->nBadCells[cluster]), 1);
                    }
                  break;
                case 1:
                  if (is_bad && energy != 0.f)
                    {
                      atomicAdd(&(moments_arr->badCellsCorrE[cluster]), weighted_energy_or_negative);
                      atomicAdd(&(moments_arr->nBadCellsCorr[cluster]), 1);
                    }
                  break;
                case 2:
                  if (!is_bad && !is_tile && ((qp.provenance() & 0x2800U) == 0x2000U) && qp.quality() > min_LAr_quality)
                    {
                      atomicAdd(&(moments_arr->badLArQFrac[cluster]), weighted_energy_or_negative);
                    }
                  break;
                case 3:
                  if (!is_bad && !is_tile && ((qp.provenance() & 0x2800U) == 0x2000U))
                    {
                      const float square_E_or_neg = weighted_energy_or_negative * weighted_energy_or_negative;
                      const float quality = qp.quality();
                      atomicAdd(&(moments_arr->avgLArQ[cluster]), square_E_or_neg * quality);
                      atomicAdd(&(CMCHack::get_temporary_array<AverageLArQNorm>(moments_arr)[cluster]), square_E_or_neg);
                    }
                  break;
                case 4:
                  if (!is_bad && is_tile && qp.tile_qual1() != 0xFFU && qp.tile_qual2() != 0xFFU)
                    {
                      const float square_E_or_neg = weighted_energy_or_negative * weighted_energy_or_negative;
                      const float max_quality = max((unsigned int) qp.tile_qual1(), (unsigned int) qp.tile_qual2());
                      atomicAdd(&(moments_arr->avgTileQ[cluster]), square_E_or_neg * max_quality);
                      atomicAdd(&(CMCHack::get_temporary_array<AverageTileQNorm>(moments_arr)[cluster]), square_E_or_neg);
                    }
                  break;
                case 5:
                  atomicAdd(&(moments_arr->PTD[cluster]), square_w_E);

                  //Comment on there:
                  //
                  //  +--------------- begin comment on there ---------------+
                  //  |                                                      |
                  //  | do not convert to pT since clusters are small and    |
                  //  | there is virtually no difference and cosh just costs |
                  //  | time ...                                             |
                  //  |                                                      |
                  //  +---------------- end comment on there ----------------+
                  //
                  //So maybe we could change this here?

                  atomicAdd(&(moments_arr->numCells[cluster]), 1);

                  if (weighted_energy_or_negative > 0)
                    {
                      atomicAdd(&(CMCHack::get_temporary_array<NumPositiveEnergyCells>(moments_arr)[cluster]), 1);
                    }

                  break;
                case 6:
                  atomicAdd(&(CMCHack::get_temporary_array<Matrix00>(moments_arr)[cluster]), square_w_E * (x - center_x) * (x - center_x));
                  break;
                case 7:
                  atomicAdd(&(CMCHack::get_temporary_array<Matrix10>(moments_arr)[cluster]), square_w_E * (x - center_x) * (y - center_y));
                  break;
                case 8:
                  atomicAdd(&(CMCHack::get_temporary_array<Matrix20>(moments_arr)[cluster]), square_w_E * (x - center_x) * (z - center_z));
                  break;
                case 9:
                  atomicAdd(&(CMCHack::get_temporary_array<Matrix11>(moments_arr)[cluster]), square_w_E * (y - center_y) * (y - center_y));
                  break;
                case 10:
                  atomicAdd(&(CMCHack::get_temporary_array<Matrix21>(moments_arr)[cluster]), square_w_E * (y - center_y) * (z - center_z));
                  break;
                case 11:
                  atomicAdd(&(CMCHack::get_temporary_array<Matrix22>(moments_arr)[cluster]), square_w_E * (z - center_z) * (z - center_z));
                  break;
                case 12:
                  atomicAdd(&(CMCHack::get_temporary_array<SumSquareEnergies>(moments_arr)[cluster]), square_w_E);
                  break;
                //Note: exclusion of PreSamplerB/E.
                case 13:
                  if ( ((is_tile && (qp.provenance() & 0x8080U)) || (!is_tile && (qp.provenance() & 0x2000U))) &&
                       sampling != CaloSampling::PreSamplerB && sampling != CaloSampling::PreSamplerE          )
                    {
                      const float normE = weight * energy;
                      const float squared_norm = normE * normE;
                      atomicAdd(&(moments_arr->time[cluster]), time * squared_norm);
                      atomicAdd(&(moments_arr->secondTime[cluster]), time * time * squared_norm);
                      atomicAdd(&(CMCHack::get_temporary_array<TimeNormalization>(moments_arr)[cluster]), squared_norm);
                    }
                  break;
                case 14:
                  atomicAdd(&(moments_arr->significance[cluster]), noise * noise);
                  break;
                case 15:
                  {
                    const float max_sig = noise > 0.f ? weighted_energy_or_negative / noise : 0.f;
                    unsigned long long int max_S_and_S = __float_as_uint(fabsf(max_sig));
                    max_S_and_S = max_S_and_S << 32 | (((unsigned long long int) sampling << 1)) | (max_sig > 0);
                    atomicMax(&(CMCHack::get_temporary_array<MaxSignificanceAndSampling>(moments_arr)[cluster]), max_S_and_S);
                  }
                  break;
                default:
                  break;
              }
          };

          if (tag.is_shared_between_clusters())
            {
              const float secondary_weight = __uint_as_float(tag.secondary_cluster_weight());
              if (in_warp_index >= WarpSize / 2)
                {
                  accumulateForCluster(tag.secondary_cluster_index(), secondary_weight, in_warp_index - WarpSize / 2);
                }
              else
                {
                  accumulateForCluster(tag.cluster_index(), 1.0f - secondary_weight, in_warp_index);
                }
            }
          else
            {
              accumulateForCluster(tag.cluster_index(), 1.0f, in_warp_index);
            }
        }
    }
}

__global__ static
void secondClusterPassKernel(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr,
                             Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                             const float max_axis_angle, const bool skip_invalid)
{
  const int cluster_number = clusters_arr->number;

  const int index   = blockIdx.x * blockDim.x + threadIdx.x;
  const int moment  = threadIdx.x % WarpSize;
  const int grid_size = gridDim.x * blockDim.x;

  for (int cluster = index / WarpSize; cluster < cluster_number; cluster += grid_size / WarpSize)
    {
      if (skip_invalid && clusters_arr->seedCellID[cluster] < 0)
        {
          continue;
        }

      const float sum_energies = moments_arr->engPos[cluster];
      const float cluster_energy = clusters_arr->clusterEnergy[cluster];
      switch (moment)
        {
          case WarpSize - 1:
          case WarpSize - 2:
          case WarpSize - 3:
            {
              const float center_x   = moments_arr->centerX[cluster];
              const float center_y   = moments_arr->centerY[cluster];
              const float center_z   = moments_arr->centerZ[cluster];
              const float center_mag_inv = rnorm3df(center_x, center_y, center_z);
              moments_arr->centerMag[cluster] = 1.0f / center_mag_inv;
              float axis_x = center_x * center_mag_inv;
              float axis_y = center_y * center_mag_inv;
              float axis_z = center_z * center_mag_inv;
              float delta_phi = 0, delta_theta = 0, delta_alpha = 0;

              if (CMCHack::get_temporary_array<NumPositiveEnergyCells>(moments_arr)[cluster] > 2)
                {
                  const float norm = 1.f / CMCHack::get_temporary_array<SumSquareEnergies>(moments_arr)[cluster];
                  RealSymmetricMatrixSolver solver { CMCHack::get_temporary_array<Matrix00>(moments_arr)[cluster] * norm,
                                                     CMCHack::get_temporary_array<Matrix11>(moments_arr)[cluster] * norm,
                                                     CMCHack::get_temporary_array<Matrix22>(moments_arr)[cluster] * norm,
                                                     CMCHack::get_temporary_array<Matrix10>(moments_arr)[cluster] * norm,
                                                     CMCHack::get_temporary_array<Matrix21>(moments_arr)[cluster] * norm,
                                                     CMCHack::get_temporary_array<Matrix20>(moments_arr)[cluster] * norm  };

                  float lambda = 0, vec[3];

                  switch (moment)
                    {
                      case WarpSize - 1:
                        solver.get_solution_pair_2(lambda, vec, true);
                        break;
                      case WarpSize - 2:
                        solver.get_solution_pair_3(lambda, vec, true);
                        break;
                      //2 and 3 are switched compared to the Eigen solution.
                      case WarpSize - 3:
                        solver.get_solution_pair_1(lambda, vec, true);
                        break;
                      default:
                        break;
                    }

                  const float min_lambdas = 1.e-6f;

                  const unsigned int mask = 0xE0000000U;
                  //The last three threads.

                  const float lambda_1 = __shfl_sync(mask, lambda, WarpSize - 3);
                  const float lambda_2 = __shfl_sync(mask, lambda, WarpSize - 2);
                  const float lambda_3 = __shfl_sync(mask, lambda, WarpSize - 1);

                  if ( solver.well_defined(lambda_1, lambda_2, lambda_3, min_lambdas)  &&
                       fabsf(lambda_1) >= min_lambdas                                  &&
                       fabsf(lambda_2) >= min_lambdas                                  &&
                       fabsf(lambda_3) >= min_lambdas                                     )
                    {
                      const float prod = (vec[0] * axis_x + vec[1] * axis_y + vec[2] * axis_z);
                      const float raw_angle = acosf(prod > 1.f ? 1.f : (prod < -1.f ? -1.f : prod));

                      float this_angle = raw_angle;

                      if (raw_angle > Helpers::Constants::pi<float> / 2)
                        {
                          this_angle = Helpers::Constants::pi<float> - raw_angle;
                          vec[0] *= -1;
                          vec[1] *= -1;
                          vec[2] *= -1;
                        }

                      const float angle_1 = __shfl_sync(mask, this_angle, WarpSize - 3);
                      const float angle_2 = __shfl_sync(mask, this_angle, WarpSize - 2);
                      const float angle_3 = __shfl_sync(mask, this_angle, WarpSize - 1);


                      float chosen_angle = 0, chosen_vec[3] = {0, 0, 0};

                      if (angle_1 <= angle_2 && angle_1 <= angle_3)
                        {
                          chosen_angle = angle_1;
                          chosen_vec[0] = __shfl_sync(mask, vec[0], WarpSize - 3);
                          chosen_vec[1] = __shfl_sync(mask, vec[1], WarpSize - 3);
                          chosen_vec[2] = __shfl_sync(mask, vec[2], WarpSize - 3);
                        }
                      else if (angle_2 < angle_1 && angle_2 <= angle_3)
                        {
                          chosen_angle = angle_2;
                          chosen_vec[0] = __shfl_sync(mask, vec[0], WarpSize - 2);
                          chosen_vec[1] = __shfl_sync(mask, vec[1], WarpSize - 2);
                          chosen_vec[2] = __shfl_sync(mask, vec[2], WarpSize - 2);
                        }
                      else if (angle_3 < angle_2 && angle_3 < angle_1)
                        {
                          chosen_angle = angle_3;
                          chosen_vec[0] = __shfl_sync(mask, vec[0], WarpSize - 1);
                          chosen_vec[1] = __shfl_sync(mask, vec[1], WarpSize - 1);
                          chosen_vec[2] = __shfl_sync(mask, vec[2], WarpSize - 1);
                        }
                      /*
                      else
                        {
                          clusters_arr->seedCellID[cluster] = -1;
                        }
                      // */

                      auto calc_phi = [](const float x, const float y, const float z)
                      {
                        return atan2f(y, x);
                      };
                      auto calc_theta = [](const float x, const float y, const float z)
                      {
                        return atan2f(1.0f, z * rhypotf(x, y));
                      };

                      switch (moment)
                        {
                          case WarpSize - 3:
                            delta_phi = Helpers::angular_difference(calc_phi(axis_x, axis_y, axis_z), calc_phi(chosen_vec[0], chosen_vec[1], chosen_vec[2]));
                            if (chosen_angle < max_axis_angle)
                              {
                                axis_x = chosen_vec[0];
                              }
                            /*
                            else
                              {
                                clusters_arr->seedCellID[cluster] = -1;
                              }
                            // */
                            break;
                          case WarpSize - 2:
                            delta_theta = calc_theta(axis_x, axis_y, axis_z) - calc_theta(chosen_vec[0], chosen_vec[1], chosen_vec[2]);
                            if (chosen_angle < max_axis_angle)
                              {
                                axis_y = chosen_vec[1];
                              }
                            /*
                            else
                              {
                                clusters_arr->seedCellID[cluster] = -1;
                              }
                            // */
                            break;
                          case WarpSize - 1:
                            delta_alpha = chosen_angle;
                            if (chosen_angle < max_axis_angle)
                              {
                                axis_z = chosen_vec[2];
                              }
                            /*
                            else
                              {
                                clusters_arr->seedCellID[cluster] = -1;
                              }
                            // */
                            break;
                          default:
                            break;
                        }

                      __syncwarp(mask);

                    }
                  /*
                  else
                    {
                      clusters_arr->seedCellID[cluster] = -1;
                    }
                  // */
                }

              switch (moment)
                {
                  case WarpSize - 3:
                    CMCHack::get_temporary_array<ShowerAxisX>(moments_arr)[cluster] = axis_x;
                    moments_arr->deltaPhi[cluster] = delta_phi;
                    break;
                  case WarpSize - 2:
                    CMCHack::get_temporary_array<ShowerAxisY>(moments_arr)[cluster] = axis_y;
                    moments_arr->deltaTheta[cluster] = delta_theta;
                    break;
                  case WarpSize - 1:
                    CMCHack::get_temporary_array<ShowerAxisZ>(moments_arr)[cluster] = axis_z;
                    moments_arr->deltaAlpha[cluster] = delta_alpha;
                    break;
                  default:
                    break;
                }

            }
            break;
          case 0:
            {
              moments_arr->badLArQFrac[cluster] /= (cluster_energy != 0.f ? cluster_energy : 1.f);
            }
            break;
          case 1:
            {
              const float prev_v = moments_arr->significance[cluster];
              moments_arr->significance[cluster] = (prev_v > 0.f ? cluster_energy * rsqrtf(prev_v) : 0.f);
            }
            break;
          case 2:
            {
              const unsigned long long int max_sig_and_samp = CMCHack::get_temporary_array<MaxSignificanceAndSampling>(moments_arr)[cluster];
              const float max_sig = __uint_as_float(max_sig_and_samp >> 32);
              const int max_samp = (max_sig_and_samp & 0xFFFFFFFEU) >> 1;
              moments_arr->cellSignificance[cluster] = max_sig * (max_sig_and_samp & 1 ? 1.f : -1.f);
              moments_arr->cellSigSampling[cluster] = max_samp;
            }
            break;
          case 3:
            {
              const float norm_LAr = CMCHack::get_temporary_array<AverageLArQNorm>(moments_arr)[cluster];
              moments_arr->avgLArQ[cluster] /= (norm_LAr > 0.f ? norm_LAr : 1.0f);
            }
            break;
          case 4:
            {
              const float norm_Tile = CMCHack::get_temporary_array<AverageTileQNorm>(moments_arr)[cluster];
              moments_arr->avgTileQ[cluster] /= (norm_Tile > 0.f ? norm_Tile : 1.0f);
            }
            break;
          case 5:
            {
              const float old = moments_arr->PTD[cluster];
              moments_arr->PTD[cluster] = 1.0f / ((sum_energies > 0.f ? sum_energies : 1.f) * rsqrtf(old));
              //See before: maybe to be revised?
            }
            break;
          case 6:
            {
              const float time_norm = CMCHack::get_temporary_array<TimeNormalization>(moments_arr)[cluster];
              if (time_norm != 0.f)
                {
                  const float real_norm = 1.0f / time_norm;
                  const float time = moments_arr->time[cluster] * real_norm;
                  const float second_sum = moments_arr->secondTime[cluster];
                  moments_arr->time[cluster] = time;
                  moments_arr->secondTime[cluster] = (second_sum * real_norm) - (time * time);
                }
              else
                {
                  moments_arr->time[cluster] = 0.f;
                  moments_arr->secondTime[cluster] = 0.f;
                }
            }
            break;
          case 7:
            if (moments_arr->numCells[cluster] <= 0)
              {
                clusters_arr->seedCellID[cluster] = -1;
              }
            break;
          default:
            break;
        }

      __syncwarp();

      //Now zero out what we need for the final (!) moments.
      //Avoid overburdening WarpSize - 3 to WarpSize - 1.
      //Also use 0 to NumSamplings - 1 for the sampling-based ones,
      //so try to load-balance with those that did less before.

      switch (moment)
        {
          case 8:
            moments_arr->firstPhi[cluster] = 0.f;
            break;
          case 9:
            moments_arr->firstEta[cluster] = 0.f;
            break;
          case 10:
            moments_arr->secondR[cluster] = 0.f;
            break;
          case 11:
            moments_arr->secondLambda[cluster] = 0.f;
            break;
          case 12:
            moments_arr->lateral[cluster] = 0.f;
            break;
          case 13:
            moments_arr->longitudinal[cluster] = 0.f;
            break;
          case 14:
            moments_arr->nExtraCellSampling[cluster] = 0;
            break;
          case 15:
            CMCHack::get_temporary_array<LateralNormalization>(moments_arr)[cluster] = 0.f;
            break;
          case 16:
            CMCHack::get_temporary_array<LongitudinalNormalization>(moments_arr)[cluster] = 0.f;
            break;
          default:
            break;
        }
      if (moment < NumSamplings)
        {
          const int sampling = moment;
          CMCHack::get_temporary_array<MaxEnergyAndCellPerSample>(moments_arr)[sampling][cluster] = 0;
          moments_arr->etaPerSample[sampling][cluster] = 0.f;
          moments_arr->phiPerSample[sampling][cluster] = 0.f;
          CMCHack::get_temporary_array<AbsoluteEnergyPerSample>(moments_arr)[sampling][cluster] = 0.f;
        }
    }
}

/******************************************************************************
 * Third pass.                                                                *
 ******************************************************************************/

__global__ static
void thirdCellPassKernel(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr,
                         const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                         const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                         const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                         const bool use_abs_energy, const float eta_inner_wheel,
                         const float min_l_longitudinal, const float min_r_lateral)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  const int grid_size = gridDim.x * blockDim.x;
  const int in_warp_index = threadIdx.x % WarpSize;

  for (int cell = index / WarpSize; cell < NCaloCells; cell += grid_size / WarpSize)
    {
      const ClusterTag tag = cell_state_arr->clusterTag[cell];
      if (tag.is_part_of_cluster())
        {
          const float energy         = cell_info_arr->energy[cell];
          const float abs_energy     = fabsf(energy);
          const float moments_energy = ((use_abs_energy || energy > 0.f) ? abs_energy : 0.f);
          const float x              = geometry->x[cell];
          const float y              = geometry->y[cell];
          const float z              = geometry->z[cell];
          const float eta            = geometry->eta[cell];
          const float phi            = geometry->phi[cell];
          const int   sampling       = geometry->sampling(cell);

          auto accumulateForCluster = [&](const int cluster, const float weight, const int this_moment)
          {
            const float weighted_energy = moments_energy * weight;
            const float center_x        = moments_arr->centerX[cluster];
            const float center_y        = moments_arr->centerY[cluster];
            const float center_z        = moments_arr->centerZ[cluster];
            const float axis_x          = CMCHack::get_temporary_array<ShowerAxisX>(moments_arr)[cluster];
            const float axis_y          = CMCHack::get_temporary_array<ShowerAxisY>(moments_arr)[cluster];
            const float axis_z          = CMCHack::get_temporary_array<ShowerAxisZ>(moments_arr)[cluster];
            const int   max_cell        = CMCHack::get_temporary_array<MaxCells>(moments_arr)[cluster];
            const int   second_max_cell = CMCHack::get_temporary_array<SecondMaxCells>(moments_arr)[cluster];

            /*
            auto cross_p_mag = [](const float x1, const float x2, const float x3,
                                  const float y1, const float y2, const float y3)
            {
              const float a = x2 * y3 - x3 * y2;
              const float b = x3 * y1 - x1 * y3;
              const float c = x1 * y2 - x2 * y1;
              return norm3df(a, b, c);
            };
            */

            auto dot_p = [](const float x1, const float x2, const float x3,
                            const float y1, const float y2, const float y3)
            {
              return x1 * y1 + x2 * y2 + x3 * y3;
            };

            //d\vec{v} = \vec{r}_{cell} - \vec{r}_{center}
            //
            //  r      = ||d\vec{v} \cross \vec{axis}||
            //  lambda = d\vec{v} \dot \vec{axis}
            //
            //  lambda = ||d\vec{v}||  cos (\theta)
            //  r      = ||d\vec{v}|| |sin(\theta)|
            //
            //  |sin(\theta)| = sqrt(1 - cos(\theta)^2)

            const float dx = x - center_x;
            const float dy = y - center_y;
            const float dz = z - center_z;

            const float d_mag_inv = rnorm3df(dx, dy, dz);

            const float lambda = dot_p(dx, dy, dz, axis_x, axis_y, axis_z);

            const float cos_theta = lambda * d_mag_inv;

            const float r = 1.0f / (rsqrtf(1 - cos_theta * cos_theta) * d_mag_inv);


            //const float r      = cross_p_mag(x - center_x, y - center_y, z - center_z, axis_x, axis_y, axis_z);
            //const float lambda = dot_p(x - center_x, y - center_y, z - center_z, axis_x, axis_y, axis_z);

            switch (this_moment)
              {
                case 0:
                  {
                    const float phi_0 = CMCHack::get_temporary_array<SeedCellPhi>(moments_arr)[cluster];
                    const float phi_real = Helpers::regularize_angle(phi, phi_0);
                    atomicAdd(&(moments_arr->firstPhi[cluster]), weighted_energy * phi_real);
                  }
                  break;
                case 1:
                  atomicAdd(&(moments_arr->firstEta[cluster]), weighted_energy * eta);
                  break;
                case 2:
                  atomicAdd(&(moments_arr->secondR[cluster]), weighted_energy * r * r);
                  break;
                case 3:
                  atomicAdd(&(moments_arr->secondLambda[cluster]), weighted_energy * lambda * lambda);
                  break;
                case 4:
                  atomicAdd(&(moments_arr->etaPerSample[sampling][cluster]), abs_energy * weight * eta);
                  break;
                case 5:
                  {
                    const float phi_0 = CMCHack::get_temporary_array<SeedCellPhi>(moments_arr)[cluster];
                    const float phi_real = Helpers::regularize_angle(phi, phi_0);
                    atomicAdd(&(moments_arr->phiPerSample[sampling][cluster]), abs_energy * weight * phi_real);
                  }
                  break;
                case 6:
                  atomicAdd(&(CMCHack::get_temporary_array<AbsoluteEnergyPerSample>(moments_arr)[sampling][cluster]), abs_energy * weight);
                  break;
                case 7:
                  if (sampling == CaloSampling::EME2 && fabsf(eta) > eta_inner_wheel)
                    {
                      atomicAdd(&(moments_arr->nExtraCellSampling[cluster]), 1);
                    }
                  break;
                case 8:
                  if (cell != max_cell && cell != second_max_cell)
                    {
                      atomicAdd(&(moments_arr->lateral[cluster]), weighted_energy * r * r);
                    }
                  break;
                case 9:
                  if (cell != max_cell && cell != second_max_cell)
                    {
                      atomicAdd(&(CMCHack::get_temporary_array<LateralNormalization>(moments_arr)[cluster]), weighted_energy * r * r);
                    }
                  else
                    {
                      const float real_r = max(r, min_r_lateral);
                      atomicAdd(&(CMCHack::get_temporary_array<LateralNormalization>(moments_arr)[cluster]), weighted_energy * real_r * real_r);
                    }
                  break;
                case 10:
                  if (cell != max_cell && cell != second_max_cell)
                    {
                      atomicAdd(&(moments_arr->longitudinal[cluster]), weighted_energy * lambda * lambda);
                    }
                  break;
                case 11:
                  if (cell != max_cell && cell != second_max_cell)
                    {
                      atomicAdd(&(CMCHack::get_temporary_array<LongitudinalNormalization>(moments_arr)[cluster]), weighted_energy * lambda * lambda);
                    }
                  else
                    {
                      const float real_lambda = max(lambda, min_l_longitudinal);
                      atomicAdd(&(CMCHack::get_temporary_array<LongitudinalNormalization>(moments_arr)[cluster]), weighted_energy * real_lambda * real_lambda);
                    }
                  break;
                case 12:
                  {
                    const unsigned int energy_pattern = __float_as_uint(energy * weight);
                    unsigned long long int E_and_cell = FloatingPointHelpers::StandardFloat::to_total_ordering(energy_pattern);
                    E_and_cell = (E_and_cell << 32) | cell;
                    atomicMax(&(CMCHack::get_temporary_array<MaxEnergyAndCellPerSample>(moments_arr)[sampling][cluster]), E_and_cell);
                  }
                  break;
                default:
                  break;
              }
          };

          if (tag.is_shared_between_clusters())
            {
              const float secondary_weight = __uint_as_float(tag.secondary_cluster_weight());
              if (in_warp_index >= WarpSize / 2)
                {
                  accumulateForCluster(tag.secondary_cluster_index(), secondary_weight, in_warp_index - WarpSize / 2);
                }
              else
                {
                  accumulateForCluster(tag.cluster_index(), 1.0f - secondary_weight, in_warp_index);
                }
            }
          else
            {
              accumulateForCluster(tag.cluster_index(), 1.0f, in_warp_index);
            }
        }
    }
}

__global__ static
void thirdClusterPassKernel(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr,
                            Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                            const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                            const bool skip_invalid)
{
  const int cluster_number = clusters_arr->number;

  const int index   = blockIdx.x * blockDim.x + threadIdx.x;
  const int moment  = threadIdx.x % WarpSize;
  const int grid_size = gridDim.x * blockDim.x;

  for (int cluster = index / WarpSize; cluster < cluster_number; cluster += grid_size / WarpSize)
    {
      if (skip_invalid && clusters_arr->seedCellID[cluster] < 0)
        {
          continue;
        }
      const float sum_energies = moments_arr->engPos[cluster];
      if (moment < NumSamplings)
        {
          const int sampling = moment;
          const float sampling_normalization = CMCHack::get_temporary_array<AbsoluteEnergyPerSample>(moments_arr)[sampling][cluster];

          const unsigned long long int energy_and_cell = CMCHack::get_temporary_array<MaxEnergyAndCellPerSample>(moments_arr)[sampling][cluster];

          const int cell = sampling_normalization > 0.f ? (int) (energy_and_cell & 0x7FFFFFFF) : -1;

          CMCHack::get_temporary_array<MaxECellPerSample>(moments_arr)[sampling][cluster] = cell;

          const float eta_phi_normalization = 1.0f / (sampling_normalization != 0.f ? sampling_normalization : 1.0f);

          moments_arr->etaPerSample[sampling][cluster] *= eta_phi_normalization;

          const float old_phi = moments_arr->phiPerSample[sampling][cluster];

          moments_arr->phiPerSample[sampling][cluster] = Helpers::regularize_angle(old_phi * eta_phi_normalization, 0.f);
        }
      else
        {
          const int thread_id = moment - NumSamplings;

          const float center_x        = moments_arr->centerX[cluster];
          const float center_y        = moments_arr->centerY[cluster];
          const float center_z        = moments_arr->centerZ[cluster];
          const float axis_x          = CMCHack::get_temporary_array<ShowerAxisX>(moments_arr)[cluster];
          const float axis_y          = CMCHack::get_temporary_array<ShowerAxisY>(moments_arr)[cluster];
          const float axis_z          = CMCHack::get_temporary_array<ShowerAxisZ>(moments_arr)[cluster];

          const float center_phi = Helpers::regularize_angle(atan2f(center_y, center_x));

          const float center_eta = Helpers::eta_from_coordinates(center_x, center_y, center_z);

          const int first_attempt_cell = geometry->get_closest_cell(CaloSampling::EMB1, center_eta, center_phi);

          float lambda_c = 0.f;

          if (first_attempt_cell >= 0)
            //Condition on CPU is r_calo == 0,
            //but, by definition, I'd expect no cell
            //to potentially overlap with the center
            //of the detector, right?!
            {
              if (thread_id == 0)
                {
                  const float r_calo = geometry->r[first_attempt_cell] - geometry->dr[first_attempt_cell] /
                                       (geometry->is_tile(first_attempt_cell) ? 2.f : 1.f);

                  const float axis_r = axis_x * axis_x + axis_y * axis_y;

                  if (axis_r > 0)
                    {
                      const float rev_axis_r = 1.0f / axis_r;
                      const float axis_and_center_r = axis_x * center_x + axis_y * center_y;
                      const float center_r = center_x * center_x + center_y * center_y - r_calo * r_calo;
                      const float det = axis_and_center_r * axis_and_center_r * (rev_axis_r * rev_axis_r) - center_r * rev_axis_r;
                      if (det > 0)
                        {
                          const float quot = -axis_and_center_r * rev_axis_r;
                          const float rootdet = sqrtf(det);
                          const float branch_1 = quot + rootdet;
                          const float branch_2 = quot - rootdet;
                          lambda_c = min(fabsf(branch_1), fabsf(branch_2));
                        }
                      /*
                      else
                        {
                          clusters_arr->seedCellID[cluster] = -1;
                        }
                      // */
                    }
                  /*
                  else
                    {
                      clusters_arr->seedCellID[cluster] = -1;
                    }
                  // */

                }
            }
          else
            {
              int this_sampling = -1;
              switch (thread_id)
                {
                  case 0:
                    this_sampling = CaloSampling::EME1;
                    break;
                  case 1:
                    this_sampling = CaloSampling::EME2;
                    break;
                  case 2:
                    this_sampling = CaloSampling::FCAL0;
                    break;
                  case 3:
                    this_sampling = CaloSampling::HEC0;
                    break;
                  default:
                    break;
                }
              if (this_sampling >= 0)
                {
                  const int this_cell = geometry->get_closest_cell(this_sampling, center_eta, center_phi);
                  float this_calc = 0.f;

                  if (this_cell >= 0)
                    {
                      const float this_z = geometry->z[this_cell];
                      this_calc = this_z + (this_z >= 0 ? -geometry->dz[this_cell] : geometry->dz[this_cell]) /
                                  (geometry->is_tile(this_cell) ? 2.f : 1.f);
                    }

                  const unsigned int mask = 0xF0000000U;
                  //The last 4 threads.

                  const float new_one = __shfl_down_sync(mask, this_calc, 1);
                  if (this_calc == 0.f)
                    {
                      this_calc = new_one;
                    }
                  //(0, 1) and (2, 3) get the wanted between the both of them.
                  const float new_two = __shfl_down_sync(mask, this_calc, 2);
                  if (this_calc == 0.f)
                    {
                      this_calc = new_two;
                    }
                  //0 got the correct one.

                  if (this_calc != 0.f && axis_z != 0.f)
                    {
                      lambda_c = fabsf( (this_calc - center_z) / axis_z );
                    }
                }
            }

          switch (thread_id)
            {
              case 0:
                if (first_attempt_cell < 0)
                  {
                    moments_arr->lateral[cluster] /= CMCHack::get_temporary_array<LateralNormalization>(moments_arr)[cluster];
                  }
                moments_arr->centerLambda[cluster] = lambda_c;
                break;
              case 1:
                if (first_attempt_cell >= 0)
                  {
                    moments_arr->lateral[cluster] /= CMCHack::get_temporary_array<LateralNormalization>(moments_arr)[cluster];
                  }
                moments_arr->longitudinal[cluster] /= CMCHack::get_temporary_array<LongitudinalNormalization>(moments_arr)[cluster];
                break;
              case 2:
                {
                  const float energy_normalization = 1.0f / (sum_energies > 0.f ? sum_energies : 1.f);
                  moments_arr->secondR[cluster]      *= energy_normalization;
                  moments_arr->secondLambda[cluster] *= energy_normalization;
                  break;
                }
              case 3:
                {
                  const float old_first_phi = moments_arr->firstPhi[cluster];
                  const float energy_normalization = 1.0f / (sum_energies > 0.f ? sum_energies : 1.f);
                  moments_arr->firstPhi[cluster] = Helpers::regularize_angle(old_first_phi * energy_normalization);
                  moments_arr->firstEta[cluster] *= energy_normalization;
                }
                break;
              default:
                break;
            }

        }
    }
}

/******************************************************************************
 * Final cleanup.                                                                *
 ******************************************************************************/

__global__ static
void finalClusterPassKernel(Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr,
                            Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                            const Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                            const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                            const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                            const bool skip_invalid)
{
  const int cluster_number = clusters_arr->number;

  const int index   = blockIdx.x * blockDim.x + threadIdx.x;
  const int thread_index  = threadIdx.x % WarpSize;
  const int grid_size = gridDim.x * blockDim.x;

  for (int cluster = index / WarpSize; cluster < cluster_number; cluster += grid_size / WarpSize)
    {
      if (skip_invalid && clusters_arr->seedCellID[cluster] < 0)
        {
          continue;
        }
      const float sum_energies = moments_arr->engPos[cluster];
      if (thread_index < NumSamplings)
        {
          const int sampling = thread_index;
          const int max_cell = CMCHack::get_temporary_array<MaxECellPerSample>(moments_arr)[sampling][cluster];
          if (max_cell >= 0 && max_cell < NCaloCells)
            {
              const ClusterTag tag = cell_state_arr->clusterTag[max_cell];
              const float sec_weight = __uint_as_float(tag.secondary_cluster_weight());

              moments_arr->maxEPerSample[sampling][cluster]   = cell_info_arr->energy[max_cell] * (tag.cluster_index() == cluster ? 1.0f - sec_weight : sec_weight);
              //The cell can belong to either the first or second cluster.

              moments_arr->maxPhiPerSample[sampling][cluster] = geometry->phi[max_cell];
              moments_arr->maxEtaPerSample[sampling][cluster] = geometry->eta[max_cell];
            }
        }
      else if (sum_energies <= 0.f)
        {
          /*
          clusters_arr->seedCellID[cluster] = -1;
          // */

          //Maybe we can use more threads in parallel?
          //Doesn't seem much likely given that the sampling stuff
          //takes quuuite a while with all the memory accesses...
          switch (thread_index - NumSamplings)
            {
              case 0:
                moments_arr->firstPhi     [cluster] = 0;
                moments_arr->firstEta     [cluster] = 0;
                moments_arr->secondR      [cluster] = 0;
                moments_arr->secondLambda [cluster] = 0;
                moments_arr->deltaPhi     [cluster] = 0;
                moments_arr->deltaTheta   [cluster] = 0;
                moments_arr->deltaAlpha   [cluster] = 0;
                moments_arr->centerX      [cluster] = 0;
                moments_arr->centerY      [cluster] = 0;
                moments_arr->centerZ      [cluster] = 0;
                break;
              case 1:
                moments_arr->centerMag     [cluster] = 0;
                moments_arr->centerLambda  [cluster] = 0;
                moments_arr->lateral       [cluster] = 0;
                moments_arr->longitudinal  [cluster] = 0;
                moments_arr->engFracEM     [cluster] = 0;
                moments_arr->engFracMax    [cluster] = 0;
                moments_arr->engFracCore   [cluster] = 0;
                moments_arr->firstEngDens  [cluster] = 0;
                moments_arr->secondEngDens [cluster] = 0;
                moments_arr->isolation     [cluster] = 0;
                break;
              case 2:
                moments_arr->engBadCells      [cluster] = 0;
                moments_arr->nBadCells        [cluster] = 0;
                moments_arr->nBadCellsCorr    [cluster] = 0;
                moments_arr->badCellsCorrE    [cluster] = 0;
                moments_arr->badLArQFrac      [cluster] = 0;
                moments_arr->significance     [cluster] = 0;
                moments_arr->cellSignificance [cluster] = 0;
                moments_arr->cellSigSampling  [cluster] = 0;
                moments_arr->avgLArQ          [cluster] = 0;
                moments_arr->avgTileQ         [cluster] = 0;
                break;
              case 3:
                moments_arr->engBadHVCells       [cluster] = 0;
                moments_arr->nBadHVCells         [cluster] = 0;
                moments_arr->PTD                 [cluster] = 0;
                moments_arr->mass                [cluster] = 0;
                moments_arr->EMProbability       [cluster] = 0;
                moments_arr->hadWeight           [cluster] = 0;
                moments_arr->OOCweight           [cluster] = 0;
                moments_arr->DMweight            [cluster] = 0;
                moments_arr->tileConfidenceLevel [cluster] = 0;
                break;
              default:
                break;
            }
        }
    }
}

/******************************************************************************
 * Actual kernel calling code.                                                *
 ******************************************************************************/

__global__ static
void calculateClusterPropertiesAndMomentsDeferKernel(Helpers::CUDA_kernel_object<ClusterInfoArr> clusters_arr,
                                                     Helpers::CUDA_kernel_object<ClusterMomentsArr> moments_arr,
                                                     Helpers::CUDA_kernel_object<CellStateArr> cell_state_arr,
                                                     const Helpers::CUDA_kernel_object<CellInfoArr> cell_info_arr,
                                                     const Helpers::CUDA_kernel_object<GeometryArr> geometry,
                                                     const Helpers::CUDA_kernel_object<CellNoiseArr> noise_arr,
                                                     const Helpers::CUDA_kernel_object<ClusterMomentCalculationOptions> opts,
                                                     const int i_dimBlockClusters,  const int i_dimBlockCells)
{
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index == 0)
    {
      const int cluster_number = clusters_arr->number;

      const int i_dimGridClusters = Helpers::int_ceil_div(cluster_number, Helpers::int_floor_div(i_dimBlockClusters, WarpSize));
      const int i_dimGridCells = Helpers::int_ceil_div(NCaloCells, Helpers::int_floor_div(i_dimBlockCells, WarpSize));

      zerothClusterPassKernel <<< i_dimGridClusters, i_dimBlockClusters>>>(moments_arr, clusters_arr, geometry,
                                                                           opts->skip_invalid_clusters);


      firstCellPassKernel <<< i_dimGridCells, i_dimBlockCells>>>(moments_arr, clusters_arr, cell_state_arr, cell_info_arr,
                                                                 geometry, opts->use_abs_energy);
      firstClusterPassKernel <<< i_dimGridClusters, i_dimBlockClusters>>>(moments_arr, clusters_arr, opts->skip_invalid_clusters);


      secondCellPassKernel <<< i_dimGridCells, i_dimBlockCells>>>(moments_arr, cell_state_arr, cell_info_arr, geometry, noise_arr,
                                                                  opts->use_abs_energy, opts->use_two_gaussian_noise, opts->min_LAr_quality);
      secondClusterPassKernel <<< i_dimGridClusters, i_dimBlockClusters>>>(moments_arr, clusters_arr,
                                                                           opts->max_axis_angle, opts->skip_invalid_clusters);


      thirdCellPassKernel <<< i_dimGridCells, i_dimBlockCells>>>(moments_arr, cell_state_arr, cell_info_arr, geometry, opts->use_abs_energy,
                                                                 opts->eta_inner_wheel, opts->min_l_longitudinal, opts->min_r_lateral);
      thirdClusterPassKernel <<< i_dimGridClusters, i_dimBlockClusters>>>(moments_arr, clusters_arr, geometry,
                                                                          opts->skip_invalid_clusters);


      finalClusterPassKernel <<< i_dimGridClusters, i_dimBlockClusters>>>(moments_arr, clusters_arr, cell_state_arr, cell_info_arr,
                                                                          geometry, opts->skip_invalid_clusters);


      //clearInvalidCells <<< i_dimGridCells, i_dimBlockCells>>>(cell_state_arr, clusters_arr);

      //We could have split this up and not rely so much on dynamic parallelism.
      //However, if not using CUDA 12 (which we probably won't be for a while),
      //we'd have to dyn-par our way through the number of clusters at every cluster-related kernel.
      //With tail calls, it's a bit simpler, but we'd have a mess of #ifdef and so on
      //until we could drop support for CUDA less than 12.
      //So I am currently taking the shortcut of just calling everything from here
      //and only calculating block sizes once...
    }
}

void ClusterMomentsCalculator::calculateClusterPropertiesAndMoments(CaloRecGPU::EventDataHolder & holder,
                                                                    const ConstantDataHolder & instance_data,
                                                                    const CMCOptionsHolder & options,
                                                                    const IGPUKernelSizeOptimizer & optimizer,
                                                                    const bool synchronize,
                                                                    CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream,
                                                                    const bool defer_instead_of_oversize)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);

  const CUDAKernelLaunchConfiguration cfg_0_clu = optimizer.get_launch_configuration("ClusterMomentsCalculator", 0);
  const CUDAKernelLaunchConfiguration cfg_1_cel = optimizer.get_launch_configuration("ClusterMomentsCalculator", 1);
  const CUDAKernelLaunchConfiguration cfg_1_clu = optimizer.get_launch_configuration("ClusterMomentsCalculator", 2);
  const CUDAKernelLaunchConfiguration cfg_2_cel = optimizer.get_launch_configuration("ClusterMomentsCalculator", 3);
  const CUDAKernelLaunchConfiguration cfg_2_clu = optimizer.get_launch_configuration("ClusterMomentsCalculator", 4);
  const CUDAKernelLaunchConfiguration cfg_3_cel = optimizer.get_launch_configuration("ClusterMomentsCalculator", 5);
  const CUDAKernelLaunchConfiguration cfg_3_clu = optimizer.get_launch_configuration("ClusterMomentsCalculator", 6);
  const CUDAKernelLaunchConfiguration cfg_f_clu = optimizer.get_launch_configuration("ClusterMomentsCalculator", 7);

  if (optimizer.use_minimal_kernel_sizes() && optimizer.can_use_dynamic_parallelism())
    {
      const CUDAKernelLaunchConfiguration cfg_blocks = optimizer.get_launch_configuration("ClusterMomentsCalculator", 0);
      const CUDAKernelLaunchConfiguration cfg_finalize  = optimizer.get_launch_configuration("ClusterMomentsCalculator", 1);

      calculateClusterPropertiesAndMomentsDeferKernel <<< 1, 1, 0, stream_to_use>>>(holder.m_clusters_dev,
                                                                                    holder.m_moments_dev,
                                                                                    holder.m_cell_state_dev,
                                                                                    holder.m_cell_info_dev,
                                                                                    instance_data.m_geometry_dev,
                                                                                    instance_data.m_cell_noise_dev,
                                                                                    options.m_options_dev,
                                                                                    cfg_2_clu.block_x, cfg_3_cel.block_x);
    }
  else
    {

      zerothClusterPassKernel <<< cfg_0_clu.grid_x, cfg_0_clu.block_x, 0, stream_to_use>>>(holder.m_moments_dev,
                                                                                           holder.m_clusters_dev,
                                                                                           instance_data.m_geometry_dev,
                                                                                           options.m_options->skip_invalid_clusters);


      firstCellPassKernel <<< cfg_1_cel.grid_x, cfg_1_cel.block_x, 0, stream_to_use>>>(holder.m_moments_dev,
                                                                                       holder.m_clusters_dev,
                                                                                       holder.m_cell_state_dev,
                                                                                       holder.m_cell_info_dev,
                                                                                       instance_data.m_geometry_dev,
                                                                                       options.m_options->use_abs_energy);
      firstClusterPassKernel <<< cfg_1_clu.grid_x, cfg_1_clu.block_x, 0, stream_to_use>>>(holder.m_moments_dev,
                                                                                          holder.m_clusters_dev,
                                                                                          options.m_options->skip_invalid_clusters);


      secondCellPassKernel <<< cfg_2_cel.grid_x, cfg_2_cel.block_x, 0, stream_to_use>>>(holder.m_moments_dev,
                                                                                        holder.m_cell_state_dev,
                                                                                        holder.m_cell_info_dev,
                                                                                        instance_data.m_geometry_dev,
                                                                                        instance_data.m_cell_noise_dev,
                                                                                        options.m_options->use_abs_energy,
                                                                                        options.m_options->use_two_gaussian_noise,
                                                                                        options.m_options->min_LAr_quality);
      secondClusterPassKernel <<< cfg_2_clu.grid_x, cfg_2_clu.block_x, 0, stream_to_use>>>(holder.m_moments_dev,
                                                                                           holder.m_clusters_dev,
                                                                                           options.m_options->max_axis_angle,
                                                                                           options.m_options->skip_invalid_clusters);


      thirdCellPassKernel <<< cfg_3_cel.grid_x, cfg_3_cel.block_x, 0, stream_to_use>>>(holder.m_moments_dev,
                                                                                       holder.m_cell_state_dev,
                                                                                       holder.m_cell_info_dev,
                                                                                       instance_data.m_geometry_dev,
                                                                                       options.m_options->use_abs_energy,
                                                                                       options.m_options->eta_inner_wheel,
                                                                                       options.m_options->min_l_longitudinal,
                                                                                       options.m_options->min_r_lateral);
      thirdClusterPassKernel <<< cfg_3_clu.grid_x, cfg_3_clu.block_x, 0, stream_to_use>>>(holder.m_moments_dev,
                                                                                          holder.m_clusters_dev,
                                                                                          instance_data.m_geometry_dev,
                                                                                          options.m_options->skip_invalid_clusters);


      finalClusterPassKernel <<< cfg_f_clu.grid_x, cfg_f_clu.block_x, 0, stream_to_use>>>(holder.m_moments_dev,
                                                                                          holder.m_clusters_dev,
                                                                                          holder.m_cell_state_dev,
                                                                                          holder.m_cell_info_dev,
                                                                                          instance_data.m_geometry_dev,
                                                                                          options.m_options->skip_invalid_clusters);


      //clearInvalidCells <<< dimGridCells, dimBlockCells, 0, stream_to_use>>>(holder.m_cell_state_dev, holder.m_clusters_dev);
    }

  if (synchronize)
    {
      CUDA_ERRCHECK(hipPeekAtLastError());
      CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
    }
}

/*******************************************************************************************************************************/

void ClusterMomentsCalculator::register_kernels(IGPUKernelSizeOptimizer & optimizer)
{
  void * kernels[] = { (void *) zerothClusterPassKernel,
                       (void *) firstCellPassKernel,
                       (void *) firstClusterPassKernel,
                       (void *) secondCellPassKernel,
                       (void *) secondClusterPassKernel,
                       (void *) thirdCellPassKernel,
                       (void *) thirdClusterPassKernel,
                       (void *) finalClusterPassKernel
                     };

  int blocksizes[] = { ClusterPassBlockSize,
                       CellPassBlockSize,
                       ClusterPassBlockSize,
                       CellPassBlockSize,
                       ClusterPassBlockSize,
                       CellPassBlockSize,
                       ClusterPassBlockSize,
                       ClusterPassBlockSize,
                     };

  int  gridsizes[] = { Helpers::int_ceil_div(NMaxClusters, Helpers::int_floor_div(ClusterPassBlockSize, WarpSize)),
                       Helpers::int_ceil_div(NCaloCells,   Helpers::int_floor_div(CellPassBlockSize,    WarpSize)),
                       Helpers::int_ceil_div(NMaxClusters, Helpers::int_floor_div(ClusterPassBlockSize, WarpSize)),
                       Helpers::int_ceil_div(NCaloCells,   Helpers::int_floor_div(CellPassBlockSize,    WarpSize)),
                       Helpers::int_ceil_div(NMaxClusters, Helpers::int_floor_div(ClusterPassBlockSize, WarpSize)),
                       Helpers::int_ceil_div(NCaloCells,   Helpers::int_floor_div(CellPassBlockSize,    WarpSize)),
                       Helpers::int_ceil_div(NMaxClusters, Helpers::int_floor_div(ClusterPassBlockSize, WarpSize)),
                       Helpers::int_ceil_div(NMaxClusters, Helpers::int_floor_div(ClusterPassBlockSize, WarpSize)),
                     };

  int   maxsizes[] = { NMaxClusters * WarpSize,
                       NCaloCells   * WarpSize,
                       NMaxClusters * WarpSize,
                       NCaloCells   * WarpSize,
                       NMaxClusters * WarpSize,
                       NCaloCells   * WarpSize,
                       NMaxClusters * WarpSize,
                       NMaxClusters * WarpSize
                     };
                     
  optimizer.register_kernels("ClusterMomentsCalculator", 8, kernels, blocksizes, gridsizes, maxsizes);
}