//
// Copyright (C) 2002-2023 CERN for the benefit of the ATLAS collaboration
//
// Dear emacs, this is -*- c++ -*-
//

#include "CaloRecGPU/Helpers.h"

void * CaloRecGPU::CUDA_Helpers::allocate(const size_t num)
{
  void * ret;
  CUDA_ERRCHECK(hipMalloc(&ret, num));
  return ret;
}

void CaloRecGPU::CUDA_Helpers::deallocate(void * address)
{
  CUDA_ERRCHECK(hipFree(address));
}


void * CaloRecGPU::CUDA_Helpers::allocate_pinned(const size_t num)
{
  void * ret;
  CUDA_ERRCHECK(hipHostMalloc(&ret, num));
  return ret;
}

void CaloRecGPU::CUDA_Helpers::deallocate_pinned(void * address)
{
  CUDA_ERRCHECK(hipHostFree(address));
}


void CaloRecGPU::CUDA_Helpers::GPU_to_CPU(void * dest, const void * const source, const size_t num)
{
  CUDA_ERRCHECK(hipMemcpy(dest, source, num, hipMemcpyDeviceToHost));
}

void CaloRecGPU::CUDA_Helpers::CPU_to_GPU(void * dest, const void * const source, const size_t num)
{
  CUDA_ERRCHECK(hipMemcpy(dest, source, num, hipMemcpyHostToDevice));
}

void CaloRecGPU::CUDA_Helpers::GPU_to_GPU(void * dest, const void * const source, const size_t num)
{
  CUDA_ERRCHECK(hipMemcpy(dest, source, num, hipMemcpyDeviceToDevice));
}



void CaloRecGPU::CUDA_Helpers::GPU_to_CPU_async(void * dest, const void * const source, const size_t num, CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  CUDA_ERRCHECK(hipMemcpyAsync(dest, source, num, hipMemcpyDeviceToHost, hipStreamPerThread));
}

void CaloRecGPU::CUDA_Helpers::CPU_to_GPU_async(void * dest, const void * const source, const size_t num, CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  CUDA_ERRCHECK(hipMemcpyAsync(dest, source, num, hipMemcpyHostToDevice, stream_to_use));
}

void CaloRecGPU::CUDA_Helpers::GPU_to_GPU_async(void * dest, const void * const source, const size_t num, CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  CUDA_ERRCHECK(hipMemcpyAsync(dest, source, num, hipMemcpyDeviceToDevice, stream_to_use));
}

void CaloRecGPU::CUDA_Helpers::GPU_synchronize(CaloRecGPU::CUDA_Helpers::CUDAStreamPtrHolder stream)
{
  const hipStream_t & stream_to_use = (stream != nullptr ? * ((hipStream_t *) stream) : hipStreamPerThread);
  CUDA_ERRCHECK(hipStreamSynchronize(stream_to_use));
}

void CaloRecGPU::CUDA_Helpers::optimize_block_and_grid_size(void * func, int & block_size, int & grid_size, const int dynamic_memory, const int block_size_limit)
{
  CUDA_ERRCHECK(hipOccupancyMaxPotentialBlockSize(&grid_size, &block_size, func, dynamic_memory, block_size_limit));
}

void CaloRecGPU::CUDA_Helpers::optimize_block_and_grid_size_for_cooperative_launch(void * func, int & block_size, int & grid_size, const int dynamic_memory, const int block_size_limit)
{
  CUDA_ERRCHECK(hipOccupancyMaxPotentialBlockSize(&grid_size, &block_size, func, dynamic_memory, block_size_limit));
  CUDA_ERRCHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&grid_size, func, block_size, dynamic_memory));

  int multi_processor_count = 1;

  CUDA_ERRCHECK(hipDeviceGetAttribute(&multi_processor_count, hipDeviceAttributeMultiprocessorCount, 0));

  grid_size *= multi_processor_count;
}


bool CaloRecGPU::CUDA_Helpers::supports_cooperative_launches()
{
  int ret = 0;
  
  CUDA_ERRCHECK(hipDeviceGetAttribute(&ret, hipDeviceAttributeCooperativeLaunch, 0));
  
  return ret;
}

bool CaloRecGPU::CUDA_Helpers::supports_dynamic_parallelism()
{
  int major = 0, minor = 0;
  
  CUDA_ERRCHECK(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, 0));
  CUDA_ERRCHECK(hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, 0));
  
  return (major > 3 || (major == 3 && minor >= 5));

}

std::string CaloRecGPU::CUDA_Helpers::GPU_name()
{
  hipDeviceProp_t properties;
  CUDA_ERRCHECK(hipGetDeviceProperties(&properties, 0));
  return properties.name;
}